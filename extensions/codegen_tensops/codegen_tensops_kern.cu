
#include <hip/hip_runtime.h>

#define WARPSZ 32
#define MODWARP(X) (X & 0x1f)



__global__
void fused_tensor_prods_example_kern(
    // <<<(batch), (WARPSZ, dim_l)>>>
    int p_0, int p_1_base, int p_1, int p_2_base, int p_2, 
    int batch, int dim_l, int dim_0, int dim_1, int dim_2,
    const float* x_0, const float* x_1, const float* x_2, const float* P_000, const float* left_000, const float* P_011, const float* left_011, const float* P_101, const float* left_101, const float* P_110, const float* left_110, const float* P_220, const float* left_220, const float* P_222, const float* left_222, const float* P_211, const float* left_211, const float* P_111, const float* left_111, const float* P_212, const float* left_212,
    float* __restrict__ y_0, float* __restrict__ y_1, float* __restrict__ y_2) {
  extern __shared__ float s[];
  float* product_000 = &s[0*p_0]; // size = 1*p_0
  float* product_011 = &s[p_1_base + 0*p_1]; // size = 3*p_1
  float* product_101 = &s[1*p_0]; // size = 3*p_0
  float* product_110 = &s[p_1_base + 3*p_1]; // size = 1*p_1
  float* product_220 = &s[p_2_base + 0*p_2]; // size = 1*p_2
  float* product_222 = &s[p_2_base + 1*p_2]; // size = 9*p_2
  float* product_211 = &s[p_1_base + 4*p_1]; // size = 3*p_1
  float* product_111 = &s[p_1_base + 7*p_1]; // size = 3*p_1
  float* product_212 = &s[p_1_base + 10*p_1]; // size = 9*p_1
  for (int idx_batch = blockIdx.x; idx_batch < batch; idx_batch += gridDim.x) {
    { // compute tensor products
      float l_000_0 = left_000[((idx_batch)*dim_l + threadIdx.y)*1 + 0];
      for (int idx_chan_in_000 = threadIdx.x; idx_chan_in_000 < dim_0; idx_chan_in_000 += blockDim.x) {
        product_000[((threadIdx.y)*dim_0 + idx_chan_in_000)*1 + 0] = l_000_0*x_0[((idx_batch)*dim_0 + idx_chan_in_000)*1 + 0];
      }
      float l_011_0 = left_011[((idx_batch)*dim_l + threadIdx.y)*1 + 0];
      for (int idx_chan_in_011 = threadIdx.x; idx_chan_in_011 < dim_1; idx_chan_in_011 += blockDim.x) {
        product_011[((threadIdx.y)*dim_1 + idx_chan_in_011)*3 + 0] = l_011_0*x_1[((idx_batch)*dim_1 + idx_chan_in_011)*3 + 0];
        product_011[((threadIdx.y)*dim_1 + idx_chan_in_011)*3 + 1] = l_011_0*x_1[((idx_batch)*dim_1 + idx_chan_in_011)*3 + 1];
        product_011[((threadIdx.y)*dim_1 + idx_chan_in_011)*3 + 2] = l_011_0*x_1[((idx_batch)*dim_1 + idx_chan_in_011)*3 + 2];
      }
      float l_101_0 = left_101[((idx_batch)*dim_l + threadIdx.y)*3 + 0];
      float l_101_1 = left_101[((idx_batch)*dim_l + threadIdx.y)*3 + 1];
      float l_101_2 = left_101[((idx_batch)*dim_l + threadIdx.y)*3 + 2];
      for (int idx_chan_in_101 = threadIdx.x; idx_chan_in_101 < dim_0; idx_chan_in_101 += blockDim.x) {
        product_101[((threadIdx.y)*dim_0 + idx_chan_in_101)*3 + 0] = l_101_0*x_0[((idx_batch)*dim_0 + idx_chan_in_101)*1 + 0];
        product_101[((threadIdx.y)*dim_0 + idx_chan_in_101)*3 + 1] = l_101_1*x_0[((idx_batch)*dim_0 + idx_chan_in_101)*1 + 0];
        product_101[((threadIdx.y)*dim_0 + idx_chan_in_101)*3 + 2] = l_101_2*x_0[((idx_batch)*dim_0 + idx_chan_in_101)*1 + 0];
      }
      float l_110_0 = left_110[((idx_batch)*dim_l + threadIdx.y)*3 + 0];
      float l_110_1 = left_110[((idx_batch)*dim_l + threadIdx.y)*3 + 1];
      float l_110_2 = left_110[((idx_batch)*dim_l + threadIdx.y)*3 + 2];
      for (int idx_chan_in_110 = threadIdx.x; idx_chan_in_110 < dim_1; idx_chan_in_110 += blockDim.x) {
        product_110[((threadIdx.y)*dim_1 + idx_chan_in_110)*1 + 0] = l_110_0*x_1[((idx_batch)*dim_1 + idx_chan_in_110)*3 + 0] + l_110_1*x_1[((idx_batch)*dim_1 + idx_chan_in_110)*3 + 1] + l_110_2*x_1[((idx_batch)*dim_1 + idx_chan_in_110)*3 + 2];
      }
      float l_220_0 = left_220[((idx_batch)*dim_l + threadIdx.y)*9 + 0];
      float l_220_1 = left_220[((idx_batch)*dim_l + threadIdx.y)*9 + 1];
      float l_220_2 = left_220[((idx_batch)*dim_l + threadIdx.y)*9 + 2];
      float l_220_3 = left_220[((idx_batch)*dim_l + threadIdx.y)*9 + 3];
      float l_220_4 = left_220[((idx_batch)*dim_l + threadIdx.y)*9 + 4];
      float l_220_5 = left_220[((idx_batch)*dim_l + threadIdx.y)*9 + 5];
      float l_220_6 = left_220[((idx_batch)*dim_l + threadIdx.y)*9 + 6];
      float l_220_7 = left_220[((idx_batch)*dim_l + threadIdx.y)*9 + 7];
      float l_220_8 = left_220[((idx_batch)*dim_l + threadIdx.y)*9 + 8];
      for (int idx_chan_in_220 = threadIdx.x; idx_chan_in_220 < dim_2; idx_chan_in_220 += blockDim.x) {
        product_220[((threadIdx.y)*dim_2 + idx_chan_in_220)*1 + 0] = l_220_0*x_2[((idx_batch)*dim_2 + idx_chan_in_220)*9 + 0] + l_220_1*x_2[((idx_batch)*dim_2 + idx_chan_in_220)*9 + 1] + l_220_2*x_2[((idx_batch)*dim_2 + idx_chan_in_220)*9 + 2] + l_220_3*x_2[((idx_batch)*dim_2 + idx_chan_in_220)*9 + 3] + l_220_4*x_2[((idx_batch)*dim_2 + idx_chan_in_220)*9 + 4] + l_220_5*x_2[((idx_batch)*dim_2 + idx_chan_in_220)*9 + 5] + l_220_6*x_2[((idx_batch)*dim_2 + idx_chan_in_220)*9 + 6] + l_220_7*x_2[((idx_batch)*dim_2 + idx_chan_in_220)*9 + 7] + l_220_8*x_2[((idx_batch)*dim_2 + idx_chan_in_220)*9 + 8];
      }
      float l_222_0 = left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 0];
      float l_222_1 = left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 1];
      float l_222_2 = left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 2];
      float l_222_3 = left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 3];
      float l_222_4 = left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 4];
      float l_222_5 = left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 5];
      float l_222_6 = left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 6];
      float l_222_7 = left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 7];
      float l_222_8 = left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 8];
      for (int idx_chan_in_222 = threadIdx.x; idx_chan_in_222 < dim_2; idx_chan_in_222 += blockDim.x) {
        product_222[((threadIdx.y)*dim_2 + idx_chan_in_222)*9 + 0] = l_222_0*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 0] + l_222_1*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 1] + l_222_2*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 2];
        product_222[((threadIdx.y)*dim_2 + idx_chan_in_222)*9 + 1] = l_222_0*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 3] + l_222_1*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 4] + l_222_2*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 5];
        product_222[((threadIdx.y)*dim_2 + idx_chan_in_222)*9 + 2] = l_222_0*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 6] + l_222_1*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 7] + l_222_2*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 8];
        product_222[((threadIdx.y)*dim_2 + idx_chan_in_222)*9 + 3] = l_222_3*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 0] + l_222_4*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 1] + l_222_5*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 2];
        product_222[((threadIdx.y)*dim_2 + idx_chan_in_222)*9 + 4] = l_222_3*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 3] + l_222_4*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 4] + l_222_5*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 5];
        product_222[((threadIdx.y)*dim_2 + idx_chan_in_222)*9 + 5] = l_222_3*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 6] + l_222_4*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 7] + l_222_5*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 8];
        product_222[((threadIdx.y)*dim_2 + idx_chan_in_222)*9 + 6] = l_222_6*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 0] + l_222_7*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 1] + l_222_8*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 2];
        product_222[((threadIdx.y)*dim_2 + idx_chan_in_222)*9 + 7] = l_222_6*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 3] + l_222_7*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 4] + l_222_8*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 5];
        product_222[((threadIdx.y)*dim_2 + idx_chan_in_222)*9 + 8] = l_222_6*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 6] + l_222_7*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 7] + l_222_8*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 8];
      }
      float l_211_0 = left_211[((idx_batch)*dim_l + threadIdx.y)*9 + 0];
      float l_211_1 = left_211[((idx_batch)*dim_l + threadIdx.y)*9 + 1];
      float l_211_2 = left_211[((idx_batch)*dim_l + threadIdx.y)*9 + 2];
      float l_211_3 = left_211[((idx_batch)*dim_l + threadIdx.y)*9 + 3];
      float l_211_4 = left_211[((idx_batch)*dim_l + threadIdx.y)*9 + 4];
      float l_211_5 = left_211[((idx_batch)*dim_l + threadIdx.y)*9 + 5];
      float l_211_6 = left_211[((idx_batch)*dim_l + threadIdx.y)*9 + 6];
      float l_211_7 = left_211[((idx_batch)*dim_l + threadIdx.y)*9 + 7];
      float l_211_8 = left_211[((idx_batch)*dim_l + threadIdx.y)*9 + 8];
      for (int idx_chan_in_211 = threadIdx.x; idx_chan_in_211 < dim_1; idx_chan_in_211 += blockDim.x) {
        product_211[((threadIdx.y)*dim_1 + idx_chan_in_211)*3 + 0] = l_211_0*x_1[((idx_batch)*dim_1 + idx_chan_in_211)*3 + 0] + l_211_1*x_1[((idx_batch)*dim_1 + idx_chan_in_211)*3 + 1] + l_211_2*x_1[((idx_batch)*dim_1 + idx_chan_in_211)*3 + 2];
        product_211[((threadIdx.y)*dim_1 + idx_chan_in_211)*3 + 1] = l_211_3*x_1[((idx_batch)*dim_1 + idx_chan_in_211)*3 + 0] + l_211_4*x_1[((idx_batch)*dim_1 + idx_chan_in_211)*3 + 1] + l_211_5*x_1[((idx_batch)*dim_1 + idx_chan_in_211)*3 + 2];
        product_211[((threadIdx.y)*dim_1 + idx_chan_in_211)*3 + 2] = l_211_6*x_1[((idx_batch)*dim_1 + idx_chan_in_211)*3 + 0] + l_211_7*x_1[((idx_batch)*dim_1 + idx_chan_in_211)*3 + 1] + l_211_8*x_1[((idx_batch)*dim_1 + idx_chan_in_211)*3 + 2];
      }
      float l_111_0 = left_111[((idx_batch)*dim_l + threadIdx.y)*3 + 0];
      float l_111_1 = left_111[((idx_batch)*dim_l + threadIdx.y)*3 + 1];
      float l_111_2 = left_111[((idx_batch)*dim_l + threadIdx.y)*3 + 2];
      for (int idx_chan_in_111 = threadIdx.x; idx_chan_in_111 < dim_1; idx_chan_in_111 += blockDim.x) {
        product_111[((threadIdx.y)*dim_1 + idx_chan_in_111)*3 + 0] = l_111_1*x_1[((idx_batch)*dim_1 + idx_chan_in_111)*3 + 2] + (-1)*l_111_2*x_1[((idx_batch)*dim_1 + idx_chan_in_111)*3 + 1];
        product_111[((threadIdx.y)*dim_1 + idx_chan_in_111)*3 + 1] = (-1)*l_111_0*x_1[((idx_batch)*dim_1 + idx_chan_in_111)*3 + 2] + l_111_2*x_1[((idx_batch)*dim_1 + idx_chan_in_111)*3 + 0];
        product_111[((threadIdx.y)*dim_1 + idx_chan_in_111)*3 + 2] = l_111_0*x_1[((idx_batch)*dim_1 + idx_chan_in_111)*3 + 1] + (-1)*l_111_1*x_1[((idx_batch)*dim_1 + idx_chan_in_111)*3 + 0];
      }
      float l_212_0 = left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 0];
      float l_212_1 = left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 1];
      float l_212_2 = left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 2];
      float l_212_3 = left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 3];
      float l_212_4 = left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 4];
      float l_212_5 = left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 5];
      float l_212_6 = left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 6];
      float l_212_7 = left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 7];
      float l_212_8 = left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 8];
      for (int idx_chan_in_212 = threadIdx.x; idx_chan_in_212 < dim_1; idx_chan_in_212 += blockDim.x) {
        product_212[((threadIdx.y)*dim_1 + idx_chan_in_212)*9 + 0] = l_212_1*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 2] + (-1)*l_212_2*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 1];
        product_212[((threadIdx.y)*dim_1 + idx_chan_in_212)*9 + 1] = (-1)*l_212_0*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 2] + l_212_2*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 0];
        product_212[((threadIdx.y)*dim_1 + idx_chan_in_212)*9 + 2] = l_212_0*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 1] + (-1)*l_212_1*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 0];
        product_212[((threadIdx.y)*dim_1 + idx_chan_in_212)*9 + 3] = l_212_4*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 2] + (-1)*l_212_5*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 1];
        product_212[((threadIdx.y)*dim_1 + idx_chan_in_212)*9 + 4] = (-1)*l_212_3*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 2] + l_212_5*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 0];
        product_212[((threadIdx.y)*dim_1 + idx_chan_in_212)*9 + 5] = l_212_3*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 1] + (-1)*l_212_4*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 0];
        product_212[((threadIdx.y)*dim_1 + idx_chan_in_212)*9 + 6] = l_212_7*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 2] + (-1)*l_212_8*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 1];
        product_212[((threadIdx.y)*dim_1 + idx_chan_in_212)*9 + 7] = (-1)*l_212_6*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 2] + l_212_8*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 0];
        product_212[((threadIdx.y)*dim_1 + idx_chan_in_212)*9 + 8] = l_212_6*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 1] + (-1)*l_212_7*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 0];
      }
    }
    __syncthreads();
    { // linear transforms to compute the outputs
      for (int idx_chan_out_0 = threadIdx.y; idx_chan_out_0 < dim_0; idx_chan_out_0 += blockDim.y) {
        float y_o_0_0 = 0.0;
        float accum_000_0 = 0.0;
        for (int idx_chan_in_000 = threadIdx.x; idx_chan_in_000 < dim_l*dim_0; idx_chan_in_000 += blockDim.x) {
          float P_oi_000 = P_000[(idx_chan_out_0)*dim_l*dim_0 + idx_chan_in_000];
          accum_000_0 += P_oi_000*product_000[(idx_chan_in_000)*1 + 0];
        }
        // reduce across the warp so that first thread in warp will have the sum 
        for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
          accum_000_0 += __shfl_down_sync(0xffffffff, accum_000_0, offset);
        }
        if (threadIdx.x == 0) {
          y_o_0_0 += accum_000_0;
        }
        float accum_110_0 = 0.0;
        for (int idx_chan_in_110 = threadIdx.x; idx_chan_in_110 < dim_l*dim_1; idx_chan_in_110 += blockDim.x) {
          float P_oi_110 = P_110[(idx_chan_out_0)*dim_l*dim_1 + idx_chan_in_110];
          accum_110_0 += P_oi_110*product_110[(idx_chan_in_110)*1 + 0];
        }
        // reduce across the warp so that first thread in warp will have the sum 
        for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
          accum_110_0 += __shfl_down_sync(0xffffffff, accum_110_0, offset);
        }
        if (threadIdx.x == 0) {
          y_o_0_0 += accum_110_0;
        }
        float accum_220_0 = 0.0;
        for (int idx_chan_in_220 = threadIdx.x; idx_chan_in_220 < dim_l*dim_2; idx_chan_in_220 += blockDim.x) {
          float P_oi_220 = P_220[(idx_chan_out_0)*dim_l*dim_2 + idx_chan_in_220];
          accum_220_0 += P_oi_220*product_220[(idx_chan_in_220)*1 + 0];
        }
        // reduce across the warp so that first thread in warp will have the sum 
        for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
          accum_220_0 += __shfl_down_sync(0xffffffff, accum_220_0, offset);
        }
        if (threadIdx.x == 0) {
          y_o_0_0 += accum_220_0;
        }
        if (threadIdx.x == 0) {
          y_0[((idx_batch)*dim_0 + idx_chan_out_0)*1 + 0] = y_o_0_0;
        }
      }
      for (int idx_chan_out_1 = threadIdx.y; idx_chan_out_1 < dim_1; idx_chan_out_1 += blockDim.y) {
        float y_o_1_0 = 0.0;
        float y_o_1_1 = 0.0;
        float y_o_1_2 = 0.0;
        float accum_011_0 = 0.0;
        float accum_011_1 = 0.0;
        float accum_011_2 = 0.0;
        for (int idx_chan_in_011 = threadIdx.x; idx_chan_in_011 < dim_l*dim_1; idx_chan_in_011 += blockDim.x) {
          float P_oi_011 = P_011[(idx_chan_out_1)*dim_l*dim_1 + idx_chan_in_011];
          accum_011_0 += P_oi_011*product_011[(idx_chan_in_011)*3 + 0];
          accum_011_1 += P_oi_011*product_011[(idx_chan_in_011)*3 + 1];
          accum_011_2 += P_oi_011*product_011[(idx_chan_in_011)*3 + 2];
        }
        // reduce across the warp so that first thread in warp will have the sum 
        for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
          accum_011_0 += __shfl_down_sync(0xffffffff, accum_011_0, offset);
          accum_011_1 += __shfl_down_sync(0xffffffff, accum_011_1, offset);
          accum_011_2 += __shfl_down_sync(0xffffffff, accum_011_2, offset);
        }
        if (threadIdx.x == 0) {
          y_o_1_0 += accum_011_0;
          y_o_1_1 += accum_011_1;
          y_o_1_2 += accum_011_2;
        }
        float accum_101_0 = 0.0;
        float accum_101_1 = 0.0;
        float accum_101_2 = 0.0;
        for (int idx_chan_in_101 = threadIdx.x; idx_chan_in_101 < dim_l*dim_0; idx_chan_in_101 += blockDim.x) {
          float P_oi_101 = P_101[(idx_chan_out_1)*dim_l*dim_0 + idx_chan_in_101];
          accum_101_0 += P_oi_101*product_101[(idx_chan_in_101)*3 + 0];
          accum_101_1 += P_oi_101*product_101[(idx_chan_in_101)*3 + 1];
          accum_101_2 += P_oi_101*product_101[(idx_chan_in_101)*3 + 2];
        }
        // reduce across the warp so that first thread in warp will have the sum 
        for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
          accum_101_0 += __shfl_down_sync(0xffffffff, accum_101_0, offset);
          accum_101_1 += __shfl_down_sync(0xffffffff, accum_101_1, offset);
          accum_101_2 += __shfl_down_sync(0xffffffff, accum_101_2, offset);
        }
        if (threadIdx.x == 0) {
          y_o_1_0 += accum_101_0;
          y_o_1_1 += accum_101_1;
          y_o_1_2 += accum_101_2;
        }
        float accum_211_0 = 0.0;
        float accum_211_1 = 0.0;
        float accum_211_2 = 0.0;
        for (int idx_chan_in_211 = threadIdx.x; idx_chan_in_211 < dim_l*dim_1; idx_chan_in_211 += blockDim.x) {
          float P_oi_211 = P_211[(idx_chan_out_1)*dim_l*dim_1 + idx_chan_in_211];
          accum_211_0 += P_oi_211*product_211[(idx_chan_in_211)*3 + 0];
          accum_211_1 += P_oi_211*product_211[(idx_chan_in_211)*3 + 1];
          accum_211_2 += P_oi_211*product_211[(idx_chan_in_211)*3 + 2];
        }
        // reduce across the warp so that first thread in warp will have the sum 
        for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
          accum_211_0 += __shfl_down_sync(0xffffffff, accum_211_0, offset);
          accum_211_1 += __shfl_down_sync(0xffffffff, accum_211_1, offset);
          accum_211_2 += __shfl_down_sync(0xffffffff, accum_211_2, offset);
        }
        if (threadIdx.x == 0) {
          y_o_1_0 += accum_211_0;
          y_o_1_1 += accum_211_1;
          y_o_1_2 += accum_211_2;
        }
        float accum_111_0 = 0.0;
        float accum_111_1 = 0.0;
        float accum_111_2 = 0.0;
        for (int idx_chan_in_111 = threadIdx.x; idx_chan_in_111 < dim_l*dim_1; idx_chan_in_111 += blockDim.x) {
          float P_oi_111 = P_111[(idx_chan_out_1)*dim_l*dim_1 + idx_chan_in_111];
          accum_111_0 += P_oi_111*product_111[(idx_chan_in_111)*3 + 0];
          accum_111_1 += P_oi_111*product_111[(idx_chan_in_111)*3 + 1];
          accum_111_2 += P_oi_111*product_111[(idx_chan_in_111)*3 + 2];
        }
        // reduce across the warp so that first thread in warp will have the sum 
        for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
          accum_111_0 += __shfl_down_sync(0xffffffff, accum_111_0, offset);
          accum_111_1 += __shfl_down_sync(0xffffffff, accum_111_1, offset);
          accum_111_2 += __shfl_down_sync(0xffffffff, accum_111_2, offset);
        }
        if (threadIdx.x == 0) {
          y_o_1_0 += accum_111_0;
          y_o_1_1 += accum_111_1;
          y_o_1_2 += accum_111_2;
        }
        if (threadIdx.x == 0) {
          y_1[((idx_batch)*dim_1 + idx_chan_out_1)*3 + 0] = y_o_1_0;
          y_1[((idx_batch)*dim_1 + idx_chan_out_1)*3 + 1] = y_o_1_1;
          y_1[((idx_batch)*dim_1 + idx_chan_out_1)*3 + 2] = y_o_1_2;
        }
      }
      for (int idx_chan_out_2 = threadIdx.y; idx_chan_out_2 < dim_2; idx_chan_out_2 += blockDim.y) {
        float y_o_2_0 = 0.0;
        float y_o_2_1 = 0.0;
        float y_o_2_2 = 0.0;
        float y_o_2_3 = 0.0;
        float y_o_2_4 = 0.0;
        float y_o_2_5 = 0.0;
        float y_o_2_6 = 0.0;
        float y_o_2_7 = 0.0;
        float y_o_2_8 = 0.0;
        float accum_222_0 = 0.0;
        float accum_222_1 = 0.0;
        float accum_222_2 = 0.0;
        float accum_222_3 = 0.0;
        float accum_222_4 = 0.0;
        float accum_222_5 = 0.0;
        float accum_222_6 = 0.0;
        float accum_222_7 = 0.0;
        float accum_222_8 = 0.0;
        for (int idx_chan_in_222 = threadIdx.x; idx_chan_in_222 < dim_l*dim_2; idx_chan_in_222 += blockDim.x) {
          float P_oi_222 = P_222[(idx_chan_out_2)*dim_l*dim_2 + idx_chan_in_222];
          accum_222_0 += P_oi_222*product_222[(idx_chan_in_222)*9 + 0];
          accum_222_1 += P_oi_222*product_222[(idx_chan_in_222)*9 + 1];
          accum_222_2 += P_oi_222*product_222[(idx_chan_in_222)*9 + 2];
          accum_222_3 += P_oi_222*product_222[(idx_chan_in_222)*9 + 3];
          accum_222_4 += P_oi_222*product_222[(idx_chan_in_222)*9 + 4];
          accum_222_5 += P_oi_222*product_222[(idx_chan_in_222)*9 + 5];
          accum_222_6 += P_oi_222*product_222[(idx_chan_in_222)*9 + 6];
          accum_222_7 += P_oi_222*product_222[(idx_chan_in_222)*9 + 7];
          accum_222_8 += P_oi_222*product_222[(idx_chan_in_222)*9 + 8];
        }
        // reduce across the warp so that first thread in warp will have the sum 
        for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
          accum_222_0 += __shfl_down_sync(0xffffffff, accum_222_0, offset);
          accum_222_1 += __shfl_down_sync(0xffffffff, accum_222_1, offset);
          accum_222_2 += __shfl_down_sync(0xffffffff, accum_222_2, offset);
          accum_222_3 += __shfl_down_sync(0xffffffff, accum_222_3, offset);
          accum_222_4 += __shfl_down_sync(0xffffffff, accum_222_4, offset);
          accum_222_5 += __shfl_down_sync(0xffffffff, accum_222_5, offset);
          accum_222_6 += __shfl_down_sync(0xffffffff, accum_222_6, offset);
          accum_222_7 += __shfl_down_sync(0xffffffff, accum_222_7, offset);
          accum_222_8 += __shfl_down_sync(0xffffffff, accum_222_8, offset);
        }
        if (threadIdx.x == 0) {
          y_o_2_0 += accum_222_0;
          y_o_2_1 += accum_222_1;
          y_o_2_2 += accum_222_2;
          y_o_2_3 += accum_222_3;
          y_o_2_4 += accum_222_4;
          y_o_2_5 += accum_222_5;
          y_o_2_6 += accum_222_6;
          y_o_2_7 += accum_222_7;
          y_o_2_8 += accum_222_8;
        }
        float accum_212_0 = 0.0;
        float accum_212_1 = 0.0;
        float accum_212_2 = 0.0;
        float accum_212_3 = 0.0;
        float accum_212_4 = 0.0;
        float accum_212_5 = 0.0;
        float accum_212_6 = 0.0;
        float accum_212_7 = 0.0;
        float accum_212_8 = 0.0;
        for (int idx_chan_in_212 = threadIdx.x; idx_chan_in_212 < dim_l*dim_1; idx_chan_in_212 += blockDim.x) {
          float P_oi_212 = P_212[(idx_chan_out_2)*dim_l*dim_1 + idx_chan_in_212];
          accum_212_0 += P_oi_212*product_212[(idx_chan_in_212)*9 + 0];
          accum_212_1 += P_oi_212*product_212[(idx_chan_in_212)*9 + 1];
          accum_212_2 += P_oi_212*product_212[(idx_chan_in_212)*9 + 2];
          accum_212_3 += P_oi_212*product_212[(idx_chan_in_212)*9 + 3];
          accum_212_4 += P_oi_212*product_212[(idx_chan_in_212)*9 + 4];
          accum_212_5 += P_oi_212*product_212[(idx_chan_in_212)*9 + 5];
          accum_212_6 += P_oi_212*product_212[(idx_chan_in_212)*9 + 6];
          accum_212_7 += P_oi_212*product_212[(idx_chan_in_212)*9 + 7];
          accum_212_8 += P_oi_212*product_212[(idx_chan_in_212)*9 + 8];
        }
        // reduce across the warp so that first thread in warp will have the sum 
        for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
          accum_212_0 += __shfl_down_sync(0xffffffff, accum_212_0, offset);
          accum_212_1 += __shfl_down_sync(0xffffffff, accum_212_1, offset);
          accum_212_2 += __shfl_down_sync(0xffffffff, accum_212_2, offset);
          accum_212_3 += __shfl_down_sync(0xffffffff, accum_212_3, offset);
          accum_212_4 += __shfl_down_sync(0xffffffff, accum_212_4, offset);
          accum_212_5 += __shfl_down_sync(0xffffffff, accum_212_5, offset);
          accum_212_6 += __shfl_down_sync(0xffffffff, accum_212_6, offset);
          accum_212_7 += __shfl_down_sync(0xffffffff, accum_212_7, offset);
          accum_212_8 += __shfl_down_sync(0xffffffff, accum_212_8, offset);
        }
        if (threadIdx.x == 0) {
          y_o_2_0 += accum_212_0;
          y_o_2_1 += accum_212_1;
          y_o_2_2 += accum_212_2;
          y_o_2_3 += accum_212_3;
          y_o_2_4 += accum_212_4;
          y_o_2_5 += accum_212_5;
          y_o_2_6 += accum_212_6;
          y_o_2_7 += accum_212_7;
          y_o_2_8 += accum_212_8;
        }
        if (threadIdx.x == 0) {
          y_2[((idx_batch)*dim_2 + idx_chan_out_2)*9 + 0] = y_o_2_0;
          y_2[((idx_batch)*dim_2 + idx_chan_out_2)*9 + 1] = y_o_2_1;
          y_2[((idx_batch)*dim_2 + idx_chan_out_2)*9 + 2] = y_o_2_2;
          y_2[((idx_batch)*dim_2 + idx_chan_out_2)*9 + 3] = y_o_2_3;
          y_2[((idx_batch)*dim_2 + idx_chan_out_2)*9 + 4] = y_o_2_4;
          y_2[((idx_batch)*dim_2 + idx_chan_out_2)*9 + 5] = y_o_2_5;
          y_2[((idx_batch)*dim_2 + idx_chan_out_2)*9 + 6] = y_o_2_6;
          y_2[((idx_batch)*dim_2 + idx_chan_out_2)*9 + 7] = y_o_2_7;
          y_2[((idx_batch)*dim_2 + idx_chan_out_2)*9 + 8] = y_o_2_8;
        }
      }
    }
  }
}


void fused_tensor_prods_example(
    int batch, int dim_l, int dim_0, int dim_1, int dim_2,
    const float* x_0, const float* x_1, const float* x_2, const float* P_000, const float* left_000, const float* P_011, const float* left_011, const float* P_101, const float* left_101, const float* P_110, const float* left_110, const float* P_220, const float* left_220, const float* P_222, const float* left_222, const float* P_211, const float* left_211, const float* P_111, const float* left_111, const float* P_212, const float* left_212,
    float* y_0, float* y_1, float* y_2) {
  
  int p_0 = dim_l*dim_0;
  int p_1 = dim_l*dim_1;
  int p_2 = dim_l*dim_2;
  int sharedmemsz = 0;
  sharedmemsz += 4*p_0;
  int p_1_base = sharedmemsz;
  sharedmemsz += 19*p_1;
  int p_2_base = sharedmemsz;
  sharedmemsz += 10*p_2;
  dim3 gridsz = dim3(batch);
  dim3 blocksz = dim3(WARPSZ, dim_l);
  fused_tensor_prods_example_kern<<<gridsz, blocksz, sharedmemsz*sizeof(float)>>>(
      p_0, p_1_base, p_1, p_2_base, p_2, 
      batch, dim_l, dim_0, dim_1, dim_2,
      x_0, x_1, x_2, P_000, left_000, P_011, left_011, P_101, left_101, P_110, left_110, P_220, left_220, P_222, left_222, P_211, left_211, P_111, left_111, P_212, left_212,
      y_0, y_1, y_2);
  
}


__global__
void fused_tensor_prods_example_backward_kern(
    // <<<(batch), (WARPSZ, dim_l)>>>
    int p_0, int p_1_base, int p_1, int p_2_base, int p_2, 
    int batch, int dim_l, int dim_0, int dim_1, int dim_2,
    const float* dy_0, const float* dy_1, const float* dy_2, const float* P_000, const float* left_000, const float* P_011, const float* left_011, const float* P_101, const float* left_101, const float* P_110, const float* left_110, const float* P_220, const float* left_220, const float* P_222, const float* left_222, const float* P_211, const float* left_211, const float* P_111, const float* left_111, const float* P_212, const float* left_212,
    float* __restrict__ dx_0, float* __restrict__ dx_1, float* __restrict__ dx_2) {
  extern __shared__ float s[];
  float* dproduct_000 = &s[0*p_0]; // size = 1*p_0
  float* dproduct_011 = &s[p_1_base + 0*p_1]; // size = 3*p_1
  float* dproduct_101 = &s[p_1_base + 3*p_1]; // size = 1*p_1
  float* dproduct_110 = &s[1*p_0]; // size = 3*p_0
  float* dproduct_220 = &s[4*p_0]; // size = 9*p_0
  float* dproduct_222 = &s[p_2_base + 0*p_2]; // size = 9*p_2
  float* dproduct_211 = &s[p_1_base + 4*p_1]; // size = 3*p_1
  float* dproduct_111 = &s[p_1_base + 7*p_1]; // size = 3*p_1
  float* dproduct_212 = &s[p_2_base + 9*p_2]; // size = 3*p_2
  for (int idx_batch = blockIdx.x; idx_batch < batch; idx_batch += gridDim.x) {
    { // compute tensor products
      float l_000_0 = left_000[((idx_batch)*dim_l + threadIdx.y)*1 + 0];
      for (int idx_chan_out_000 = threadIdx.x; idx_chan_out_000 < dim_0; idx_chan_out_000 += blockDim.x) {
        dproduct_000[((threadIdx.y)*dim_0 + idx_chan_out_000)*1 + 0] = l_000_0*dy_0[((idx_batch)*dim_0 + idx_chan_out_000)*1 + 0];
      }
      float l_011_0 = left_011[((idx_batch)*dim_l + threadIdx.y)*1 + 0];
      for (int idx_chan_out_011 = threadIdx.x; idx_chan_out_011 < dim_1; idx_chan_out_011 += blockDim.x) {
        dproduct_011[((threadIdx.y)*dim_1 + idx_chan_out_011)*3 + 0] = l_011_0*dy_1[((idx_batch)*dim_1 + idx_chan_out_011)*3 + 0];
        dproduct_011[((threadIdx.y)*dim_1 + idx_chan_out_011)*3 + 1] = l_011_0*dy_1[((idx_batch)*dim_1 + idx_chan_out_011)*3 + 1];
        dproduct_011[((threadIdx.y)*dim_1 + idx_chan_out_011)*3 + 2] = l_011_0*dy_1[((idx_batch)*dim_1 + idx_chan_out_011)*3 + 2];
      }
      float l_101_0 = left_101[((idx_batch)*dim_l + threadIdx.y)*3 + 0];
      float l_101_1 = left_101[((idx_batch)*dim_l + threadIdx.y)*3 + 1];
      float l_101_2 = left_101[((idx_batch)*dim_l + threadIdx.y)*3 + 2];
      for (int idx_chan_out_101 = threadIdx.x; idx_chan_out_101 < dim_1; idx_chan_out_101 += blockDim.x) {
        dproduct_101[((threadIdx.y)*dim_1 + idx_chan_out_101)*1 + 0] = l_101_0*dy_1[((idx_batch)*dim_1 + idx_chan_out_101)*3 + 0] + l_101_1*dy_1[((idx_batch)*dim_1 + idx_chan_out_101)*3 + 1] + l_101_2*dy_1[((idx_batch)*dim_1 + idx_chan_out_101)*3 + 2];
      }
      float l_110_0 = left_110[((idx_batch)*dim_l + threadIdx.y)*3 + 0];
      float l_110_1 = left_110[((idx_batch)*dim_l + threadIdx.y)*3 + 1];
      float l_110_2 = left_110[((idx_batch)*dim_l + threadIdx.y)*3 + 2];
      for (int idx_chan_out_110 = threadIdx.x; idx_chan_out_110 < dim_0; idx_chan_out_110 += blockDim.x) {
        dproduct_110[((threadIdx.y)*dim_0 + idx_chan_out_110)*3 + 0] = l_110_0*dy_0[((idx_batch)*dim_0 + idx_chan_out_110)*1 + 0];
        dproduct_110[((threadIdx.y)*dim_0 + idx_chan_out_110)*3 + 1] = l_110_1*dy_0[((idx_batch)*dim_0 + idx_chan_out_110)*1 + 0];
        dproduct_110[((threadIdx.y)*dim_0 + idx_chan_out_110)*3 + 2] = l_110_2*dy_0[((idx_batch)*dim_0 + idx_chan_out_110)*1 + 0];
      }
      float l_220_0 = left_220[((idx_batch)*dim_l + threadIdx.y)*9 + 0];
      float l_220_1 = left_220[((idx_batch)*dim_l + threadIdx.y)*9 + 1];
      float l_220_2 = left_220[((idx_batch)*dim_l + threadIdx.y)*9 + 2];
      float l_220_3 = left_220[((idx_batch)*dim_l + threadIdx.y)*9 + 3];
      float l_220_4 = left_220[((idx_batch)*dim_l + threadIdx.y)*9 + 4];
      float l_220_5 = left_220[((idx_batch)*dim_l + threadIdx.y)*9 + 5];
      float l_220_6 = left_220[((idx_batch)*dim_l + threadIdx.y)*9 + 6];
      float l_220_7 = left_220[((idx_batch)*dim_l + threadIdx.y)*9 + 7];
      float l_220_8 = left_220[((idx_batch)*dim_l + threadIdx.y)*9 + 8];
      for (int idx_chan_out_220 = threadIdx.x; idx_chan_out_220 < dim_0; idx_chan_out_220 += blockDim.x) {
        dproduct_220[((threadIdx.y)*dim_0 + idx_chan_out_220)*9 + 0] = l_220_0*dy_0[((idx_batch)*dim_0 + idx_chan_out_220)*1 + 0];
        dproduct_220[((threadIdx.y)*dim_0 + idx_chan_out_220)*9 + 1] = l_220_1*dy_0[((idx_batch)*dim_0 + idx_chan_out_220)*1 + 0];
        dproduct_220[((threadIdx.y)*dim_0 + idx_chan_out_220)*9 + 2] = l_220_2*dy_0[((idx_batch)*dim_0 + idx_chan_out_220)*1 + 0];
        dproduct_220[((threadIdx.y)*dim_0 + idx_chan_out_220)*9 + 3] = l_220_3*dy_0[((idx_batch)*dim_0 + idx_chan_out_220)*1 + 0];
        dproduct_220[((threadIdx.y)*dim_0 + idx_chan_out_220)*9 + 4] = l_220_4*dy_0[((idx_batch)*dim_0 + idx_chan_out_220)*1 + 0];
        dproduct_220[((threadIdx.y)*dim_0 + idx_chan_out_220)*9 + 5] = l_220_5*dy_0[((idx_batch)*dim_0 + idx_chan_out_220)*1 + 0];
        dproduct_220[((threadIdx.y)*dim_0 + idx_chan_out_220)*9 + 6] = l_220_6*dy_0[((idx_batch)*dim_0 + idx_chan_out_220)*1 + 0];
        dproduct_220[((threadIdx.y)*dim_0 + idx_chan_out_220)*9 + 7] = l_220_7*dy_0[((idx_batch)*dim_0 + idx_chan_out_220)*1 + 0];
        dproduct_220[((threadIdx.y)*dim_0 + idx_chan_out_220)*9 + 8] = l_220_8*dy_0[((idx_batch)*dim_0 + idx_chan_out_220)*1 + 0];
      }
      float l_222_0 = left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 0];
      float l_222_1 = left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 1];
      float l_222_2 = left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 2];
      float l_222_3 = left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 3];
      float l_222_4 = left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 4];
      float l_222_5 = left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 5];
      float l_222_6 = left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 6];
      float l_222_7 = left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 7];
      float l_222_8 = left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 8];
      for (int idx_chan_out_222 = threadIdx.x; idx_chan_out_222 < dim_2; idx_chan_out_222 += blockDim.x) {
        dproduct_222[((threadIdx.y)*dim_2 + idx_chan_out_222)*9 + 0] = l_222_0*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 0] + l_222_3*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 3] + l_222_6*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 6];
        dproduct_222[((threadIdx.y)*dim_2 + idx_chan_out_222)*9 + 1] = l_222_1*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 0] + l_222_4*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 3] + l_222_7*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 6];
        dproduct_222[((threadIdx.y)*dim_2 + idx_chan_out_222)*9 + 2] = l_222_2*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 0] + l_222_5*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 3] + l_222_8*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 6];
        dproduct_222[((threadIdx.y)*dim_2 + idx_chan_out_222)*9 + 3] = l_222_0*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 1] + l_222_3*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 4] + l_222_6*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 7];
        dproduct_222[((threadIdx.y)*dim_2 + idx_chan_out_222)*9 + 4] = l_222_1*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 1] + l_222_4*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 4] + l_222_7*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 7];
        dproduct_222[((threadIdx.y)*dim_2 + idx_chan_out_222)*9 + 5] = l_222_2*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 1] + l_222_5*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 4] + l_222_8*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 7];
        dproduct_222[((threadIdx.y)*dim_2 + idx_chan_out_222)*9 + 6] = l_222_0*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 2] + l_222_3*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 5] + l_222_6*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 8];
        dproduct_222[((threadIdx.y)*dim_2 + idx_chan_out_222)*9 + 7] = l_222_1*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 2] + l_222_4*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 5] + l_222_7*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 8];
        dproduct_222[((threadIdx.y)*dim_2 + idx_chan_out_222)*9 + 8] = l_222_2*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 2] + l_222_5*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 5] + l_222_8*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 8];
      }
      float l_211_0 = left_211[((idx_batch)*dim_l + threadIdx.y)*9 + 0];
      float l_211_1 = left_211[((idx_batch)*dim_l + threadIdx.y)*9 + 1];
      float l_211_2 = left_211[((idx_batch)*dim_l + threadIdx.y)*9 + 2];
      float l_211_3 = left_211[((idx_batch)*dim_l + threadIdx.y)*9 + 3];
      float l_211_4 = left_211[((idx_batch)*dim_l + threadIdx.y)*9 + 4];
      float l_211_5 = left_211[((idx_batch)*dim_l + threadIdx.y)*9 + 5];
      float l_211_6 = left_211[((idx_batch)*dim_l + threadIdx.y)*9 + 6];
      float l_211_7 = left_211[((idx_batch)*dim_l + threadIdx.y)*9 + 7];
      float l_211_8 = left_211[((idx_batch)*dim_l + threadIdx.y)*9 + 8];
      for (int idx_chan_out_211 = threadIdx.x; idx_chan_out_211 < dim_1; idx_chan_out_211 += blockDim.x) {
        dproduct_211[((threadIdx.y)*dim_1 + idx_chan_out_211)*3 + 0] = l_211_0*dy_1[((idx_batch)*dim_1 + idx_chan_out_211)*3 + 0] + l_211_3*dy_1[((idx_batch)*dim_1 + idx_chan_out_211)*3 + 1] + l_211_6*dy_1[((idx_batch)*dim_1 + idx_chan_out_211)*3 + 2];
        dproduct_211[((threadIdx.y)*dim_1 + idx_chan_out_211)*3 + 1] = l_211_1*dy_1[((idx_batch)*dim_1 + idx_chan_out_211)*3 + 0] + l_211_4*dy_1[((idx_batch)*dim_1 + idx_chan_out_211)*3 + 1] + l_211_7*dy_1[((idx_batch)*dim_1 + idx_chan_out_211)*3 + 2];
        dproduct_211[((threadIdx.y)*dim_1 + idx_chan_out_211)*3 + 2] = l_211_2*dy_1[((idx_batch)*dim_1 + idx_chan_out_211)*3 + 0] + l_211_5*dy_1[((idx_batch)*dim_1 + idx_chan_out_211)*3 + 1] + l_211_8*dy_1[((idx_batch)*dim_1 + idx_chan_out_211)*3 + 2];
      }
      float l_111_0 = left_111[((idx_batch)*dim_l + threadIdx.y)*3 + 0];
      float l_111_1 = left_111[((idx_batch)*dim_l + threadIdx.y)*3 + 1];
      float l_111_2 = left_111[((idx_batch)*dim_l + threadIdx.y)*3 + 2];
      for (int idx_chan_out_111 = threadIdx.x; idx_chan_out_111 < dim_1; idx_chan_out_111 += blockDim.x) {
        dproduct_111[((threadIdx.y)*dim_1 + idx_chan_out_111)*3 + 0] = l_111_2*dy_1[((idx_batch)*dim_1 + idx_chan_out_111)*3 + 1] + (-1)*l_111_1*dy_1[((idx_batch)*dim_1 + idx_chan_out_111)*3 + 2];
        dproduct_111[((threadIdx.y)*dim_1 + idx_chan_out_111)*3 + 1] = (-1)*l_111_2*dy_1[((idx_batch)*dim_1 + idx_chan_out_111)*3 + 0] + l_111_0*dy_1[((idx_batch)*dim_1 + idx_chan_out_111)*3 + 2];
        dproduct_111[((threadIdx.y)*dim_1 + idx_chan_out_111)*3 + 2] = l_111_1*dy_1[((idx_batch)*dim_1 + idx_chan_out_111)*3 + 0] + (-1)*l_111_0*dy_1[((idx_batch)*dim_1 + idx_chan_out_111)*3 + 1];
      }
      float l_212_0 = left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 0];
      float l_212_1 = left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 1];
      float l_212_2 = left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 2];
      float l_212_3 = left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 3];
      float l_212_4 = left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 4];
      float l_212_5 = left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 5];
      float l_212_6 = left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 6];
      float l_212_7 = left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 7];
      float l_212_8 = left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 8];
      for (int idx_chan_out_212 = threadIdx.x; idx_chan_out_212 < dim_2; idx_chan_out_212 += blockDim.x) {
        dproduct_212[((threadIdx.y)*dim_2 + idx_chan_out_212)*3 + 0] = l_212_2*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 1] + (-1)*l_212_1*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 2] + l_212_5*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 4] + (-1)*l_212_4*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 5] + l_212_8*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 7] + (-1)*l_212_7*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 8];
        dproduct_212[((threadIdx.y)*dim_2 + idx_chan_out_212)*3 + 1] = (-1)*l_212_2*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 0] + l_212_0*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 2] + (-1)*l_212_5*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 3] + l_212_3*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 5] + (-1)*l_212_8*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 6] + l_212_6*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 8];
        dproduct_212[((threadIdx.y)*dim_2 + idx_chan_out_212)*3 + 2] = l_212_1*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 0] + (-1)*l_212_0*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 1] + l_212_4*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 3] + (-1)*l_212_3*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 4] + l_212_7*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 6] + (-1)*l_212_6*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 7];
      }
    }
    __syncthreads();
    { // linear transforms to compute dx
      for (int idx_chan_in_0 = threadIdx.y; idx_chan_in_0 < dim_0; idx_chan_in_0 += blockDim.y) {
        float dx_o_0_0 = 0.0;
        float accum_000_0 = 0.0;
        for (int idx_l_000 = 0; idx_l_000 < dim_l; idx_l_000 += 1) {
          for (int idx_chan_out_000 = threadIdx.x; idx_chan_out_000 < dim_0; idx_chan_out_000 += blockDim.x) {
            float P_oi_000 = P_000[((idx_chan_out_000)*dim_l + idx_l_000)*dim_0 + idx_chan_in_0];
            accum_000_0 += P_oi_000*dproduct_000[((idx_l_000)*dim_0 + idx_chan_out_000)*1 + 0];
          }
        }
        // reduce across the warp so that first thread in warp will have the sum 
        for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
          accum_000_0 += __shfl_down_sync(0xffffffff, accum_000_0, offset);
        }
        if (threadIdx.x == 0) {
          dx_o_0_0 += accum_000_0;
        }
        float accum_101_0 = 0.0;
        for (int idx_l_101 = 0; idx_l_101 < dim_l; idx_l_101 += 1) {
          for (int idx_chan_out_101 = threadIdx.x; idx_chan_out_101 < dim_1; idx_chan_out_101 += blockDim.x) {
            float P_oi_101 = P_101[((idx_chan_out_101)*dim_l + idx_l_101)*dim_0 + idx_chan_in_0];
            accum_101_0 += P_oi_101*dproduct_101[((idx_l_101)*dim_1 + idx_chan_out_101)*1 + 0];
          }
        }
        // reduce across the warp so that first thread in warp will have the sum 
        for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
          accum_101_0 += __shfl_down_sync(0xffffffff, accum_101_0, offset);
        }
        if (threadIdx.x == 0) {
          dx_o_0_0 += accum_101_0;
        }
        if (threadIdx.x == 0) {
          dx_0[((idx_batch)*dim_0 + idx_chan_in_0)*1 + 0] = dx_o_0_0;
        }
      }
      for (int idx_chan_in_1 = threadIdx.y; idx_chan_in_1 < dim_1; idx_chan_in_1 += blockDim.y) {
        float dx_o_1_0 = 0.0;
        float dx_o_1_1 = 0.0;
        float dx_o_1_2 = 0.0;
        float accum_011_0 = 0.0;
        float accum_011_1 = 0.0;
        float accum_011_2 = 0.0;
        for (int idx_l_011 = 0; idx_l_011 < dim_l; idx_l_011 += 1) {
          for (int idx_chan_out_011 = threadIdx.x; idx_chan_out_011 < dim_1; idx_chan_out_011 += blockDim.x) {
            float P_oi_011 = P_011[((idx_chan_out_011)*dim_l + idx_l_011)*dim_1 + idx_chan_in_1];
            accum_011_0 += P_oi_011*dproduct_011[((idx_l_011)*dim_1 + idx_chan_out_011)*3 + 0];
            accum_011_1 += P_oi_011*dproduct_011[((idx_l_011)*dim_1 + idx_chan_out_011)*3 + 1];
            accum_011_2 += P_oi_011*dproduct_011[((idx_l_011)*dim_1 + idx_chan_out_011)*3 + 2];
          }
        }
        // reduce across the warp so that first thread in warp will have the sum 
        for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
          accum_011_0 += __shfl_down_sync(0xffffffff, accum_011_0, offset);
          accum_011_1 += __shfl_down_sync(0xffffffff, accum_011_1, offset);
          accum_011_2 += __shfl_down_sync(0xffffffff, accum_011_2, offset);
        }
        if (threadIdx.x == 0) {
          dx_o_1_0 += accum_011_0;
          dx_o_1_1 += accum_011_1;
          dx_o_1_2 += accum_011_2;
        }
        float accum_110_0 = 0.0;
        float accum_110_1 = 0.0;
        float accum_110_2 = 0.0;
        for (int idx_l_110 = 0; idx_l_110 < dim_l; idx_l_110 += 1) {
          for (int idx_chan_out_110 = threadIdx.x; idx_chan_out_110 < dim_0; idx_chan_out_110 += blockDim.x) {
            float P_oi_110 = P_110[((idx_chan_out_110)*dim_l + idx_l_110)*dim_1 + idx_chan_in_1];
            accum_110_0 += P_oi_110*dproduct_110[((idx_l_110)*dim_0 + idx_chan_out_110)*3 + 0];
            accum_110_1 += P_oi_110*dproduct_110[((idx_l_110)*dim_0 + idx_chan_out_110)*3 + 1];
            accum_110_2 += P_oi_110*dproduct_110[((idx_l_110)*dim_0 + idx_chan_out_110)*3 + 2];
          }
        }
        // reduce across the warp so that first thread in warp will have the sum 
        for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
          accum_110_0 += __shfl_down_sync(0xffffffff, accum_110_0, offset);
          accum_110_1 += __shfl_down_sync(0xffffffff, accum_110_1, offset);
          accum_110_2 += __shfl_down_sync(0xffffffff, accum_110_2, offset);
        }
        if (threadIdx.x == 0) {
          dx_o_1_0 += accum_110_0;
          dx_o_1_1 += accum_110_1;
          dx_o_1_2 += accum_110_2;
        }
        float accum_211_0 = 0.0;
        float accum_211_1 = 0.0;
        float accum_211_2 = 0.0;
        for (int idx_l_211 = 0; idx_l_211 < dim_l; idx_l_211 += 1) {
          for (int idx_chan_out_211 = threadIdx.x; idx_chan_out_211 < dim_1; idx_chan_out_211 += blockDim.x) {
            float P_oi_211 = P_211[((idx_chan_out_211)*dim_l + idx_l_211)*dim_1 + idx_chan_in_1];
            accum_211_0 += P_oi_211*dproduct_211[((idx_l_211)*dim_1 + idx_chan_out_211)*3 + 0];
            accum_211_1 += P_oi_211*dproduct_211[((idx_l_211)*dim_1 + idx_chan_out_211)*3 + 1];
            accum_211_2 += P_oi_211*dproduct_211[((idx_l_211)*dim_1 + idx_chan_out_211)*3 + 2];
          }
        }
        // reduce across the warp so that first thread in warp will have the sum 
        for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
          accum_211_0 += __shfl_down_sync(0xffffffff, accum_211_0, offset);
          accum_211_1 += __shfl_down_sync(0xffffffff, accum_211_1, offset);
          accum_211_2 += __shfl_down_sync(0xffffffff, accum_211_2, offset);
        }
        if (threadIdx.x == 0) {
          dx_o_1_0 += accum_211_0;
          dx_o_1_1 += accum_211_1;
          dx_o_1_2 += accum_211_2;
        }
        float accum_111_0 = 0.0;
        float accum_111_1 = 0.0;
        float accum_111_2 = 0.0;
        for (int idx_l_111 = 0; idx_l_111 < dim_l; idx_l_111 += 1) {
          for (int idx_chan_out_111 = threadIdx.x; idx_chan_out_111 < dim_1; idx_chan_out_111 += blockDim.x) {
            float P_oi_111 = P_111[((idx_chan_out_111)*dim_l + idx_l_111)*dim_1 + idx_chan_in_1];
            accum_111_0 += P_oi_111*dproduct_111[((idx_l_111)*dim_1 + idx_chan_out_111)*3 + 0];
            accum_111_1 += P_oi_111*dproduct_111[((idx_l_111)*dim_1 + idx_chan_out_111)*3 + 1];
            accum_111_2 += P_oi_111*dproduct_111[((idx_l_111)*dim_1 + idx_chan_out_111)*3 + 2];
          }
        }
        // reduce across the warp so that first thread in warp will have the sum 
        for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
          accum_111_0 += __shfl_down_sync(0xffffffff, accum_111_0, offset);
          accum_111_1 += __shfl_down_sync(0xffffffff, accum_111_1, offset);
          accum_111_2 += __shfl_down_sync(0xffffffff, accum_111_2, offset);
        }
        if (threadIdx.x == 0) {
          dx_o_1_0 += accum_111_0;
          dx_o_1_1 += accum_111_1;
          dx_o_1_2 += accum_111_2;
        }
        float accum_212_0 = 0.0;
        float accum_212_1 = 0.0;
        float accum_212_2 = 0.0;
        for (int idx_l_212 = 0; idx_l_212 < dim_l; idx_l_212 += 1) {
          for (int idx_chan_out_212 = threadIdx.x; idx_chan_out_212 < dim_2; idx_chan_out_212 += blockDim.x) {
            float P_oi_212 = P_212[((idx_chan_out_212)*dim_l + idx_l_212)*dim_1 + idx_chan_in_1];
            accum_212_0 += P_oi_212*dproduct_212[((idx_l_212)*dim_2 + idx_chan_out_212)*3 + 0];
            accum_212_1 += P_oi_212*dproduct_212[((idx_l_212)*dim_2 + idx_chan_out_212)*3 + 1];
            accum_212_2 += P_oi_212*dproduct_212[((idx_l_212)*dim_2 + idx_chan_out_212)*3 + 2];
          }
        }
        // reduce across the warp so that first thread in warp will have the sum 
        for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
          accum_212_0 += __shfl_down_sync(0xffffffff, accum_212_0, offset);
          accum_212_1 += __shfl_down_sync(0xffffffff, accum_212_1, offset);
          accum_212_2 += __shfl_down_sync(0xffffffff, accum_212_2, offset);
        }
        if (threadIdx.x == 0) {
          dx_o_1_0 += accum_212_0;
          dx_o_1_1 += accum_212_1;
          dx_o_1_2 += accum_212_2;
        }
        if (threadIdx.x == 0) {
          dx_1[((idx_batch)*dim_1 + idx_chan_in_1)*3 + 0] = dx_o_1_0;
          dx_1[((idx_batch)*dim_1 + idx_chan_in_1)*3 + 1] = dx_o_1_1;
          dx_1[((idx_batch)*dim_1 + idx_chan_in_1)*3 + 2] = dx_o_1_2;
        }
      }
      for (int idx_chan_in_2 = threadIdx.y; idx_chan_in_2 < dim_2; idx_chan_in_2 += blockDim.y) {
        float dx_o_2_0 = 0.0;
        float dx_o_2_1 = 0.0;
        float dx_o_2_2 = 0.0;
        float dx_o_2_3 = 0.0;
        float dx_o_2_4 = 0.0;
        float dx_o_2_5 = 0.0;
        float dx_o_2_6 = 0.0;
        float dx_o_2_7 = 0.0;
        float dx_o_2_8 = 0.0;
        float accum_220_0 = 0.0;
        float accum_220_1 = 0.0;
        float accum_220_2 = 0.0;
        float accum_220_3 = 0.0;
        float accum_220_4 = 0.0;
        float accum_220_5 = 0.0;
        float accum_220_6 = 0.0;
        float accum_220_7 = 0.0;
        float accum_220_8 = 0.0;
        for (int idx_l_220 = 0; idx_l_220 < dim_l; idx_l_220 += 1) {
          for (int idx_chan_out_220 = threadIdx.x; idx_chan_out_220 < dim_0; idx_chan_out_220 += blockDim.x) {
            float P_oi_220 = P_220[((idx_chan_out_220)*dim_l + idx_l_220)*dim_2 + idx_chan_in_2];
            accum_220_0 += P_oi_220*dproduct_220[((idx_l_220)*dim_0 + idx_chan_out_220)*9 + 0];
            accum_220_1 += P_oi_220*dproduct_220[((idx_l_220)*dim_0 + idx_chan_out_220)*9 + 1];
            accum_220_2 += P_oi_220*dproduct_220[((idx_l_220)*dim_0 + idx_chan_out_220)*9 + 2];
            accum_220_3 += P_oi_220*dproduct_220[((idx_l_220)*dim_0 + idx_chan_out_220)*9 + 3];
            accum_220_4 += P_oi_220*dproduct_220[((idx_l_220)*dim_0 + idx_chan_out_220)*9 + 4];
            accum_220_5 += P_oi_220*dproduct_220[((idx_l_220)*dim_0 + idx_chan_out_220)*9 + 5];
            accum_220_6 += P_oi_220*dproduct_220[((idx_l_220)*dim_0 + idx_chan_out_220)*9 + 6];
            accum_220_7 += P_oi_220*dproduct_220[((idx_l_220)*dim_0 + idx_chan_out_220)*9 + 7];
            accum_220_8 += P_oi_220*dproduct_220[((idx_l_220)*dim_0 + idx_chan_out_220)*9 + 8];
          }
        }
        // reduce across the warp so that first thread in warp will have the sum 
        for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
          accum_220_0 += __shfl_down_sync(0xffffffff, accum_220_0, offset);
          accum_220_1 += __shfl_down_sync(0xffffffff, accum_220_1, offset);
          accum_220_2 += __shfl_down_sync(0xffffffff, accum_220_2, offset);
          accum_220_3 += __shfl_down_sync(0xffffffff, accum_220_3, offset);
          accum_220_4 += __shfl_down_sync(0xffffffff, accum_220_4, offset);
          accum_220_5 += __shfl_down_sync(0xffffffff, accum_220_5, offset);
          accum_220_6 += __shfl_down_sync(0xffffffff, accum_220_6, offset);
          accum_220_7 += __shfl_down_sync(0xffffffff, accum_220_7, offset);
          accum_220_8 += __shfl_down_sync(0xffffffff, accum_220_8, offset);
        }
        if (threadIdx.x == 0) {
          dx_o_2_0 += accum_220_0;
          dx_o_2_1 += accum_220_1;
          dx_o_2_2 += accum_220_2;
          dx_o_2_3 += accum_220_3;
          dx_o_2_4 += accum_220_4;
          dx_o_2_5 += accum_220_5;
          dx_o_2_6 += accum_220_6;
          dx_o_2_7 += accum_220_7;
          dx_o_2_8 += accum_220_8;
        }
        float accum_222_0 = 0.0;
        float accum_222_1 = 0.0;
        float accum_222_2 = 0.0;
        float accum_222_3 = 0.0;
        float accum_222_4 = 0.0;
        float accum_222_5 = 0.0;
        float accum_222_6 = 0.0;
        float accum_222_7 = 0.0;
        float accum_222_8 = 0.0;
        for (int idx_l_222 = 0; idx_l_222 < dim_l; idx_l_222 += 1) {
          for (int idx_chan_out_222 = threadIdx.x; idx_chan_out_222 < dim_2; idx_chan_out_222 += blockDim.x) {
            float P_oi_222 = P_222[((idx_chan_out_222)*dim_l + idx_l_222)*dim_2 + idx_chan_in_2];
            accum_222_0 += P_oi_222*dproduct_222[((idx_l_222)*dim_2 + idx_chan_out_222)*9 + 0];
            accum_222_1 += P_oi_222*dproduct_222[((idx_l_222)*dim_2 + idx_chan_out_222)*9 + 1];
            accum_222_2 += P_oi_222*dproduct_222[((idx_l_222)*dim_2 + idx_chan_out_222)*9 + 2];
            accum_222_3 += P_oi_222*dproduct_222[((idx_l_222)*dim_2 + idx_chan_out_222)*9 + 3];
            accum_222_4 += P_oi_222*dproduct_222[((idx_l_222)*dim_2 + idx_chan_out_222)*9 + 4];
            accum_222_5 += P_oi_222*dproduct_222[((idx_l_222)*dim_2 + idx_chan_out_222)*9 + 5];
            accum_222_6 += P_oi_222*dproduct_222[((idx_l_222)*dim_2 + idx_chan_out_222)*9 + 6];
            accum_222_7 += P_oi_222*dproduct_222[((idx_l_222)*dim_2 + idx_chan_out_222)*9 + 7];
            accum_222_8 += P_oi_222*dproduct_222[((idx_l_222)*dim_2 + idx_chan_out_222)*9 + 8];
          }
        }
        // reduce across the warp so that first thread in warp will have the sum 
        for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
          accum_222_0 += __shfl_down_sync(0xffffffff, accum_222_0, offset);
          accum_222_1 += __shfl_down_sync(0xffffffff, accum_222_1, offset);
          accum_222_2 += __shfl_down_sync(0xffffffff, accum_222_2, offset);
          accum_222_3 += __shfl_down_sync(0xffffffff, accum_222_3, offset);
          accum_222_4 += __shfl_down_sync(0xffffffff, accum_222_4, offset);
          accum_222_5 += __shfl_down_sync(0xffffffff, accum_222_5, offset);
          accum_222_6 += __shfl_down_sync(0xffffffff, accum_222_6, offset);
          accum_222_7 += __shfl_down_sync(0xffffffff, accum_222_7, offset);
          accum_222_8 += __shfl_down_sync(0xffffffff, accum_222_8, offset);
        }
        if (threadIdx.x == 0) {
          dx_o_2_0 += accum_222_0;
          dx_o_2_1 += accum_222_1;
          dx_o_2_2 += accum_222_2;
          dx_o_2_3 += accum_222_3;
          dx_o_2_4 += accum_222_4;
          dx_o_2_5 += accum_222_5;
          dx_o_2_6 += accum_222_6;
          dx_o_2_7 += accum_222_7;
          dx_o_2_8 += accum_222_8;
        }
        if (threadIdx.x == 0) {
          dx_2[((idx_batch)*dim_2 + idx_chan_in_2)*9 + 0] = dx_o_2_0;
          dx_2[((idx_batch)*dim_2 + idx_chan_in_2)*9 + 1] = dx_o_2_1;
          dx_2[((idx_batch)*dim_2 + idx_chan_in_2)*9 + 2] = dx_o_2_2;
          dx_2[((idx_batch)*dim_2 + idx_chan_in_2)*9 + 3] = dx_o_2_3;
          dx_2[((idx_batch)*dim_2 + idx_chan_in_2)*9 + 4] = dx_o_2_4;
          dx_2[((idx_batch)*dim_2 + idx_chan_in_2)*9 + 5] = dx_o_2_5;
          dx_2[((idx_batch)*dim_2 + idx_chan_in_2)*9 + 6] = dx_o_2_6;
          dx_2[((idx_batch)*dim_2 + idx_chan_in_2)*9 + 7] = dx_o_2_7;
          dx_2[((idx_batch)*dim_2 + idx_chan_in_2)*9 + 8] = dx_o_2_8;
        }
      }
    }
  }
}


void fused_tensor_prods_example_backward(
    int batch, int dim_l, int dim_0, int dim_1, int dim_2,
    const float* dy_0, const float* dy_1, const float* dy_2, const float* P_000, const float* left_000, const float* P_011, const float* left_011, const float* P_101, const float* left_101, const float* P_110, const float* left_110, const float* P_220, const float* left_220, const float* P_222, const float* left_222, const float* P_211, const float* left_211, const float* P_111, const float* left_111, const float* P_212, const float* left_212,
    float* dx_0, float* dx_1, float* dx_2) {
  
  int p_0 = dim_l*dim_0;
  int p_1 = dim_l*dim_1;
  int p_2 = dim_l*dim_2;
  int sharedmemsz = 0;
  sharedmemsz += 13*p_0;
  int p_1_base = sharedmemsz;
  sharedmemsz += 10*p_1;
  int p_2_base = sharedmemsz;
  sharedmemsz += 12*p_2;
  dim3 gridsz = dim3(batch);
  dim3 blocksz = dim3(WARPSZ, dim_l);
  fused_tensor_prods_example_backward_kern<<<gridsz, blocksz, sharedmemsz*sizeof(float)>>>(
      p_0, p_1_base, p_1, p_2_base, p_2, 
      batch, dim_l, dim_0, dim_1, dim_2,
      dy_0, dy_1, dy_2, P_000, left_000, P_011, left_011, P_101, left_101, P_110, left_110, P_220, left_220, P_222, left_222, P_211, left_211, P_111, left_111, P_212, left_212,
      dx_0, dx_1, dx_2);
  
}


__global__
void fused_tensor_prods_example_backleft_kern(
    // <<<(batch), (WARPSZ, dim_l)>>>
    
    int batch, int dim_l, int dim_0, int dim_1, int dim_2,
    const float* x_0, const float* x_1, const float* x_2, const float* dy_0, const float* dy_1, const float* dy_2, const float* P_000, const float* P_011, const float* P_101, const float* P_110, const float* P_220, const float* P_222, const float* P_211, const float* P_111, const float* P_212,
    float* __restrict__ dleft_000, float* __restrict__ dleft_011, float* __restrict__ dleft_101, float* __restrict__ dleft_110, float* __restrict__ dleft_220, float* __restrict__ dleft_222, float* __restrict__ dleft_211, float* __restrict__ dleft_111, float* __restrict__ dleft_212) {
  extern __shared__ float s[];
  for (int idx_batch = blockIdx.x; idx_batch < batch; idx_batch += gridDim.x) {
    { // compute left derivative tensor products
      float accum_000_0 = 0.0;
      for (int idx_chan_in_000 = threadIdx.x; idx_chan_in_000 < dim_0; idx_chan_in_000 += blockDim.x) {
        for (int idx_chan_out_000 = 0; idx_chan_out_000 < dim_0; idx_chan_out_000 += 1) {
          float l_000_0 = x_0[((idx_batch)*dim_0 + idx_chan_in_000)*1 + 0]*dy_0[((idx_batch)*dim_0 + idx_chan_out_000)*1 + 0];
          float P_oi_000 = P_000[((idx_chan_out_000)*blockDim.y + threadIdx.y)*dim_0 + idx_chan_in_000];
          accum_000_0 += P_oi_000*l_000_0;
        }
      }
      // reduce across the warp so that first thread in warp will have the sum 
      for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
        accum_000_0 += __shfl_down_sync(0xffffffff, accum_000_0, offset);
      }
      if (threadIdx.x == 0) {
        dleft_000[((idx_batch)*dim_l + threadIdx.y)*1 + 0] = accum_000_0;
      }
      float accum_011_0 = 0.0;
      for (int idx_chan_in_011 = threadIdx.x; idx_chan_in_011 < dim_1; idx_chan_in_011 += blockDim.x) {
        for (int idx_chan_out_011 = 0; idx_chan_out_011 < dim_1; idx_chan_out_011 += 1) {
          float l_011_0 = x_1[((idx_batch)*dim_1 + idx_chan_in_011)*3 + 0]*dy_1[((idx_batch)*dim_1 + idx_chan_out_011)*3 + 0] + x_1[((idx_batch)*dim_1 + idx_chan_in_011)*3 + 1]*dy_1[((idx_batch)*dim_1 + idx_chan_out_011)*3 + 1] + x_1[((idx_batch)*dim_1 + idx_chan_in_011)*3 + 2]*dy_1[((idx_batch)*dim_1 + idx_chan_out_011)*3 + 2];
          float P_oi_011 = P_011[((idx_chan_out_011)*blockDim.y + threadIdx.y)*dim_1 + idx_chan_in_011];
          accum_011_0 += P_oi_011*l_011_0;
        }
      }
      // reduce across the warp so that first thread in warp will have the sum 
      for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
        accum_011_0 += __shfl_down_sync(0xffffffff, accum_011_0, offset);
      }
      if (threadIdx.x == 0) {
        dleft_011[((idx_batch)*dim_l + threadIdx.y)*1 + 0] = accum_011_0;
      }
      float accum_101_0 = 0.0;
      float accum_101_1 = 0.0;
      float accum_101_2 = 0.0;
      for (int idx_chan_in_101 = threadIdx.x; idx_chan_in_101 < dim_0; idx_chan_in_101 += blockDim.x) {
        for (int idx_chan_out_101 = 0; idx_chan_out_101 < dim_1; idx_chan_out_101 += 1) {
          float l_101_0 = x_0[((idx_batch)*dim_0 + idx_chan_in_101)*1 + 0]*dy_1[((idx_batch)*dim_1 + idx_chan_out_101)*3 + 0];
          float l_101_1 = x_0[((idx_batch)*dim_0 + idx_chan_in_101)*1 + 0]*dy_1[((idx_batch)*dim_1 + idx_chan_out_101)*3 + 1];
          float l_101_2 = x_0[((idx_batch)*dim_0 + idx_chan_in_101)*1 + 0]*dy_1[((idx_batch)*dim_1 + idx_chan_out_101)*3 + 2];
          float P_oi_101 = P_101[((idx_chan_out_101)*blockDim.y + threadIdx.y)*dim_0 + idx_chan_in_101];
          accum_101_0 += P_oi_101*l_101_0;
          accum_101_1 += P_oi_101*l_101_1;
          accum_101_2 += P_oi_101*l_101_2;
        }
      }
      // reduce across the warp so that first thread in warp will have the sum 
      for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
        accum_101_0 += __shfl_down_sync(0xffffffff, accum_101_0, offset);
        accum_101_1 += __shfl_down_sync(0xffffffff, accum_101_1, offset);
        accum_101_2 += __shfl_down_sync(0xffffffff, accum_101_2, offset);
      }
      if (threadIdx.x == 0) {
        dleft_101[((idx_batch)*dim_l + threadIdx.y)*3 + 0] = accum_101_0;
        dleft_101[((idx_batch)*dim_l + threadIdx.y)*3 + 1] = accum_101_1;
        dleft_101[((idx_batch)*dim_l + threadIdx.y)*3 + 2] = accum_101_2;
      }
      float accum_110_0 = 0.0;
      float accum_110_1 = 0.0;
      float accum_110_2 = 0.0;
      for (int idx_chan_in_110 = threadIdx.x; idx_chan_in_110 < dim_1; idx_chan_in_110 += blockDim.x) {
        for (int idx_chan_out_110 = 0; idx_chan_out_110 < dim_0; idx_chan_out_110 += 1) {
          float l_110_0 = x_1[((idx_batch)*dim_1 + idx_chan_in_110)*3 + 0]*dy_0[((idx_batch)*dim_0 + idx_chan_out_110)*1 + 0];
          float l_110_1 = x_1[((idx_batch)*dim_1 + idx_chan_in_110)*3 + 1]*dy_0[((idx_batch)*dim_0 + idx_chan_out_110)*1 + 0];
          float l_110_2 = x_1[((idx_batch)*dim_1 + idx_chan_in_110)*3 + 2]*dy_0[((idx_batch)*dim_0 + idx_chan_out_110)*1 + 0];
          float P_oi_110 = P_110[((idx_chan_out_110)*blockDim.y + threadIdx.y)*dim_1 + idx_chan_in_110];
          accum_110_0 += P_oi_110*l_110_0;
          accum_110_1 += P_oi_110*l_110_1;
          accum_110_2 += P_oi_110*l_110_2;
        }
      }
      // reduce across the warp so that first thread in warp will have the sum 
      for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
        accum_110_0 += __shfl_down_sync(0xffffffff, accum_110_0, offset);
        accum_110_1 += __shfl_down_sync(0xffffffff, accum_110_1, offset);
        accum_110_2 += __shfl_down_sync(0xffffffff, accum_110_2, offset);
      }
      if (threadIdx.x == 0) {
        dleft_110[((idx_batch)*dim_l + threadIdx.y)*3 + 0] = accum_110_0;
        dleft_110[((idx_batch)*dim_l + threadIdx.y)*3 + 1] = accum_110_1;
        dleft_110[((idx_batch)*dim_l + threadIdx.y)*3 + 2] = accum_110_2;
      }
      float accum_220_0 = 0.0;
      float accum_220_1 = 0.0;
      float accum_220_2 = 0.0;
      float accum_220_3 = 0.0;
      float accum_220_4 = 0.0;
      float accum_220_5 = 0.0;
      float accum_220_6 = 0.0;
      float accum_220_7 = 0.0;
      float accum_220_8 = 0.0;
      for (int idx_chan_in_220 = threadIdx.x; idx_chan_in_220 < dim_2; idx_chan_in_220 += blockDim.x) {
        for (int idx_chan_out_220 = 0; idx_chan_out_220 < dim_0; idx_chan_out_220 += 1) {
          float l_220_0 = x_2[((idx_batch)*dim_2 + idx_chan_in_220)*9 + 0]*dy_0[((idx_batch)*dim_0 + idx_chan_out_220)*1 + 0];
          float l_220_1 = x_2[((idx_batch)*dim_2 + idx_chan_in_220)*9 + 1]*dy_0[((idx_batch)*dim_0 + idx_chan_out_220)*1 + 0];
          float l_220_2 = x_2[((idx_batch)*dim_2 + idx_chan_in_220)*9 + 2]*dy_0[((idx_batch)*dim_0 + idx_chan_out_220)*1 + 0];
          float l_220_3 = x_2[((idx_batch)*dim_2 + idx_chan_in_220)*9 + 3]*dy_0[((idx_batch)*dim_0 + idx_chan_out_220)*1 + 0];
          float l_220_4 = x_2[((idx_batch)*dim_2 + idx_chan_in_220)*9 + 4]*dy_0[((idx_batch)*dim_0 + idx_chan_out_220)*1 + 0];
          float l_220_5 = x_2[((idx_batch)*dim_2 + idx_chan_in_220)*9 + 5]*dy_0[((idx_batch)*dim_0 + idx_chan_out_220)*1 + 0];
          float l_220_6 = x_2[((idx_batch)*dim_2 + idx_chan_in_220)*9 + 6]*dy_0[((idx_batch)*dim_0 + idx_chan_out_220)*1 + 0];
          float l_220_7 = x_2[((idx_batch)*dim_2 + idx_chan_in_220)*9 + 7]*dy_0[((idx_batch)*dim_0 + idx_chan_out_220)*1 + 0];
          float l_220_8 = x_2[((idx_batch)*dim_2 + idx_chan_in_220)*9 + 8]*dy_0[((idx_batch)*dim_0 + idx_chan_out_220)*1 + 0];
          float P_oi_220 = P_220[((idx_chan_out_220)*blockDim.y + threadIdx.y)*dim_2 + idx_chan_in_220];
          accum_220_0 += P_oi_220*l_220_0;
          accum_220_1 += P_oi_220*l_220_1;
          accum_220_2 += P_oi_220*l_220_2;
          accum_220_3 += P_oi_220*l_220_3;
          accum_220_4 += P_oi_220*l_220_4;
          accum_220_5 += P_oi_220*l_220_5;
          accum_220_6 += P_oi_220*l_220_6;
          accum_220_7 += P_oi_220*l_220_7;
          accum_220_8 += P_oi_220*l_220_8;
        }
      }
      // reduce across the warp so that first thread in warp will have the sum 
      for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
        accum_220_0 += __shfl_down_sync(0xffffffff, accum_220_0, offset);
        accum_220_1 += __shfl_down_sync(0xffffffff, accum_220_1, offset);
        accum_220_2 += __shfl_down_sync(0xffffffff, accum_220_2, offset);
        accum_220_3 += __shfl_down_sync(0xffffffff, accum_220_3, offset);
        accum_220_4 += __shfl_down_sync(0xffffffff, accum_220_4, offset);
        accum_220_5 += __shfl_down_sync(0xffffffff, accum_220_5, offset);
        accum_220_6 += __shfl_down_sync(0xffffffff, accum_220_6, offset);
        accum_220_7 += __shfl_down_sync(0xffffffff, accum_220_7, offset);
        accum_220_8 += __shfl_down_sync(0xffffffff, accum_220_8, offset);
      }
      if (threadIdx.x == 0) {
        dleft_220[((idx_batch)*dim_l + threadIdx.y)*9 + 0] = accum_220_0;
        dleft_220[((idx_batch)*dim_l + threadIdx.y)*9 + 1] = accum_220_1;
        dleft_220[((idx_batch)*dim_l + threadIdx.y)*9 + 2] = accum_220_2;
        dleft_220[((idx_batch)*dim_l + threadIdx.y)*9 + 3] = accum_220_3;
        dleft_220[((idx_batch)*dim_l + threadIdx.y)*9 + 4] = accum_220_4;
        dleft_220[((idx_batch)*dim_l + threadIdx.y)*9 + 5] = accum_220_5;
        dleft_220[((idx_batch)*dim_l + threadIdx.y)*9 + 6] = accum_220_6;
        dleft_220[((idx_batch)*dim_l + threadIdx.y)*9 + 7] = accum_220_7;
        dleft_220[((idx_batch)*dim_l + threadIdx.y)*9 + 8] = accum_220_8;
      }
      float accum_222_0 = 0.0;
      float accum_222_1 = 0.0;
      float accum_222_2 = 0.0;
      float accum_222_3 = 0.0;
      float accum_222_4 = 0.0;
      float accum_222_5 = 0.0;
      float accum_222_6 = 0.0;
      float accum_222_7 = 0.0;
      float accum_222_8 = 0.0;
      for (int idx_chan_in_222 = threadIdx.x; idx_chan_in_222 < dim_2; idx_chan_in_222 += blockDim.x) {
        for (int idx_chan_out_222 = 0; idx_chan_out_222 < dim_2; idx_chan_out_222 += 1) {
          float l_222_0 = x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 0]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 0] + x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 3]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 1] + x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 6]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 2];
          float l_222_1 = x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 1]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 0] + x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 4]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 1] + x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 7]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 2];
          float l_222_2 = x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 2]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 0] + x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 5]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 1] + x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 8]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 2];
          float l_222_3 = x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 0]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 3] + x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 3]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 4] + x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 6]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 5];
          float l_222_4 = x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 1]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 3] + x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 4]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 4] + x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 7]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 5];
          float l_222_5 = x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 2]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 3] + x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 5]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 4] + x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 8]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 5];
          float l_222_6 = x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 0]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 6] + x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 3]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 7] + x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 6]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 8];
          float l_222_7 = x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 1]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 6] + x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 4]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 7] + x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 7]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 8];
          float l_222_8 = x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 2]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 6] + x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 5]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 7] + x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 8]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 8];
          float P_oi_222 = P_222[((idx_chan_out_222)*blockDim.y + threadIdx.y)*dim_2 + idx_chan_in_222];
          accum_222_0 += P_oi_222*l_222_0;
          accum_222_1 += P_oi_222*l_222_1;
          accum_222_2 += P_oi_222*l_222_2;
          accum_222_3 += P_oi_222*l_222_3;
          accum_222_4 += P_oi_222*l_222_4;
          accum_222_5 += P_oi_222*l_222_5;
          accum_222_6 += P_oi_222*l_222_6;
          accum_222_7 += P_oi_222*l_222_7;
          accum_222_8 += P_oi_222*l_222_8;
        }
      }
      // reduce across the warp so that first thread in warp will have the sum 
      for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
        accum_222_0 += __shfl_down_sync(0xffffffff, accum_222_0, offset);
        accum_222_1 += __shfl_down_sync(0xffffffff, accum_222_1, offset);
        accum_222_2 += __shfl_down_sync(0xffffffff, accum_222_2, offset);
        accum_222_3 += __shfl_down_sync(0xffffffff, accum_222_3, offset);
        accum_222_4 += __shfl_down_sync(0xffffffff, accum_222_4, offset);
        accum_222_5 += __shfl_down_sync(0xffffffff, accum_222_5, offset);
        accum_222_6 += __shfl_down_sync(0xffffffff, accum_222_6, offset);
        accum_222_7 += __shfl_down_sync(0xffffffff, accum_222_7, offset);
        accum_222_8 += __shfl_down_sync(0xffffffff, accum_222_8, offset);
      }
      if (threadIdx.x == 0) {
        dleft_222[((idx_batch)*dim_l + threadIdx.y)*9 + 0] = accum_222_0;
        dleft_222[((idx_batch)*dim_l + threadIdx.y)*9 + 1] = accum_222_1;
        dleft_222[((idx_batch)*dim_l + threadIdx.y)*9 + 2] = accum_222_2;
        dleft_222[((idx_batch)*dim_l + threadIdx.y)*9 + 3] = accum_222_3;
        dleft_222[((idx_batch)*dim_l + threadIdx.y)*9 + 4] = accum_222_4;
        dleft_222[((idx_batch)*dim_l + threadIdx.y)*9 + 5] = accum_222_5;
        dleft_222[((idx_batch)*dim_l + threadIdx.y)*9 + 6] = accum_222_6;
        dleft_222[((idx_batch)*dim_l + threadIdx.y)*9 + 7] = accum_222_7;
        dleft_222[((idx_batch)*dim_l + threadIdx.y)*9 + 8] = accum_222_8;
      }
      float accum_211_0 = 0.0;
      float accum_211_1 = 0.0;
      float accum_211_2 = 0.0;
      float accum_211_3 = 0.0;
      float accum_211_4 = 0.0;
      float accum_211_5 = 0.0;
      float accum_211_6 = 0.0;
      float accum_211_7 = 0.0;
      float accum_211_8 = 0.0;
      for (int idx_chan_in_211 = threadIdx.x; idx_chan_in_211 < dim_1; idx_chan_in_211 += blockDim.x) {
        for (int idx_chan_out_211 = 0; idx_chan_out_211 < dim_1; idx_chan_out_211 += 1) {
          float l_211_0 = x_1[((idx_batch)*dim_1 + idx_chan_in_211)*3 + 0]*dy_1[((idx_batch)*dim_1 + idx_chan_out_211)*3 + 0];
          float l_211_1 = x_1[((idx_batch)*dim_1 + idx_chan_in_211)*3 + 1]*dy_1[((idx_batch)*dim_1 + idx_chan_out_211)*3 + 0];
          float l_211_2 = x_1[((idx_batch)*dim_1 + idx_chan_in_211)*3 + 2]*dy_1[((idx_batch)*dim_1 + idx_chan_out_211)*3 + 0];
          float l_211_3 = x_1[((idx_batch)*dim_1 + idx_chan_in_211)*3 + 0]*dy_1[((idx_batch)*dim_1 + idx_chan_out_211)*3 + 1];
          float l_211_4 = x_1[((idx_batch)*dim_1 + idx_chan_in_211)*3 + 1]*dy_1[((idx_batch)*dim_1 + idx_chan_out_211)*3 + 1];
          float l_211_5 = x_1[((idx_batch)*dim_1 + idx_chan_in_211)*3 + 2]*dy_1[((idx_batch)*dim_1 + idx_chan_out_211)*3 + 1];
          float l_211_6 = x_1[((idx_batch)*dim_1 + idx_chan_in_211)*3 + 0]*dy_1[((idx_batch)*dim_1 + idx_chan_out_211)*3 + 2];
          float l_211_7 = x_1[((idx_batch)*dim_1 + idx_chan_in_211)*3 + 1]*dy_1[((idx_batch)*dim_1 + idx_chan_out_211)*3 + 2];
          float l_211_8 = x_1[((idx_batch)*dim_1 + idx_chan_in_211)*3 + 2]*dy_1[((idx_batch)*dim_1 + idx_chan_out_211)*3 + 2];
          float P_oi_211 = P_211[((idx_chan_out_211)*blockDim.y + threadIdx.y)*dim_1 + idx_chan_in_211];
          accum_211_0 += P_oi_211*l_211_0;
          accum_211_1 += P_oi_211*l_211_1;
          accum_211_2 += P_oi_211*l_211_2;
          accum_211_3 += P_oi_211*l_211_3;
          accum_211_4 += P_oi_211*l_211_4;
          accum_211_5 += P_oi_211*l_211_5;
          accum_211_6 += P_oi_211*l_211_6;
          accum_211_7 += P_oi_211*l_211_7;
          accum_211_8 += P_oi_211*l_211_8;
        }
      }
      // reduce across the warp so that first thread in warp will have the sum 
      for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
        accum_211_0 += __shfl_down_sync(0xffffffff, accum_211_0, offset);
        accum_211_1 += __shfl_down_sync(0xffffffff, accum_211_1, offset);
        accum_211_2 += __shfl_down_sync(0xffffffff, accum_211_2, offset);
        accum_211_3 += __shfl_down_sync(0xffffffff, accum_211_3, offset);
        accum_211_4 += __shfl_down_sync(0xffffffff, accum_211_4, offset);
        accum_211_5 += __shfl_down_sync(0xffffffff, accum_211_5, offset);
        accum_211_6 += __shfl_down_sync(0xffffffff, accum_211_6, offset);
        accum_211_7 += __shfl_down_sync(0xffffffff, accum_211_7, offset);
        accum_211_8 += __shfl_down_sync(0xffffffff, accum_211_8, offset);
      }
      if (threadIdx.x == 0) {
        dleft_211[((idx_batch)*dim_l + threadIdx.y)*9 + 0] = accum_211_0;
        dleft_211[((idx_batch)*dim_l + threadIdx.y)*9 + 1] = accum_211_1;
        dleft_211[((idx_batch)*dim_l + threadIdx.y)*9 + 2] = accum_211_2;
        dleft_211[((idx_batch)*dim_l + threadIdx.y)*9 + 3] = accum_211_3;
        dleft_211[((idx_batch)*dim_l + threadIdx.y)*9 + 4] = accum_211_4;
        dleft_211[((idx_batch)*dim_l + threadIdx.y)*9 + 5] = accum_211_5;
        dleft_211[((idx_batch)*dim_l + threadIdx.y)*9 + 6] = accum_211_6;
        dleft_211[((idx_batch)*dim_l + threadIdx.y)*9 + 7] = accum_211_7;
        dleft_211[((idx_batch)*dim_l + threadIdx.y)*9 + 8] = accum_211_8;
      }
      float accum_111_0 = 0.0;
      float accum_111_1 = 0.0;
      float accum_111_2 = 0.0;
      for (int idx_chan_in_111 = threadIdx.x; idx_chan_in_111 < dim_1; idx_chan_in_111 += blockDim.x) {
        for (int idx_chan_out_111 = 0; idx_chan_out_111 < dim_1; idx_chan_out_111 += 1) {
          float l_111_0 = (-1)*x_1[((idx_batch)*dim_1 + idx_chan_in_111)*3 + 2]*dy_1[((idx_batch)*dim_1 + idx_chan_out_111)*3 + 1] + x_1[((idx_batch)*dim_1 + idx_chan_in_111)*3 + 1]*dy_1[((idx_batch)*dim_1 + idx_chan_out_111)*3 + 2];
          float l_111_1 = x_1[((idx_batch)*dim_1 + idx_chan_in_111)*3 + 2]*dy_1[((idx_batch)*dim_1 + idx_chan_out_111)*3 + 0] + (-1)*x_1[((idx_batch)*dim_1 + idx_chan_in_111)*3 + 0]*dy_1[((idx_batch)*dim_1 + idx_chan_out_111)*3 + 2];
          float l_111_2 = (-1)*x_1[((idx_batch)*dim_1 + idx_chan_in_111)*3 + 1]*dy_1[((idx_batch)*dim_1 + idx_chan_out_111)*3 + 0] + x_1[((idx_batch)*dim_1 + idx_chan_in_111)*3 + 0]*dy_1[((idx_batch)*dim_1 + idx_chan_out_111)*3 + 1];
          float P_oi_111 = P_111[((idx_chan_out_111)*blockDim.y + threadIdx.y)*dim_1 + idx_chan_in_111];
          accum_111_0 += P_oi_111*l_111_0;
          accum_111_1 += P_oi_111*l_111_1;
          accum_111_2 += P_oi_111*l_111_2;
        }
      }
      // reduce across the warp so that first thread in warp will have the sum 
      for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
        accum_111_0 += __shfl_down_sync(0xffffffff, accum_111_0, offset);
        accum_111_1 += __shfl_down_sync(0xffffffff, accum_111_1, offset);
        accum_111_2 += __shfl_down_sync(0xffffffff, accum_111_2, offset);
      }
      if (threadIdx.x == 0) {
        dleft_111[((idx_batch)*dim_l + threadIdx.y)*3 + 0] = accum_111_0;
        dleft_111[((idx_batch)*dim_l + threadIdx.y)*3 + 1] = accum_111_1;
        dleft_111[((idx_batch)*dim_l + threadIdx.y)*3 + 2] = accum_111_2;
      }
      float accum_212_0 = 0.0;
      float accum_212_1 = 0.0;
      float accum_212_2 = 0.0;
      float accum_212_3 = 0.0;
      float accum_212_4 = 0.0;
      float accum_212_5 = 0.0;
      float accum_212_6 = 0.0;
      float accum_212_7 = 0.0;
      float accum_212_8 = 0.0;
      for (int idx_chan_in_212 = threadIdx.x; idx_chan_in_212 < dim_1; idx_chan_in_212 += blockDim.x) {
        for (int idx_chan_out_212 = 0; idx_chan_out_212 < dim_2; idx_chan_out_212 += 1) {
          float l_212_0 = (-1)*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 2]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 1] + x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 1]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 2];
          float l_212_1 = x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 2]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 0] + (-1)*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 0]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 2];
          float l_212_2 = (-1)*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 1]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 0] + x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 0]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 1];
          float l_212_3 = (-1)*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 2]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 4] + x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 1]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 5];
          float l_212_4 = x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 2]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 3] + (-1)*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 0]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 5];
          float l_212_5 = (-1)*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 1]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 3] + x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 0]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 4];
          float l_212_6 = (-1)*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 2]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 7] + x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 1]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 8];
          float l_212_7 = x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 2]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 6] + (-1)*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 0]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 8];
          float l_212_8 = (-1)*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 1]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 6] + x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 0]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 7];
          float P_oi_212 = P_212[((idx_chan_out_212)*blockDim.y + threadIdx.y)*dim_1 + idx_chan_in_212];
          accum_212_0 += P_oi_212*l_212_0;
          accum_212_1 += P_oi_212*l_212_1;
          accum_212_2 += P_oi_212*l_212_2;
          accum_212_3 += P_oi_212*l_212_3;
          accum_212_4 += P_oi_212*l_212_4;
          accum_212_5 += P_oi_212*l_212_5;
          accum_212_6 += P_oi_212*l_212_6;
          accum_212_7 += P_oi_212*l_212_7;
          accum_212_8 += P_oi_212*l_212_8;
        }
      }
      // reduce across the warp so that first thread in warp will have the sum 
      for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
        accum_212_0 += __shfl_down_sync(0xffffffff, accum_212_0, offset);
        accum_212_1 += __shfl_down_sync(0xffffffff, accum_212_1, offset);
        accum_212_2 += __shfl_down_sync(0xffffffff, accum_212_2, offset);
        accum_212_3 += __shfl_down_sync(0xffffffff, accum_212_3, offset);
        accum_212_4 += __shfl_down_sync(0xffffffff, accum_212_4, offset);
        accum_212_5 += __shfl_down_sync(0xffffffff, accum_212_5, offset);
        accum_212_6 += __shfl_down_sync(0xffffffff, accum_212_6, offset);
        accum_212_7 += __shfl_down_sync(0xffffffff, accum_212_7, offset);
        accum_212_8 += __shfl_down_sync(0xffffffff, accum_212_8, offset);
      }
      if (threadIdx.x == 0) {
        dleft_212[((idx_batch)*dim_l + threadIdx.y)*9 + 0] = accum_212_0;
        dleft_212[((idx_batch)*dim_l + threadIdx.y)*9 + 1] = accum_212_1;
        dleft_212[((idx_batch)*dim_l + threadIdx.y)*9 + 2] = accum_212_2;
        dleft_212[((idx_batch)*dim_l + threadIdx.y)*9 + 3] = accum_212_3;
        dleft_212[((idx_batch)*dim_l + threadIdx.y)*9 + 4] = accum_212_4;
        dleft_212[((idx_batch)*dim_l + threadIdx.y)*9 + 5] = accum_212_5;
        dleft_212[((idx_batch)*dim_l + threadIdx.y)*9 + 6] = accum_212_6;
        dleft_212[((idx_batch)*dim_l + threadIdx.y)*9 + 7] = accum_212_7;
        dleft_212[((idx_batch)*dim_l + threadIdx.y)*9 + 8] = accum_212_8;
      }
    }
  }
}


void fused_tensor_prods_example_backleft(
    int batch, int dim_l, int dim_0, int dim_1, int dim_2,
    const float* x_0, const float* x_1, const float* x_2, const float* dy_0, const float* dy_1, const float* dy_2, const float* P_000, const float* P_011, const float* P_101, const float* P_110, const float* P_220, const float* P_222, const float* P_211, const float* P_111, const float* P_212,
    float* dleft_000, float* dleft_011, float* dleft_101, float* dleft_110, float* dleft_220, float* dleft_222, float* dleft_211, float* dleft_111, float* dleft_212) {
  
  
  int sharedmemsz = 0;
  dim3 gridsz = dim3(batch);
  dim3 blocksz = dim3(WARPSZ, dim_l);
  fused_tensor_prods_example_backleft_kern<<<gridsz, blocksz, sharedmemsz*sizeof(float)>>>(
      
      batch, dim_l, dim_0, dim_1, dim_2,
      x_0, x_1, x_2, dy_0, dy_1, dy_2, P_000, P_011, P_101, P_110, P_220, P_222, P_211, P_111, P_212,
      dleft_000, dleft_011, dleft_101, dleft_110, dleft_220, dleft_222, dleft_211, dleft_111, dleft_212);
  
}


__global__
void fused_tensor_prods_example_wtsback_kern(
    // <<<(WARPSZ, WARPSZ), (WARPSZ, dim_l)>>>
    
    int batch, int dim_l, int dim_0, int dim_1, int dim_2,
    const float* x_0, const float* x_1, const float* x_2, const float* dy_0, const float* dy_1, const float* dy_2, const float* left_000, const float* left_011, const float* left_101, const float* left_110, const float* left_220, const float* left_222, const float* left_211, const float* left_111, const float* left_212,
    float* __restrict__ dP_000, float* __restrict__ dP_011, float* __restrict__ dP_101, float* __restrict__ dP_110, float* __restrict__ dP_220, float* __restrict__ dP_222, float* __restrict__ dP_211, float* __restrict__ dP_111, float* __restrict__ dP_212) {
  extern __shared__ float s[];
  for (int idx_chan_in_000 = blockIdx.x; idx_chan_in_000 < dim_0; idx_chan_in_000 += gridDim.x) {
    for (int idx_chan_out_000 = blockIdx.y; idx_chan_out_000 < dim_0; idx_chan_out_000 += gridDim.y) {
      float dP_oi = 0.0;
      for (int idx_batch = threadIdx.x; idx_batch < batch; idx_batch += blockDim.x) {
        dP_oi += (left_000[((idx_batch)*dim_l + threadIdx.y)*1 + 0]*x_0[((idx_batch)*dim_0 + idx_chan_in_000)*1 + 0]*dy_0[((idx_batch)*dim_0 + idx_chan_out_000)*1 + 0]);
      }
      // reduce across the warp so that first thread in warp will have the sum 
      for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
        dP_oi += __shfl_down_sync(0xffffffff, dP_oi, offset);
      }
      if (threadIdx.x == 0) {
        dP_000[((idx_chan_out_000)*blockDim.y + threadIdx.y)*dim_0 + idx_chan_in_000] = dP_oi;
      }
    }
  }
  for (int idx_chan_in_011 = blockIdx.x; idx_chan_in_011 < dim_1; idx_chan_in_011 += gridDim.x) {
    for (int idx_chan_out_011 = blockIdx.y; idx_chan_out_011 < dim_1; idx_chan_out_011 += gridDim.y) {
      float dP_oi = 0.0;
      for (int idx_batch = threadIdx.x; idx_batch < batch; idx_batch += blockDim.x) {
        dP_oi += (left_011[((idx_batch)*dim_l + threadIdx.y)*1 + 0]*x_1[((idx_batch)*dim_1 + idx_chan_in_011)*3 + 0]*dy_1[((idx_batch)*dim_1 + idx_chan_out_011)*3 + 0] + left_011[((idx_batch)*dim_l + threadIdx.y)*1 + 0]*x_1[((idx_batch)*dim_1 + idx_chan_in_011)*3 + 1]*dy_1[((idx_batch)*dim_1 + idx_chan_out_011)*3 + 1] + left_011[((idx_batch)*dim_l + threadIdx.y)*1 + 0]*x_1[((idx_batch)*dim_1 + idx_chan_in_011)*3 + 2]*dy_1[((idx_batch)*dim_1 + idx_chan_out_011)*3 + 2]);
      }
      // reduce across the warp so that first thread in warp will have the sum 
      for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
        dP_oi += __shfl_down_sync(0xffffffff, dP_oi, offset);
      }
      if (threadIdx.x == 0) {
        dP_011[((idx_chan_out_011)*blockDim.y + threadIdx.y)*dim_1 + idx_chan_in_011] = dP_oi;
      }
    }
  }
  for (int idx_chan_in_101 = blockIdx.x; idx_chan_in_101 < dim_0; idx_chan_in_101 += gridDim.x) {
    for (int idx_chan_out_101 = blockIdx.y; idx_chan_out_101 < dim_1; idx_chan_out_101 += gridDim.y) {
      float dP_oi = 0.0;
      for (int idx_batch = threadIdx.x; idx_batch < batch; idx_batch += blockDim.x) {
        dP_oi += (left_101[((idx_batch)*dim_l + threadIdx.y)*3 + 0]*x_0[((idx_batch)*dim_0 + idx_chan_in_101)*1 + 0]*dy_1[((idx_batch)*dim_1 + idx_chan_out_101)*3 + 0] + left_101[((idx_batch)*dim_l + threadIdx.y)*3 + 1]*x_0[((idx_batch)*dim_0 + idx_chan_in_101)*1 + 0]*dy_1[((idx_batch)*dim_1 + idx_chan_out_101)*3 + 1] + left_101[((idx_batch)*dim_l + threadIdx.y)*3 + 2]*x_0[((idx_batch)*dim_0 + idx_chan_in_101)*1 + 0]*dy_1[((idx_batch)*dim_1 + idx_chan_out_101)*3 + 2]);
      }
      // reduce across the warp so that first thread in warp will have the sum 
      for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
        dP_oi += __shfl_down_sync(0xffffffff, dP_oi, offset);
      }
      if (threadIdx.x == 0) {
        dP_101[((idx_chan_out_101)*blockDim.y + threadIdx.y)*dim_0 + idx_chan_in_101] = dP_oi;
      }
    }
  }
  for (int idx_chan_in_110 = blockIdx.x; idx_chan_in_110 < dim_1; idx_chan_in_110 += gridDim.x) {
    for (int idx_chan_out_110 = blockIdx.y; idx_chan_out_110 < dim_0; idx_chan_out_110 += gridDim.y) {
      float dP_oi = 0.0;
      for (int idx_batch = threadIdx.x; idx_batch < batch; idx_batch += blockDim.x) {
        dP_oi += (left_110[((idx_batch)*dim_l + threadIdx.y)*3 + 0]*x_1[((idx_batch)*dim_1 + idx_chan_in_110)*3 + 0]*dy_0[((idx_batch)*dim_0 + idx_chan_out_110)*1 + 0] + left_110[((idx_batch)*dim_l + threadIdx.y)*3 + 1]*x_1[((idx_batch)*dim_1 + idx_chan_in_110)*3 + 1]*dy_0[((idx_batch)*dim_0 + idx_chan_out_110)*1 + 0] + left_110[((idx_batch)*dim_l + threadIdx.y)*3 + 2]*x_1[((idx_batch)*dim_1 + idx_chan_in_110)*3 + 2]*dy_0[((idx_batch)*dim_0 + idx_chan_out_110)*1 + 0]);
      }
      // reduce across the warp so that first thread in warp will have the sum 
      for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
        dP_oi += __shfl_down_sync(0xffffffff, dP_oi, offset);
      }
      if (threadIdx.x == 0) {
        dP_110[((idx_chan_out_110)*blockDim.y + threadIdx.y)*dim_1 + idx_chan_in_110] = dP_oi;
      }
    }
  }
  for (int idx_chan_in_220 = blockIdx.x; idx_chan_in_220 < dim_2; idx_chan_in_220 += gridDim.x) {
    for (int idx_chan_out_220 = blockIdx.y; idx_chan_out_220 < dim_0; idx_chan_out_220 += gridDim.y) {
      float dP_oi = 0.0;
      for (int idx_batch = threadIdx.x; idx_batch < batch; idx_batch += blockDim.x) {
        dP_oi += (left_220[((idx_batch)*dim_l + threadIdx.y)*9 + 0]*x_2[((idx_batch)*dim_2 + idx_chan_in_220)*9 + 0]*dy_0[((idx_batch)*dim_0 + idx_chan_out_220)*1 + 0] + left_220[((idx_batch)*dim_l + threadIdx.y)*9 + 1]*x_2[((idx_batch)*dim_2 + idx_chan_in_220)*9 + 1]*dy_0[((idx_batch)*dim_0 + idx_chan_out_220)*1 + 0] + left_220[((idx_batch)*dim_l + threadIdx.y)*9 + 2]*x_2[((idx_batch)*dim_2 + idx_chan_in_220)*9 + 2]*dy_0[((idx_batch)*dim_0 + idx_chan_out_220)*1 + 0] + left_220[((idx_batch)*dim_l + threadIdx.y)*9 + 3]*x_2[((idx_batch)*dim_2 + idx_chan_in_220)*9 + 3]*dy_0[((idx_batch)*dim_0 + idx_chan_out_220)*1 + 0] + left_220[((idx_batch)*dim_l + threadIdx.y)*9 + 4]*x_2[((idx_batch)*dim_2 + idx_chan_in_220)*9 + 4]*dy_0[((idx_batch)*dim_0 + idx_chan_out_220)*1 + 0] + left_220[((idx_batch)*dim_l + threadIdx.y)*9 + 5]*x_2[((idx_batch)*dim_2 + idx_chan_in_220)*9 + 5]*dy_0[((idx_batch)*dim_0 + idx_chan_out_220)*1 + 0] + left_220[((idx_batch)*dim_l + threadIdx.y)*9 + 6]*x_2[((idx_batch)*dim_2 + idx_chan_in_220)*9 + 6]*dy_0[((idx_batch)*dim_0 + idx_chan_out_220)*1 + 0] + left_220[((idx_batch)*dim_l + threadIdx.y)*9 + 7]*x_2[((idx_batch)*dim_2 + idx_chan_in_220)*9 + 7]*dy_0[((idx_batch)*dim_0 + idx_chan_out_220)*1 + 0] + left_220[((idx_batch)*dim_l + threadIdx.y)*9 + 8]*x_2[((idx_batch)*dim_2 + idx_chan_in_220)*9 + 8]*dy_0[((idx_batch)*dim_0 + idx_chan_out_220)*1 + 0]);
      }
      // reduce across the warp so that first thread in warp will have the sum 
      for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
        dP_oi += __shfl_down_sync(0xffffffff, dP_oi, offset);
      }
      if (threadIdx.x == 0) {
        dP_220[((idx_chan_out_220)*blockDim.y + threadIdx.y)*dim_2 + idx_chan_in_220] = dP_oi;
      }
    }
  }
  for (int idx_chan_in_222 = blockIdx.x; idx_chan_in_222 < dim_2; idx_chan_in_222 += gridDim.x) {
    for (int idx_chan_out_222 = blockIdx.y; idx_chan_out_222 < dim_2; idx_chan_out_222 += gridDim.y) {
      float dP_oi = 0.0;
      for (int idx_batch = threadIdx.x; idx_batch < batch; idx_batch += blockDim.x) {
        dP_oi += (left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 0]*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 0]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 0] + left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 1]*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 1]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 0] + left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 2]*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 2]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 0] + left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 0]*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 3]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 1] + left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 1]*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 4]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 1] + left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 2]*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 5]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 1] + left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 0]*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 6]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 2] + left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 1]*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 7]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 2] + left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 2]*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 8]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 2] + left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 3]*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 0]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 3] + left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 4]*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 1]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 3] + left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 5]*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 2]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 3] + left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 3]*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 3]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 4] + left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 4]*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 4]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 4] + left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 5]*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 5]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 4] + left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 3]*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 6]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 5] + left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 4]*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 7]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 5] + left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 5]*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 8]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 5] + left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 6]*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 0]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 6] + left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 7]*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 1]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 6] + left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 8]*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 2]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 6] + left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 6]*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 3]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 7] + left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 7]*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 4]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 7] + left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 8]*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 5]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 7] + left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 6]*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 6]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 8] + left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 7]*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 7]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 8] + left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 8]*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 8]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 8]);
      }
      // reduce across the warp so that first thread in warp will have the sum 
      for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
        dP_oi += __shfl_down_sync(0xffffffff, dP_oi, offset);
      }
      if (threadIdx.x == 0) {
        dP_222[((idx_chan_out_222)*blockDim.y + threadIdx.y)*dim_2 + idx_chan_in_222] = dP_oi;
      }
    }
  }
  for (int idx_chan_in_211 = blockIdx.x; idx_chan_in_211 < dim_1; idx_chan_in_211 += gridDim.x) {
    for (int idx_chan_out_211 = blockIdx.y; idx_chan_out_211 < dim_1; idx_chan_out_211 += gridDim.y) {
      float dP_oi = 0.0;
      for (int idx_batch = threadIdx.x; idx_batch < batch; idx_batch += blockDim.x) {
        dP_oi += (left_211[((idx_batch)*dim_l + threadIdx.y)*9 + 0]*x_1[((idx_batch)*dim_1 + idx_chan_in_211)*3 + 0]*dy_1[((idx_batch)*dim_1 + idx_chan_out_211)*3 + 0] + left_211[((idx_batch)*dim_l + threadIdx.y)*9 + 1]*x_1[((idx_batch)*dim_1 + idx_chan_in_211)*3 + 1]*dy_1[((idx_batch)*dim_1 + idx_chan_out_211)*3 + 0] + left_211[((idx_batch)*dim_l + threadIdx.y)*9 + 2]*x_1[((idx_batch)*dim_1 + idx_chan_in_211)*3 + 2]*dy_1[((idx_batch)*dim_1 + idx_chan_out_211)*3 + 0] + left_211[((idx_batch)*dim_l + threadIdx.y)*9 + 3]*x_1[((idx_batch)*dim_1 + idx_chan_in_211)*3 + 0]*dy_1[((idx_batch)*dim_1 + idx_chan_out_211)*3 + 1] + left_211[((idx_batch)*dim_l + threadIdx.y)*9 + 4]*x_1[((idx_batch)*dim_1 + idx_chan_in_211)*3 + 1]*dy_1[((idx_batch)*dim_1 + idx_chan_out_211)*3 + 1] + left_211[((idx_batch)*dim_l + threadIdx.y)*9 + 5]*x_1[((idx_batch)*dim_1 + idx_chan_in_211)*3 + 2]*dy_1[((idx_batch)*dim_1 + idx_chan_out_211)*3 + 1] + left_211[((idx_batch)*dim_l + threadIdx.y)*9 + 6]*x_1[((idx_batch)*dim_1 + idx_chan_in_211)*3 + 0]*dy_1[((idx_batch)*dim_1 + idx_chan_out_211)*3 + 2] + left_211[((idx_batch)*dim_l + threadIdx.y)*9 + 7]*x_1[((idx_batch)*dim_1 + idx_chan_in_211)*3 + 1]*dy_1[((idx_batch)*dim_1 + idx_chan_out_211)*3 + 2] + left_211[((idx_batch)*dim_l + threadIdx.y)*9 + 8]*x_1[((idx_batch)*dim_1 + idx_chan_in_211)*3 + 2]*dy_1[((idx_batch)*dim_1 + idx_chan_out_211)*3 + 2]);
      }
      // reduce across the warp so that first thread in warp will have the sum 
      for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
        dP_oi += __shfl_down_sync(0xffffffff, dP_oi, offset);
      }
      if (threadIdx.x == 0) {
        dP_211[((idx_chan_out_211)*blockDim.y + threadIdx.y)*dim_1 + idx_chan_in_211] = dP_oi;
      }
    }
  }
  for (int idx_chan_in_111 = blockIdx.x; idx_chan_in_111 < dim_1; idx_chan_in_111 += gridDim.x) {
    for (int idx_chan_out_111 = blockIdx.y; idx_chan_out_111 < dim_1; idx_chan_out_111 += gridDim.y) {
      float dP_oi = 0.0;
      for (int idx_batch = threadIdx.x; idx_batch < batch; idx_batch += blockDim.x) {
        dP_oi += (left_111[((idx_batch)*dim_l + threadIdx.y)*3 + 1]*x_1[((idx_batch)*dim_1 + idx_chan_in_111)*3 + 2]*dy_1[((idx_batch)*dim_1 + idx_chan_out_111)*3 + 0] + (-1)*left_111[((idx_batch)*dim_l + threadIdx.y)*3 + 2]*x_1[((idx_batch)*dim_1 + idx_chan_in_111)*3 + 1]*dy_1[((idx_batch)*dim_1 + idx_chan_out_111)*3 + 0] + (-1)*left_111[((idx_batch)*dim_l + threadIdx.y)*3 + 0]*x_1[((idx_batch)*dim_1 + idx_chan_in_111)*3 + 2]*dy_1[((idx_batch)*dim_1 + idx_chan_out_111)*3 + 1] + left_111[((idx_batch)*dim_l + threadIdx.y)*3 + 2]*x_1[((idx_batch)*dim_1 + idx_chan_in_111)*3 + 0]*dy_1[((idx_batch)*dim_1 + idx_chan_out_111)*3 + 1] + left_111[((idx_batch)*dim_l + threadIdx.y)*3 + 0]*x_1[((idx_batch)*dim_1 + idx_chan_in_111)*3 + 1]*dy_1[((idx_batch)*dim_1 + idx_chan_out_111)*3 + 2] + (-1)*left_111[((idx_batch)*dim_l + threadIdx.y)*3 + 1]*x_1[((idx_batch)*dim_1 + idx_chan_in_111)*3 + 0]*dy_1[((idx_batch)*dim_1 + idx_chan_out_111)*3 + 2]);
      }
      // reduce across the warp so that first thread in warp will have the sum 
      for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
        dP_oi += __shfl_down_sync(0xffffffff, dP_oi, offset);
      }
      if (threadIdx.x == 0) {
        dP_111[((idx_chan_out_111)*blockDim.y + threadIdx.y)*dim_1 + idx_chan_in_111] = dP_oi;
      }
    }
  }
  for (int idx_chan_in_212 = blockIdx.x; idx_chan_in_212 < dim_1; idx_chan_in_212 += gridDim.x) {
    for (int idx_chan_out_212 = blockIdx.y; idx_chan_out_212 < dim_2; idx_chan_out_212 += gridDim.y) {
      float dP_oi = 0.0;
      for (int idx_batch = threadIdx.x; idx_batch < batch; idx_batch += blockDim.x) {
        dP_oi += (left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 1]*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 2]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 0] + (-1)*left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 2]*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 1]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 0] + (-1)*left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 0]*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 2]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 1] + left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 2]*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 0]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 1] + left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 0]*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 1]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 2] + (-1)*left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 1]*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 0]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 2] + left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 4]*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 2]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 3] + (-1)*left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 5]*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 1]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 3] + (-1)*left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 3]*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 2]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 4] + left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 5]*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 0]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 4] + left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 3]*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 1]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 5] + (-1)*left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 4]*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 0]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 5] + left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 7]*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 2]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 6] + (-1)*left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 8]*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 1]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 6] + (-1)*left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 6]*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 2]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 7] + left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 8]*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 0]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 7] + left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 6]*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 1]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 8] + (-1)*left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 7]*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 0]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 8]);
      }
      // reduce across the warp so that first thread in warp will have the sum 
      for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
        dP_oi += __shfl_down_sync(0xffffffff, dP_oi, offset);
      }
      if (threadIdx.x == 0) {
        dP_212[((idx_chan_out_212)*blockDim.y + threadIdx.y)*dim_1 + idx_chan_in_212] = dP_oi;
      }
    }
  }
}


void fused_tensor_prods_example_wtsback(
    int batch, int dim_l, int dim_0, int dim_1, int dim_2,
    const float* x_0, const float* x_1, const float* x_2, const float* dy_0, const float* dy_1, const float* dy_2, const float* left_000, const float* left_011, const float* left_101, const float* left_110, const float* left_220, const float* left_222, const float* left_211, const float* left_111, const float* left_212,
    float* dP_000, float* dP_011, float* dP_101, float* dP_110, float* dP_220, float* dP_222, float* dP_211, float* dP_111, float* dP_212) {
  
  
  int sharedmemsz = 0;
  dim3 gridsz = dim3(WARPSZ, WARPSZ);
  dim3 blocksz = dim3(WARPSZ, dim_l);
  fused_tensor_prods_example_wtsback_kern<<<gridsz, blocksz, sharedmemsz*sizeof(float)>>>(
      
      batch, dim_l, dim_0, dim_1, dim_2,
      x_0, x_1, x_2, dy_0, dy_1, dy_2, left_000, left_011, left_101, left_110, left_220, left_222, left_211, left_111, left_212,
      dP_000, dP_011, dP_101, dP_110, dP_220, dP_222, dP_211, dP_111, dP_212);
  
}


__global__
void ant16_o0_kern(
    // <<<(batch), (WARPSZ, dim_l)>>>
    int p_0, int p_1_base, int p_1, int p_2_base, int p_2, 
    int batch, int dim_l, int dim_0, int dim_1, int dim_2,
    const float* x_0, const float* x_1, const float* x_2, const float* P_000, const float* left_000, const float* P_110, const float* left_110, const float* P_220, const float* left_220,
    float* __restrict__ y_0) {
  extern __shared__ float s[];
  float* product_000 = &s[0*p_0]; // size = 1*p_0
  float* product_110 = &s[p_1_base + 0*p_1]; // size = 1*p_1
  float* product_220 = &s[p_2_base + 0*p_2]; // size = 1*p_2
  for (int idx_batch = blockIdx.x; idx_batch < batch; idx_batch += gridDim.x) {
    { // compute tensor products
      float l_000_0 = left_000[((idx_batch)*dim_l + threadIdx.y)*1 + 0];
      for (int idx_chan_in_000 = threadIdx.x; idx_chan_in_000 < dim_0; idx_chan_in_000 += blockDim.x) {
        product_000[((threadIdx.y)*dim_0 + idx_chan_in_000)*1 + 0] = l_000_0*x_0[((idx_batch)*dim_0 + idx_chan_in_000)*1 + 0];
      }
      float l_110_0 = left_110[((idx_batch)*dim_l + threadIdx.y)*3 + 0];
      float l_110_1 = left_110[((idx_batch)*dim_l + threadIdx.y)*3 + 1];
      float l_110_2 = left_110[((idx_batch)*dim_l + threadIdx.y)*3 + 2];
      for (int idx_chan_in_110 = threadIdx.x; idx_chan_in_110 < dim_1; idx_chan_in_110 += blockDim.x) {
        product_110[((threadIdx.y)*dim_1 + idx_chan_in_110)*1 + 0] = l_110_0*x_1[((idx_batch)*dim_1 + idx_chan_in_110)*3 + 0] + l_110_1*x_1[((idx_batch)*dim_1 + idx_chan_in_110)*3 + 1] + l_110_2*x_1[((idx_batch)*dim_1 + idx_chan_in_110)*3 + 2];
      }
      float l_220_0 = left_220[((idx_batch)*dim_l + threadIdx.y)*9 + 0];
      float l_220_1 = left_220[((idx_batch)*dim_l + threadIdx.y)*9 + 1];
      float l_220_2 = left_220[((idx_batch)*dim_l + threadIdx.y)*9 + 2];
      float l_220_3 = left_220[((idx_batch)*dim_l + threadIdx.y)*9 + 3];
      float l_220_4 = left_220[((idx_batch)*dim_l + threadIdx.y)*9 + 4];
      float l_220_5 = left_220[((idx_batch)*dim_l + threadIdx.y)*9 + 5];
      float l_220_6 = left_220[((idx_batch)*dim_l + threadIdx.y)*9 + 6];
      float l_220_7 = left_220[((idx_batch)*dim_l + threadIdx.y)*9 + 7];
      float l_220_8 = left_220[((idx_batch)*dim_l + threadIdx.y)*9 + 8];
      for (int idx_chan_in_220 = threadIdx.x; idx_chan_in_220 < dim_2; idx_chan_in_220 += blockDim.x) {
        product_220[((threadIdx.y)*dim_2 + idx_chan_in_220)*1 + 0] = l_220_0*x_2[((idx_batch)*dim_2 + idx_chan_in_220)*9 + 0] + l_220_1*x_2[((idx_batch)*dim_2 + idx_chan_in_220)*9 + 1] + l_220_2*x_2[((idx_batch)*dim_2 + idx_chan_in_220)*9 + 2] + l_220_3*x_2[((idx_batch)*dim_2 + idx_chan_in_220)*9 + 3] + l_220_4*x_2[((idx_batch)*dim_2 + idx_chan_in_220)*9 + 4] + l_220_5*x_2[((idx_batch)*dim_2 + idx_chan_in_220)*9 + 5] + l_220_6*x_2[((idx_batch)*dim_2 + idx_chan_in_220)*9 + 6] + l_220_7*x_2[((idx_batch)*dim_2 + idx_chan_in_220)*9 + 7] + l_220_8*x_2[((idx_batch)*dim_2 + idx_chan_in_220)*9 + 8];
      }
    }
    __syncthreads();
    { // linear transforms to compute the outputs
      for (int idx_chan_out_0 = threadIdx.y; idx_chan_out_0 < dim_0; idx_chan_out_0 += blockDim.y) {
        float y_o_0_0 = 0.0;
        float accum_000_0 = 0.0;
        for (int idx_chan_in_000 = threadIdx.x; idx_chan_in_000 < dim_l*dim_0; idx_chan_in_000 += blockDim.x) {
          float P_oi_000 = P_000[(idx_chan_out_0)*dim_l*dim_0 + idx_chan_in_000];
          accum_000_0 += P_oi_000*product_000[(idx_chan_in_000)*1 + 0];
        }
        // reduce across the warp so that first thread in warp will have the sum 
        for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
          accum_000_0 += __shfl_down_sync(0xffffffff, accum_000_0, offset);
        }
        if (threadIdx.x == 0) {
          y_o_0_0 += accum_000_0;
        }
        float accum_110_0 = 0.0;
        for (int idx_chan_in_110 = threadIdx.x; idx_chan_in_110 < dim_l*dim_1; idx_chan_in_110 += blockDim.x) {
          float P_oi_110 = P_110[(idx_chan_out_0)*dim_l*dim_1 + idx_chan_in_110];
          accum_110_0 += P_oi_110*product_110[(idx_chan_in_110)*1 + 0];
        }
        // reduce across the warp so that first thread in warp will have the sum 
        for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
          accum_110_0 += __shfl_down_sync(0xffffffff, accum_110_0, offset);
        }
        if (threadIdx.x == 0) {
          y_o_0_0 += accum_110_0;
        }
        float accum_220_0 = 0.0;
        for (int idx_chan_in_220 = threadIdx.x; idx_chan_in_220 < dim_l*dim_2; idx_chan_in_220 += blockDim.x) {
          float P_oi_220 = P_220[(idx_chan_out_0)*dim_l*dim_2 + idx_chan_in_220];
          accum_220_0 += P_oi_220*product_220[(idx_chan_in_220)*1 + 0];
        }
        // reduce across the warp so that first thread in warp will have the sum 
        for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
          accum_220_0 += __shfl_down_sync(0xffffffff, accum_220_0, offset);
        }
        if (threadIdx.x == 0) {
          y_o_0_0 += accum_220_0;
        }
        if (threadIdx.x == 0) {
          y_0[((idx_batch)*dim_0 + idx_chan_out_0)*1 + 0] = y_o_0_0;
        }
      }
    }
  }
}


void ant16_o0(
    int batch, int dim_l, int dim_0, int dim_1, int dim_2,
    const float* x_0, const float* x_1, const float* x_2, const float* P_000, const float* left_000, const float* P_110, const float* left_110, const float* P_220, const float* left_220,
    float* y_0) {
  
  int p_0 = dim_l*dim_0;
  int p_1 = dim_l*dim_1;
  int p_2 = dim_l*dim_2;
  int sharedmemsz = 0;
  sharedmemsz += 1*p_0;
  int p_1_base = sharedmemsz;
  sharedmemsz += 1*p_1;
  int p_2_base = sharedmemsz;
  sharedmemsz += 1*p_2;
  dim3 gridsz = dim3(batch);
  dim3 blocksz = dim3(WARPSZ, dim_l);
  ant16_o0_kern<<<gridsz, blocksz, sharedmemsz*sizeof(float)>>>(
      p_0, p_1_base, p_1, p_2_base, p_2, 
      batch, dim_l, dim_0, dim_1, dim_2,
      x_0, x_1, x_2, P_000, left_000, P_110, left_110, P_220, left_220,
      y_0);
  
}


__global__
void ant16_o0_backward_kern(
    // <<<(batch), (WARPSZ, dim_l)>>>
    int p_0, 
    int batch, int dim_l, int dim_0, int dim_1, int dim_2,
    const float* dy_0, const float* P_000, const float* left_000, const float* P_110, const float* left_110, const float* P_220, const float* left_220,
    float* __restrict__ dx_0, float* __restrict__ dx_1, float* __restrict__ dx_2) {
  extern __shared__ float s[];
  float* dproduct_000 = &s[0*p_0]; // size = 1*p_0
  float* dproduct_110 = &s[1*p_0]; // size = 3*p_0
  float* dproduct_220 = &s[4*p_0]; // size = 9*p_0
  for (int idx_batch = blockIdx.x; idx_batch < batch; idx_batch += gridDim.x) {
    { // compute tensor products
      float l_000_0 = left_000[((idx_batch)*dim_l + threadIdx.y)*1 + 0];
      for (int idx_chan_out_000 = threadIdx.x; idx_chan_out_000 < dim_0; idx_chan_out_000 += blockDim.x) {
        dproduct_000[((threadIdx.y)*dim_0 + idx_chan_out_000)*1 + 0] = l_000_0*dy_0[((idx_batch)*dim_0 + idx_chan_out_000)*1 + 0];
      }
      float l_110_0 = left_110[((idx_batch)*dim_l + threadIdx.y)*3 + 0];
      float l_110_1 = left_110[((idx_batch)*dim_l + threadIdx.y)*3 + 1];
      float l_110_2 = left_110[((idx_batch)*dim_l + threadIdx.y)*3 + 2];
      for (int idx_chan_out_110 = threadIdx.x; idx_chan_out_110 < dim_0; idx_chan_out_110 += blockDim.x) {
        dproduct_110[((threadIdx.y)*dim_0 + idx_chan_out_110)*3 + 0] = l_110_0*dy_0[((idx_batch)*dim_0 + idx_chan_out_110)*1 + 0];
        dproduct_110[((threadIdx.y)*dim_0 + idx_chan_out_110)*3 + 1] = l_110_1*dy_0[((idx_batch)*dim_0 + idx_chan_out_110)*1 + 0];
        dproduct_110[((threadIdx.y)*dim_0 + idx_chan_out_110)*3 + 2] = l_110_2*dy_0[((idx_batch)*dim_0 + idx_chan_out_110)*1 + 0];
      }
      float l_220_0 = left_220[((idx_batch)*dim_l + threadIdx.y)*9 + 0];
      float l_220_1 = left_220[((idx_batch)*dim_l + threadIdx.y)*9 + 1];
      float l_220_2 = left_220[((idx_batch)*dim_l + threadIdx.y)*9 + 2];
      float l_220_3 = left_220[((idx_batch)*dim_l + threadIdx.y)*9 + 3];
      float l_220_4 = left_220[((idx_batch)*dim_l + threadIdx.y)*9 + 4];
      float l_220_5 = left_220[((idx_batch)*dim_l + threadIdx.y)*9 + 5];
      float l_220_6 = left_220[((idx_batch)*dim_l + threadIdx.y)*9 + 6];
      float l_220_7 = left_220[((idx_batch)*dim_l + threadIdx.y)*9 + 7];
      float l_220_8 = left_220[((idx_batch)*dim_l + threadIdx.y)*9 + 8];
      for (int idx_chan_out_220 = threadIdx.x; idx_chan_out_220 < dim_0; idx_chan_out_220 += blockDim.x) {
        dproduct_220[((threadIdx.y)*dim_0 + idx_chan_out_220)*9 + 0] = l_220_0*dy_0[((idx_batch)*dim_0 + idx_chan_out_220)*1 + 0];
        dproduct_220[((threadIdx.y)*dim_0 + idx_chan_out_220)*9 + 1] = l_220_1*dy_0[((idx_batch)*dim_0 + idx_chan_out_220)*1 + 0];
        dproduct_220[((threadIdx.y)*dim_0 + idx_chan_out_220)*9 + 2] = l_220_2*dy_0[((idx_batch)*dim_0 + idx_chan_out_220)*1 + 0];
        dproduct_220[((threadIdx.y)*dim_0 + idx_chan_out_220)*9 + 3] = l_220_3*dy_0[((idx_batch)*dim_0 + idx_chan_out_220)*1 + 0];
        dproduct_220[((threadIdx.y)*dim_0 + idx_chan_out_220)*9 + 4] = l_220_4*dy_0[((idx_batch)*dim_0 + idx_chan_out_220)*1 + 0];
        dproduct_220[((threadIdx.y)*dim_0 + idx_chan_out_220)*9 + 5] = l_220_5*dy_0[((idx_batch)*dim_0 + idx_chan_out_220)*1 + 0];
        dproduct_220[((threadIdx.y)*dim_0 + idx_chan_out_220)*9 + 6] = l_220_6*dy_0[((idx_batch)*dim_0 + idx_chan_out_220)*1 + 0];
        dproduct_220[((threadIdx.y)*dim_0 + idx_chan_out_220)*9 + 7] = l_220_7*dy_0[((idx_batch)*dim_0 + idx_chan_out_220)*1 + 0];
        dproduct_220[((threadIdx.y)*dim_0 + idx_chan_out_220)*9 + 8] = l_220_8*dy_0[((idx_batch)*dim_0 + idx_chan_out_220)*1 + 0];
      }
    }
    __syncthreads();
    { // linear transforms to compute dx
      for (int idx_chan_in_0 = threadIdx.y; idx_chan_in_0 < dim_0; idx_chan_in_0 += blockDim.y) {
        float dx_o_0_0 = 0.0;
        float accum_000_0 = 0.0;
        for (int idx_l_000 = 0; idx_l_000 < dim_l; idx_l_000 += 1) {
          for (int idx_chan_out_000 = threadIdx.x; idx_chan_out_000 < dim_0; idx_chan_out_000 += blockDim.x) {
            float P_oi_000 = P_000[((idx_chan_out_000)*dim_l + idx_l_000)*dim_0 + idx_chan_in_0];
            accum_000_0 += P_oi_000*dproduct_000[((idx_l_000)*dim_0 + idx_chan_out_000)*1 + 0];
          }
        }
        // reduce across the warp so that first thread in warp will have the sum 
        for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
          accum_000_0 += __shfl_down_sync(0xffffffff, accum_000_0, offset);
        }
        if (threadIdx.x == 0) {
          dx_o_0_0 += accum_000_0;
        }
        if (threadIdx.x == 0) {
          dx_0[((idx_batch)*dim_0 + idx_chan_in_0)*1 + 0] = dx_o_0_0;
        }
      }
      for (int idx_chan_in_1 = threadIdx.y; idx_chan_in_1 < dim_1; idx_chan_in_1 += blockDim.y) {
        float dx_o_1_0 = 0.0;
        float dx_o_1_1 = 0.0;
        float dx_o_1_2 = 0.0;
        float accum_110_0 = 0.0;
        float accum_110_1 = 0.0;
        float accum_110_2 = 0.0;
        for (int idx_l_110 = 0; idx_l_110 < dim_l; idx_l_110 += 1) {
          for (int idx_chan_out_110 = threadIdx.x; idx_chan_out_110 < dim_0; idx_chan_out_110 += blockDim.x) {
            float P_oi_110 = P_110[((idx_chan_out_110)*dim_l + idx_l_110)*dim_1 + idx_chan_in_1];
            accum_110_0 += P_oi_110*dproduct_110[((idx_l_110)*dim_0 + idx_chan_out_110)*3 + 0];
            accum_110_1 += P_oi_110*dproduct_110[((idx_l_110)*dim_0 + idx_chan_out_110)*3 + 1];
            accum_110_2 += P_oi_110*dproduct_110[((idx_l_110)*dim_0 + idx_chan_out_110)*3 + 2];
          }
        }
        // reduce across the warp so that first thread in warp will have the sum 
        for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
          accum_110_0 += __shfl_down_sync(0xffffffff, accum_110_0, offset);
          accum_110_1 += __shfl_down_sync(0xffffffff, accum_110_1, offset);
          accum_110_2 += __shfl_down_sync(0xffffffff, accum_110_2, offset);
        }
        if (threadIdx.x == 0) {
          dx_o_1_0 += accum_110_0;
          dx_o_1_1 += accum_110_1;
          dx_o_1_2 += accum_110_2;
        }
        if (threadIdx.x == 0) {
          dx_1[((idx_batch)*dim_1 + idx_chan_in_1)*3 + 0] = dx_o_1_0;
          dx_1[((idx_batch)*dim_1 + idx_chan_in_1)*3 + 1] = dx_o_1_1;
          dx_1[((idx_batch)*dim_1 + idx_chan_in_1)*3 + 2] = dx_o_1_2;
        }
      }
      for (int idx_chan_in_2 = threadIdx.y; idx_chan_in_2 < dim_2; idx_chan_in_2 += blockDim.y) {
        float dx_o_2_0 = 0.0;
        float dx_o_2_1 = 0.0;
        float dx_o_2_2 = 0.0;
        float dx_o_2_3 = 0.0;
        float dx_o_2_4 = 0.0;
        float dx_o_2_5 = 0.0;
        float dx_o_2_6 = 0.0;
        float dx_o_2_7 = 0.0;
        float dx_o_2_8 = 0.0;
        float accum_220_0 = 0.0;
        float accum_220_1 = 0.0;
        float accum_220_2 = 0.0;
        float accum_220_3 = 0.0;
        float accum_220_4 = 0.0;
        float accum_220_5 = 0.0;
        float accum_220_6 = 0.0;
        float accum_220_7 = 0.0;
        float accum_220_8 = 0.0;
        for (int idx_l_220 = 0; idx_l_220 < dim_l; idx_l_220 += 1) {
          for (int idx_chan_out_220 = threadIdx.x; idx_chan_out_220 < dim_0; idx_chan_out_220 += blockDim.x) {
            float P_oi_220 = P_220[((idx_chan_out_220)*dim_l + idx_l_220)*dim_2 + idx_chan_in_2];
            accum_220_0 += P_oi_220*dproduct_220[((idx_l_220)*dim_0 + idx_chan_out_220)*9 + 0];
            accum_220_1 += P_oi_220*dproduct_220[((idx_l_220)*dim_0 + idx_chan_out_220)*9 + 1];
            accum_220_2 += P_oi_220*dproduct_220[((idx_l_220)*dim_0 + idx_chan_out_220)*9 + 2];
            accum_220_3 += P_oi_220*dproduct_220[((idx_l_220)*dim_0 + idx_chan_out_220)*9 + 3];
            accum_220_4 += P_oi_220*dproduct_220[((idx_l_220)*dim_0 + idx_chan_out_220)*9 + 4];
            accum_220_5 += P_oi_220*dproduct_220[((idx_l_220)*dim_0 + idx_chan_out_220)*9 + 5];
            accum_220_6 += P_oi_220*dproduct_220[((idx_l_220)*dim_0 + idx_chan_out_220)*9 + 6];
            accum_220_7 += P_oi_220*dproduct_220[((idx_l_220)*dim_0 + idx_chan_out_220)*9 + 7];
            accum_220_8 += P_oi_220*dproduct_220[((idx_l_220)*dim_0 + idx_chan_out_220)*9 + 8];
          }
        }
        // reduce across the warp so that first thread in warp will have the sum 
        for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
          accum_220_0 += __shfl_down_sync(0xffffffff, accum_220_0, offset);
          accum_220_1 += __shfl_down_sync(0xffffffff, accum_220_1, offset);
          accum_220_2 += __shfl_down_sync(0xffffffff, accum_220_2, offset);
          accum_220_3 += __shfl_down_sync(0xffffffff, accum_220_3, offset);
          accum_220_4 += __shfl_down_sync(0xffffffff, accum_220_4, offset);
          accum_220_5 += __shfl_down_sync(0xffffffff, accum_220_5, offset);
          accum_220_6 += __shfl_down_sync(0xffffffff, accum_220_6, offset);
          accum_220_7 += __shfl_down_sync(0xffffffff, accum_220_7, offset);
          accum_220_8 += __shfl_down_sync(0xffffffff, accum_220_8, offset);
        }
        if (threadIdx.x == 0) {
          dx_o_2_0 += accum_220_0;
          dx_o_2_1 += accum_220_1;
          dx_o_2_2 += accum_220_2;
          dx_o_2_3 += accum_220_3;
          dx_o_2_4 += accum_220_4;
          dx_o_2_5 += accum_220_5;
          dx_o_2_6 += accum_220_6;
          dx_o_2_7 += accum_220_7;
          dx_o_2_8 += accum_220_8;
        }
        if (threadIdx.x == 0) {
          dx_2[((idx_batch)*dim_2 + idx_chan_in_2)*9 + 0] = dx_o_2_0;
          dx_2[((idx_batch)*dim_2 + idx_chan_in_2)*9 + 1] = dx_o_2_1;
          dx_2[((idx_batch)*dim_2 + idx_chan_in_2)*9 + 2] = dx_o_2_2;
          dx_2[((idx_batch)*dim_2 + idx_chan_in_2)*9 + 3] = dx_o_2_3;
          dx_2[((idx_batch)*dim_2 + idx_chan_in_2)*9 + 4] = dx_o_2_4;
          dx_2[((idx_batch)*dim_2 + idx_chan_in_2)*9 + 5] = dx_o_2_5;
          dx_2[((idx_batch)*dim_2 + idx_chan_in_2)*9 + 6] = dx_o_2_6;
          dx_2[((idx_batch)*dim_2 + idx_chan_in_2)*9 + 7] = dx_o_2_7;
          dx_2[((idx_batch)*dim_2 + idx_chan_in_2)*9 + 8] = dx_o_2_8;
        }
      }
    }
  }
}


void ant16_o0_backward(
    int batch, int dim_l, int dim_0, int dim_1, int dim_2,
    const float* dy_0, const float* P_000, const float* left_000, const float* P_110, const float* left_110, const float* P_220, const float* left_220,
    float* dx_0, float* dx_1, float* dx_2) {
  
  int p_0 = dim_l*dim_0;
  int sharedmemsz = 0;
  sharedmemsz += 13*p_0;
  dim3 gridsz = dim3(batch);
  dim3 blocksz = dim3(WARPSZ, dim_l);
  ant16_o0_backward_kern<<<gridsz, blocksz, sharedmemsz*sizeof(float)>>>(
      p_0, 
      batch, dim_l, dim_0, dim_1, dim_2,
      dy_0, P_000, left_000, P_110, left_110, P_220, left_220,
      dx_0, dx_1, dx_2);
  
}


__global__
void ant16_o0_backleft_kern(
    // <<<(batch), (WARPSZ, dim_l)>>>
    
    int batch, int dim_l, int dim_0, int dim_1, int dim_2,
    const float* x_0, const float* x_1, const float* x_2, const float* dy_0, const float* P_000, const float* P_110, const float* P_220,
    float* __restrict__ dleft_000, float* __restrict__ dleft_110, float* __restrict__ dleft_220) {
  extern __shared__ float s[];
  for (int idx_batch = blockIdx.x; idx_batch < batch; idx_batch += gridDim.x) {
    { // compute left derivative tensor products
      float accum_000_0 = 0.0;
      for (int idx_chan_in_000 = threadIdx.x; idx_chan_in_000 < dim_0; idx_chan_in_000 += blockDim.x) {
        for (int idx_chan_out_000 = 0; idx_chan_out_000 < dim_0; idx_chan_out_000 += 1) {
          float l_000_0 = x_0[((idx_batch)*dim_0 + idx_chan_in_000)*1 + 0]*dy_0[((idx_batch)*dim_0 + idx_chan_out_000)*1 + 0];
          float P_oi_000 = P_000[((idx_chan_out_000)*blockDim.y + threadIdx.y)*dim_0 + idx_chan_in_000];
          accum_000_0 += P_oi_000*l_000_0;
        }
      }
      // reduce across the warp so that first thread in warp will have the sum 
      for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
        accum_000_0 += __shfl_down_sync(0xffffffff, accum_000_0, offset);
      }
      if (threadIdx.x == 0) {
        dleft_000[((idx_batch)*dim_l + threadIdx.y)*1 + 0] = accum_000_0;
      }
      float accum_110_0 = 0.0;
      float accum_110_1 = 0.0;
      float accum_110_2 = 0.0;
      for (int idx_chan_in_110 = threadIdx.x; idx_chan_in_110 < dim_1; idx_chan_in_110 += blockDim.x) {
        for (int idx_chan_out_110 = 0; idx_chan_out_110 < dim_0; idx_chan_out_110 += 1) {
          float l_110_0 = x_1[((idx_batch)*dim_1 + idx_chan_in_110)*3 + 0]*dy_0[((idx_batch)*dim_0 + idx_chan_out_110)*1 + 0];
          float l_110_1 = x_1[((idx_batch)*dim_1 + idx_chan_in_110)*3 + 1]*dy_0[((idx_batch)*dim_0 + idx_chan_out_110)*1 + 0];
          float l_110_2 = x_1[((idx_batch)*dim_1 + idx_chan_in_110)*3 + 2]*dy_0[((idx_batch)*dim_0 + idx_chan_out_110)*1 + 0];
          float P_oi_110 = P_110[((idx_chan_out_110)*blockDim.y + threadIdx.y)*dim_1 + idx_chan_in_110];
          accum_110_0 += P_oi_110*l_110_0;
          accum_110_1 += P_oi_110*l_110_1;
          accum_110_2 += P_oi_110*l_110_2;
        }
      }
      // reduce across the warp so that first thread in warp will have the sum 
      for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
        accum_110_0 += __shfl_down_sync(0xffffffff, accum_110_0, offset);
        accum_110_1 += __shfl_down_sync(0xffffffff, accum_110_1, offset);
        accum_110_2 += __shfl_down_sync(0xffffffff, accum_110_2, offset);
      }
      if (threadIdx.x == 0) {
        dleft_110[((idx_batch)*dim_l + threadIdx.y)*3 + 0] = accum_110_0;
        dleft_110[((idx_batch)*dim_l + threadIdx.y)*3 + 1] = accum_110_1;
        dleft_110[((idx_batch)*dim_l + threadIdx.y)*3 + 2] = accum_110_2;
      }
      float accum_220_0 = 0.0;
      float accum_220_1 = 0.0;
      float accum_220_2 = 0.0;
      float accum_220_3 = 0.0;
      float accum_220_4 = 0.0;
      float accum_220_5 = 0.0;
      float accum_220_6 = 0.0;
      float accum_220_7 = 0.0;
      float accum_220_8 = 0.0;
      for (int idx_chan_in_220 = threadIdx.x; idx_chan_in_220 < dim_2; idx_chan_in_220 += blockDim.x) {
        for (int idx_chan_out_220 = 0; idx_chan_out_220 < dim_0; idx_chan_out_220 += 1) {
          float l_220_0 = x_2[((idx_batch)*dim_2 + idx_chan_in_220)*9 + 0]*dy_0[((idx_batch)*dim_0 + idx_chan_out_220)*1 + 0];
          float l_220_1 = x_2[((idx_batch)*dim_2 + idx_chan_in_220)*9 + 1]*dy_0[((idx_batch)*dim_0 + idx_chan_out_220)*1 + 0];
          float l_220_2 = x_2[((idx_batch)*dim_2 + idx_chan_in_220)*9 + 2]*dy_0[((idx_batch)*dim_0 + idx_chan_out_220)*1 + 0];
          float l_220_3 = x_2[((idx_batch)*dim_2 + idx_chan_in_220)*9 + 3]*dy_0[((idx_batch)*dim_0 + idx_chan_out_220)*1 + 0];
          float l_220_4 = x_2[((idx_batch)*dim_2 + idx_chan_in_220)*9 + 4]*dy_0[((idx_batch)*dim_0 + idx_chan_out_220)*1 + 0];
          float l_220_5 = x_2[((idx_batch)*dim_2 + idx_chan_in_220)*9 + 5]*dy_0[((idx_batch)*dim_0 + idx_chan_out_220)*1 + 0];
          float l_220_6 = x_2[((idx_batch)*dim_2 + idx_chan_in_220)*9 + 6]*dy_0[((idx_batch)*dim_0 + idx_chan_out_220)*1 + 0];
          float l_220_7 = x_2[((idx_batch)*dim_2 + idx_chan_in_220)*9 + 7]*dy_0[((idx_batch)*dim_0 + idx_chan_out_220)*1 + 0];
          float l_220_8 = x_2[((idx_batch)*dim_2 + idx_chan_in_220)*9 + 8]*dy_0[((idx_batch)*dim_0 + idx_chan_out_220)*1 + 0];
          float P_oi_220 = P_220[((idx_chan_out_220)*blockDim.y + threadIdx.y)*dim_2 + idx_chan_in_220];
          accum_220_0 += P_oi_220*l_220_0;
          accum_220_1 += P_oi_220*l_220_1;
          accum_220_2 += P_oi_220*l_220_2;
          accum_220_3 += P_oi_220*l_220_3;
          accum_220_4 += P_oi_220*l_220_4;
          accum_220_5 += P_oi_220*l_220_5;
          accum_220_6 += P_oi_220*l_220_6;
          accum_220_7 += P_oi_220*l_220_7;
          accum_220_8 += P_oi_220*l_220_8;
        }
      }
      // reduce across the warp so that first thread in warp will have the sum 
      for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
        accum_220_0 += __shfl_down_sync(0xffffffff, accum_220_0, offset);
        accum_220_1 += __shfl_down_sync(0xffffffff, accum_220_1, offset);
        accum_220_2 += __shfl_down_sync(0xffffffff, accum_220_2, offset);
        accum_220_3 += __shfl_down_sync(0xffffffff, accum_220_3, offset);
        accum_220_4 += __shfl_down_sync(0xffffffff, accum_220_4, offset);
        accum_220_5 += __shfl_down_sync(0xffffffff, accum_220_5, offset);
        accum_220_6 += __shfl_down_sync(0xffffffff, accum_220_6, offset);
        accum_220_7 += __shfl_down_sync(0xffffffff, accum_220_7, offset);
        accum_220_8 += __shfl_down_sync(0xffffffff, accum_220_8, offset);
      }
      if (threadIdx.x == 0) {
        dleft_220[((idx_batch)*dim_l + threadIdx.y)*9 + 0] = accum_220_0;
        dleft_220[((idx_batch)*dim_l + threadIdx.y)*9 + 1] = accum_220_1;
        dleft_220[((idx_batch)*dim_l + threadIdx.y)*9 + 2] = accum_220_2;
        dleft_220[((idx_batch)*dim_l + threadIdx.y)*9 + 3] = accum_220_3;
        dleft_220[((idx_batch)*dim_l + threadIdx.y)*9 + 4] = accum_220_4;
        dleft_220[((idx_batch)*dim_l + threadIdx.y)*9 + 5] = accum_220_5;
        dleft_220[((idx_batch)*dim_l + threadIdx.y)*9 + 6] = accum_220_6;
        dleft_220[((idx_batch)*dim_l + threadIdx.y)*9 + 7] = accum_220_7;
        dleft_220[((idx_batch)*dim_l + threadIdx.y)*9 + 8] = accum_220_8;
      }
    }
  }
}


void ant16_o0_backleft(
    int batch, int dim_l, int dim_0, int dim_1, int dim_2,
    const float* x_0, const float* x_1, const float* x_2, const float* dy_0, const float* P_000, const float* P_110, const float* P_220,
    float* dleft_000, float* dleft_110, float* dleft_220) {
  
  
  int sharedmemsz = 0;
  dim3 gridsz = dim3(batch);
  dim3 blocksz = dim3(WARPSZ, dim_l);
  ant16_o0_backleft_kern<<<gridsz, blocksz, sharedmemsz*sizeof(float)>>>(
      
      batch, dim_l, dim_0, dim_1, dim_2,
      x_0, x_1, x_2, dy_0, P_000, P_110, P_220,
      dleft_000, dleft_110, dleft_220);
  
}


__global__
void ant16_o0_wtsback_kern(
    // <<<(WARPSZ, WARPSZ), (WARPSZ, dim_l)>>>
    
    int batch, int dim_l, int dim_0, int dim_1, int dim_2,
    const float* x_0, const float* x_1, const float* x_2, const float* dy_0, const float* left_000, const float* left_110, const float* left_220,
    float* __restrict__ dP_000, float* __restrict__ dP_110, float* __restrict__ dP_220) {
  extern __shared__ float s[];
  for (int idx_chan_in_000 = blockIdx.x; idx_chan_in_000 < dim_0; idx_chan_in_000 += gridDim.x) {
    for (int idx_chan_out_000 = blockIdx.y; idx_chan_out_000 < dim_0; idx_chan_out_000 += gridDim.y) {
      float dP_oi = 0.0;
      for (int idx_batch = threadIdx.x; idx_batch < batch; idx_batch += blockDim.x) {
        dP_oi += (left_000[((idx_batch)*dim_l + threadIdx.y)*1 + 0]*x_0[((idx_batch)*dim_0 + idx_chan_in_000)*1 + 0]*dy_0[((idx_batch)*dim_0 + idx_chan_out_000)*1 + 0]);
      }
      // reduce across the warp so that first thread in warp will have the sum 
      for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
        dP_oi += __shfl_down_sync(0xffffffff, dP_oi, offset);
      }
      if (threadIdx.x == 0) {
        dP_000[((idx_chan_out_000)*blockDim.y + threadIdx.y)*dim_0 + idx_chan_in_000] = dP_oi;
      }
    }
  }
  for (int idx_chan_in_110 = blockIdx.x; idx_chan_in_110 < dim_1; idx_chan_in_110 += gridDim.x) {
    for (int idx_chan_out_110 = blockIdx.y; idx_chan_out_110 < dim_0; idx_chan_out_110 += gridDim.y) {
      float dP_oi = 0.0;
      for (int idx_batch = threadIdx.x; idx_batch < batch; idx_batch += blockDim.x) {
        dP_oi += (left_110[((idx_batch)*dim_l + threadIdx.y)*3 + 0]*x_1[((idx_batch)*dim_1 + idx_chan_in_110)*3 + 0]*dy_0[((idx_batch)*dim_0 + idx_chan_out_110)*1 + 0] + left_110[((idx_batch)*dim_l + threadIdx.y)*3 + 1]*x_1[((idx_batch)*dim_1 + idx_chan_in_110)*3 + 1]*dy_0[((idx_batch)*dim_0 + idx_chan_out_110)*1 + 0] + left_110[((idx_batch)*dim_l + threadIdx.y)*3 + 2]*x_1[((idx_batch)*dim_1 + idx_chan_in_110)*3 + 2]*dy_0[((idx_batch)*dim_0 + idx_chan_out_110)*1 + 0]);
      }
      // reduce across the warp so that first thread in warp will have the sum 
      for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
        dP_oi += __shfl_down_sync(0xffffffff, dP_oi, offset);
      }
      if (threadIdx.x == 0) {
        dP_110[((idx_chan_out_110)*blockDim.y + threadIdx.y)*dim_1 + idx_chan_in_110] = dP_oi;
      }
    }
  }
  for (int idx_chan_in_220 = blockIdx.x; idx_chan_in_220 < dim_2; idx_chan_in_220 += gridDim.x) {
    for (int idx_chan_out_220 = blockIdx.y; idx_chan_out_220 < dim_0; idx_chan_out_220 += gridDim.y) {
      float dP_oi = 0.0;
      for (int idx_batch = threadIdx.x; idx_batch < batch; idx_batch += blockDim.x) {
        dP_oi += (left_220[((idx_batch)*dim_l + threadIdx.y)*9 + 0]*x_2[((idx_batch)*dim_2 + idx_chan_in_220)*9 + 0]*dy_0[((idx_batch)*dim_0 + idx_chan_out_220)*1 + 0] + left_220[((idx_batch)*dim_l + threadIdx.y)*9 + 1]*x_2[((idx_batch)*dim_2 + idx_chan_in_220)*9 + 1]*dy_0[((idx_batch)*dim_0 + idx_chan_out_220)*1 + 0] + left_220[((idx_batch)*dim_l + threadIdx.y)*9 + 2]*x_2[((idx_batch)*dim_2 + idx_chan_in_220)*9 + 2]*dy_0[((idx_batch)*dim_0 + idx_chan_out_220)*1 + 0] + left_220[((idx_batch)*dim_l + threadIdx.y)*9 + 3]*x_2[((idx_batch)*dim_2 + idx_chan_in_220)*9 + 3]*dy_0[((idx_batch)*dim_0 + idx_chan_out_220)*1 + 0] + left_220[((idx_batch)*dim_l + threadIdx.y)*9 + 4]*x_2[((idx_batch)*dim_2 + idx_chan_in_220)*9 + 4]*dy_0[((idx_batch)*dim_0 + idx_chan_out_220)*1 + 0] + left_220[((idx_batch)*dim_l + threadIdx.y)*9 + 5]*x_2[((idx_batch)*dim_2 + idx_chan_in_220)*9 + 5]*dy_0[((idx_batch)*dim_0 + idx_chan_out_220)*1 + 0] + left_220[((idx_batch)*dim_l + threadIdx.y)*9 + 6]*x_2[((idx_batch)*dim_2 + idx_chan_in_220)*9 + 6]*dy_0[((idx_batch)*dim_0 + idx_chan_out_220)*1 + 0] + left_220[((idx_batch)*dim_l + threadIdx.y)*9 + 7]*x_2[((idx_batch)*dim_2 + idx_chan_in_220)*9 + 7]*dy_0[((idx_batch)*dim_0 + idx_chan_out_220)*1 + 0] + left_220[((idx_batch)*dim_l + threadIdx.y)*9 + 8]*x_2[((idx_batch)*dim_2 + idx_chan_in_220)*9 + 8]*dy_0[((idx_batch)*dim_0 + idx_chan_out_220)*1 + 0]);
      }
      // reduce across the warp so that first thread in warp will have the sum 
      for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
        dP_oi += __shfl_down_sync(0xffffffff, dP_oi, offset);
      }
      if (threadIdx.x == 0) {
        dP_220[((idx_chan_out_220)*blockDim.y + threadIdx.y)*dim_2 + idx_chan_in_220] = dP_oi;
      }
    }
  }
}


void ant16_o0_wtsback(
    int batch, int dim_l, int dim_0, int dim_1, int dim_2,
    const float* x_0, const float* x_1, const float* x_2, const float* dy_0, const float* left_000, const float* left_110, const float* left_220,
    float* dP_000, float* dP_110, float* dP_220) {
  
  
  int sharedmemsz = 0;
  dim3 gridsz = dim3(WARPSZ, WARPSZ);
  dim3 blocksz = dim3(WARPSZ, dim_l);
  ant16_o0_wtsback_kern<<<gridsz, blocksz, sharedmemsz*sizeof(float)>>>(
      
      batch, dim_l, dim_0, dim_1, dim_2,
      x_0, x_1, x_2, dy_0, left_000, left_110, left_220,
      dP_000, dP_110, dP_220);
  
}


__global__
void ant16_o1_kern(
    // <<<(batch), (WARPSZ, dim_l)>>>
    int p_1, int p_0_base, int p_0, int p_2_base, int p_2, 
    int batch, int dim_l, int dim_0, int dim_1, int dim_2,
    const float* x_0, const float* x_1, const float* x_2, const float* P_011, const float* left_011, const float* P_101, const float* left_101, const float* P_121, const float* left_121, const float* P_211, const float* left_211,
    float* __restrict__ y_1) {
  extern __shared__ float s[];
  float* product_011 = &s[0*p_1]; // size = 3*p_1
  float* product_101 = &s[p_0_base + 0*p_0]; // size = 3*p_0
  float* product_121 = &s[p_2_base + 0*p_2]; // size = 3*p_2
  float* product_211 = &s[3*p_1]; // size = 3*p_1
  for (int idx_batch = blockIdx.x; idx_batch < batch; idx_batch += gridDim.x) {
    { // compute tensor products
      float l_011_0 = left_011[((idx_batch)*dim_l + threadIdx.y)*1 + 0];
      for (int idx_chan_in_011 = threadIdx.x; idx_chan_in_011 < dim_1; idx_chan_in_011 += blockDim.x) {
        product_011[((threadIdx.y)*dim_1 + idx_chan_in_011)*3 + 0] = l_011_0*x_1[((idx_batch)*dim_1 + idx_chan_in_011)*3 + 0];
        product_011[((threadIdx.y)*dim_1 + idx_chan_in_011)*3 + 1] = l_011_0*x_1[((idx_batch)*dim_1 + idx_chan_in_011)*3 + 1];
        product_011[((threadIdx.y)*dim_1 + idx_chan_in_011)*3 + 2] = l_011_0*x_1[((idx_batch)*dim_1 + idx_chan_in_011)*3 + 2];
      }
      float l_101_0 = left_101[((idx_batch)*dim_l + threadIdx.y)*3 + 0];
      float l_101_1 = left_101[((idx_batch)*dim_l + threadIdx.y)*3 + 1];
      float l_101_2 = left_101[((idx_batch)*dim_l + threadIdx.y)*3 + 2];
      for (int idx_chan_in_101 = threadIdx.x; idx_chan_in_101 < dim_0; idx_chan_in_101 += blockDim.x) {
        product_101[((threadIdx.y)*dim_0 + idx_chan_in_101)*3 + 0] = l_101_0*x_0[((idx_batch)*dim_0 + idx_chan_in_101)*1 + 0];
        product_101[((threadIdx.y)*dim_0 + idx_chan_in_101)*3 + 1] = l_101_1*x_0[((idx_batch)*dim_0 + idx_chan_in_101)*1 + 0];
        product_101[((threadIdx.y)*dim_0 + idx_chan_in_101)*3 + 2] = l_101_2*x_0[((idx_batch)*dim_0 + idx_chan_in_101)*1 + 0];
      }
      float l_121_0 = left_121[((idx_batch)*dim_l + threadIdx.y)*3 + 0];
      float l_121_1 = left_121[((idx_batch)*dim_l + threadIdx.y)*3 + 1];
      float l_121_2 = left_121[((idx_batch)*dim_l + threadIdx.y)*3 + 2];
      for (int idx_chan_in_121 = threadIdx.x; idx_chan_in_121 < dim_2; idx_chan_in_121 += blockDim.x) {
        product_121[((threadIdx.y)*dim_2 + idx_chan_in_121)*3 + 0] = l_121_0*x_2[((idx_batch)*dim_2 + idx_chan_in_121)*9 + 0] + l_121_1*x_2[((idx_batch)*dim_2 + idx_chan_in_121)*9 + 1] + l_121_2*x_2[((idx_batch)*dim_2 + idx_chan_in_121)*9 + 2];
        product_121[((threadIdx.y)*dim_2 + idx_chan_in_121)*3 + 1] = l_121_0*x_2[((idx_batch)*dim_2 + idx_chan_in_121)*9 + 3] + l_121_1*x_2[((idx_batch)*dim_2 + idx_chan_in_121)*9 + 4] + l_121_2*x_2[((idx_batch)*dim_2 + idx_chan_in_121)*9 + 5];
        product_121[((threadIdx.y)*dim_2 + idx_chan_in_121)*3 + 2] = l_121_0*x_2[((idx_batch)*dim_2 + idx_chan_in_121)*9 + 6] + l_121_1*x_2[((idx_batch)*dim_2 + idx_chan_in_121)*9 + 7] + l_121_2*x_2[((idx_batch)*dim_2 + idx_chan_in_121)*9 + 8];
      }
      float l_211_0 = left_211[((idx_batch)*dim_l + threadIdx.y)*9 + 0];
      float l_211_1 = left_211[((idx_batch)*dim_l + threadIdx.y)*9 + 1];
      float l_211_2 = left_211[((idx_batch)*dim_l + threadIdx.y)*9 + 2];
      float l_211_3 = left_211[((idx_batch)*dim_l + threadIdx.y)*9 + 3];
      float l_211_4 = left_211[((idx_batch)*dim_l + threadIdx.y)*9 + 4];
      float l_211_5 = left_211[((idx_batch)*dim_l + threadIdx.y)*9 + 5];
      float l_211_6 = left_211[((idx_batch)*dim_l + threadIdx.y)*9 + 6];
      float l_211_7 = left_211[((idx_batch)*dim_l + threadIdx.y)*9 + 7];
      float l_211_8 = left_211[((idx_batch)*dim_l + threadIdx.y)*9 + 8];
      for (int idx_chan_in_211 = threadIdx.x; idx_chan_in_211 < dim_1; idx_chan_in_211 += blockDim.x) {
        product_211[((threadIdx.y)*dim_1 + idx_chan_in_211)*3 + 0] = l_211_0*x_1[((idx_batch)*dim_1 + idx_chan_in_211)*3 + 0] + l_211_1*x_1[((idx_batch)*dim_1 + idx_chan_in_211)*3 + 1] + l_211_2*x_1[((idx_batch)*dim_1 + idx_chan_in_211)*3 + 2];
        product_211[((threadIdx.y)*dim_1 + idx_chan_in_211)*3 + 1] = l_211_3*x_1[((idx_batch)*dim_1 + idx_chan_in_211)*3 + 0] + l_211_4*x_1[((idx_batch)*dim_1 + idx_chan_in_211)*3 + 1] + l_211_5*x_1[((idx_batch)*dim_1 + idx_chan_in_211)*3 + 2];
        product_211[((threadIdx.y)*dim_1 + idx_chan_in_211)*3 + 2] = l_211_6*x_1[((idx_batch)*dim_1 + idx_chan_in_211)*3 + 0] + l_211_7*x_1[((idx_batch)*dim_1 + idx_chan_in_211)*3 + 1] + l_211_8*x_1[((idx_batch)*dim_1 + idx_chan_in_211)*3 + 2];
      }
    }
    __syncthreads();
    { // linear transforms to compute the outputs
      for (int idx_chan_out_1 = threadIdx.y; idx_chan_out_1 < dim_1; idx_chan_out_1 += blockDim.y) {
        float y_o_1_0 = 0.0;
        float y_o_1_1 = 0.0;
        float y_o_1_2 = 0.0;
        float accum_011_0 = 0.0;
        float accum_011_1 = 0.0;
        float accum_011_2 = 0.0;
        for (int idx_chan_in_011 = threadIdx.x; idx_chan_in_011 < dim_l*dim_1; idx_chan_in_011 += blockDim.x) {
          float P_oi_011 = P_011[(idx_chan_out_1)*dim_l*dim_1 + idx_chan_in_011];
          accum_011_0 += P_oi_011*product_011[(idx_chan_in_011)*3 + 0];
          accum_011_1 += P_oi_011*product_011[(idx_chan_in_011)*3 + 1];
          accum_011_2 += P_oi_011*product_011[(idx_chan_in_011)*3 + 2];
        }
        // reduce across the warp so that first thread in warp will have the sum 
        for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
          accum_011_0 += __shfl_down_sync(0xffffffff, accum_011_0, offset);
          accum_011_1 += __shfl_down_sync(0xffffffff, accum_011_1, offset);
          accum_011_2 += __shfl_down_sync(0xffffffff, accum_011_2, offset);
        }
        if (threadIdx.x == 0) {
          y_o_1_0 += accum_011_0;
          y_o_1_1 += accum_011_1;
          y_o_1_2 += accum_011_2;
        }
        float accum_101_0 = 0.0;
        float accum_101_1 = 0.0;
        float accum_101_2 = 0.0;
        for (int idx_chan_in_101 = threadIdx.x; idx_chan_in_101 < dim_l*dim_0; idx_chan_in_101 += blockDim.x) {
          float P_oi_101 = P_101[(idx_chan_out_1)*dim_l*dim_0 + idx_chan_in_101];
          accum_101_0 += P_oi_101*product_101[(idx_chan_in_101)*3 + 0];
          accum_101_1 += P_oi_101*product_101[(idx_chan_in_101)*3 + 1];
          accum_101_2 += P_oi_101*product_101[(idx_chan_in_101)*3 + 2];
        }
        // reduce across the warp so that first thread in warp will have the sum 
        for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
          accum_101_0 += __shfl_down_sync(0xffffffff, accum_101_0, offset);
          accum_101_1 += __shfl_down_sync(0xffffffff, accum_101_1, offset);
          accum_101_2 += __shfl_down_sync(0xffffffff, accum_101_2, offset);
        }
        if (threadIdx.x == 0) {
          y_o_1_0 += accum_101_0;
          y_o_1_1 += accum_101_1;
          y_o_1_2 += accum_101_2;
        }
        float accum_121_0 = 0.0;
        float accum_121_1 = 0.0;
        float accum_121_2 = 0.0;
        for (int idx_chan_in_121 = threadIdx.x; idx_chan_in_121 < dim_l*dim_2; idx_chan_in_121 += blockDim.x) {
          float P_oi_121 = P_121[(idx_chan_out_1)*dim_l*dim_2 + idx_chan_in_121];
          accum_121_0 += P_oi_121*product_121[(idx_chan_in_121)*3 + 0];
          accum_121_1 += P_oi_121*product_121[(idx_chan_in_121)*3 + 1];
          accum_121_2 += P_oi_121*product_121[(idx_chan_in_121)*3 + 2];
        }
        // reduce across the warp so that first thread in warp will have the sum 
        for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
          accum_121_0 += __shfl_down_sync(0xffffffff, accum_121_0, offset);
          accum_121_1 += __shfl_down_sync(0xffffffff, accum_121_1, offset);
          accum_121_2 += __shfl_down_sync(0xffffffff, accum_121_2, offset);
        }
        if (threadIdx.x == 0) {
          y_o_1_0 += accum_121_0;
          y_o_1_1 += accum_121_1;
          y_o_1_2 += accum_121_2;
        }
        float accum_211_0 = 0.0;
        float accum_211_1 = 0.0;
        float accum_211_2 = 0.0;
        for (int idx_chan_in_211 = threadIdx.x; idx_chan_in_211 < dim_l*dim_1; idx_chan_in_211 += blockDim.x) {
          float P_oi_211 = P_211[(idx_chan_out_1)*dim_l*dim_1 + idx_chan_in_211];
          accum_211_0 += P_oi_211*product_211[(idx_chan_in_211)*3 + 0];
          accum_211_1 += P_oi_211*product_211[(idx_chan_in_211)*3 + 1];
          accum_211_2 += P_oi_211*product_211[(idx_chan_in_211)*3 + 2];
        }
        // reduce across the warp so that first thread in warp will have the sum 
        for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
          accum_211_0 += __shfl_down_sync(0xffffffff, accum_211_0, offset);
          accum_211_1 += __shfl_down_sync(0xffffffff, accum_211_1, offset);
          accum_211_2 += __shfl_down_sync(0xffffffff, accum_211_2, offset);
        }
        if (threadIdx.x == 0) {
          y_o_1_0 += accum_211_0;
          y_o_1_1 += accum_211_1;
          y_o_1_2 += accum_211_2;
        }
        if (threadIdx.x == 0) {
          y_1[((idx_batch)*dim_1 + idx_chan_out_1)*3 + 0] = y_o_1_0;
          y_1[((idx_batch)*dim_1 + idx_chan_out_1)*3 + 1] = y_o_1_1;
          y_1[((idx_batch)*dim_1 + idx_chan_out_1)*3 + 2] = y_o_1_2;
        }
      }
    }
  }
}


void ant16_o1(
    int batch, int dim_l, int dim_0, int dim_1, int dim_2,
    const float* x_0, const float* x_1, const float* x_2, const float* P_011, const float* left_011, const float* P_101, const float* left_101, const float* P_121, const float* left_121, const float* P_211, const float* left_211,
    float* y_1) {
  
  int p_1 = dim_l*dim_1;
  int p_0 = dim_l*dim_0;
  int p_2 = dim_l*dim_2;
  int sharedmemsz = 0;
  sharedmemsz += 6*p_1;
  int p_0_base = sharedmemsz;
  sharedmemsz += 3*p_0;
  int p_2_base = sharedmemsz;
  sharedmemsz += 3*p_2;
  dim3 gridsz = dim3(batch);
  dim3 blocksz = dim3(WARPSZ, dim_l);
  ant16_o1_kern<<<gridsz, blocksz, sharedmemsz*sizeof(float)>>>(
      p_1, p_0_base, p_0, p_2_base, p_2, 
      batch, dim_l, dim_0, dim_1, dim_2,
      x_0, x_1, x_2, P_011, left_011, P_101, left_101, P_121, left_121, P_211, left_211,
      y_1);
  
}


__global__
void ant16_o1_backward_kern(
    // <<<(batch), (WARPSZ, dim_l)>>>
    int p_1, 
    int batch, int dim_l, int dim_0, int dim_1, int dim_2,
    const float* dy_1, const float* P_011, const float* left_011, const float* P_101, const float* left_101, const float* P_121, const float* left_121, const float* P_211, const float* left_211,
    float* __restrict__ dx_0, float* __restrict__ dx_1, float* __restrict__ dx_2) {
  extern __shared__ float s[];
  float* dproduct_011 = &s[0*p_1]; // size = 3*p_1
  float* dproduct_101 = &s[3*p_1]; // size = 1*p_1
  float* dproduct_121 = &s[4*p_1]; // size = 9*p_1
  float* dproduct_211 = &s[13*p_1]; // size = 3*p_1
  for (int idx_batch = blockIdx.x; idx_batch < batch; idx_batch += gridDim.x) {
    { // compute tensor products
      float l_011_0 = left_011[((idx_batch)*dim_l + threadIdx.y)*1 + 0];
      for (int idx_chan_out_011 = threadIdx.x; idx_chan_out_011 < dim_1; idx_chan_out_011 += blockDim.x) {
        dproduct_011[((threadIdx.y)*dim_1 + idx_chan_out_011)*3 + 0] = l_011_0*dy_1[((idx_batch)*dim_1 + idx_chan_out_011)*3 + 0];
        dproduct_011[((threadIdx.y)*dim_1 + idx_chan_out_011)*3 + 1] = l_011_0*dy_1[((idx_batch)*dim_1 + idx_chan_out_011)*3 + 1];
        dproduct_011[((threadIdx.y)*dim_1 + idx_chan_out_011)*3 + 2] = l_011_0*dy_1[((idx_batch)*dim_1 + idx_chan_out_011)*3 + 2];
      }
      float l_101_0 = left_101[((idx_batch)*dim_l + threadIdx.y)*3 + 0];
      float l_101_1 = left_101[((idx_batch)*dim_l + threadIdx.y)*3 + 1];
      float l_101_2 = left_101[((idx_batch)*dim_l + threadIdx.y)*3 + 2];
      for (int idx_chan_out_101 = threadIdx.x; idx_chan_out_101 < dim_1; idx_chan_out_101 += blockDim.x) {
        dproduct_101[((threadIdx.y)*dim_1 + idx_chan_out_101)*1 + 0] = l_101_0*dy_1[((idx_batch)*dim_1 + idx_chan_out_101)*3 + 0] + l_101_1*dy_1[((idx_batch)*dim_1 + idx_chan_out_101)*3 + 1] + l_101_2*dy_1[((idx_batch)*dim_1 + idx_chan_out_101)*3 + 2];
      }
      float l_121_0 = left_121[((idx_batch)*dim_l + threadIdx.y)*3 + 0];
      float l_121_1 = left_121[((idx_batch)*dim_l + threadIdx.y)*3 + 1];
      float l_121_2 = left_121[((idx_batch)*dim_l + threadIdx.y)*3 + 2];
      for (int idx_chan_out_121 = threadIdx.x; idx_chan_out_121 < dim_1; idx_chan_out_121 += blockDim.x) {
        dproduct_121[((threadIdx.y)*dim_1 + idx_chan_out_121)*9 + 0] = l_121_0*dy_1[((idx_batch)*dim_1 + idx_chan_out_121)*3 + 0];
        dproduct_121[((threadIdx.y)*dim_1 + idx_chan_out_121)*9 + 1] = l_121_1*dy_1[((idx_batch)*dim_1 + idx_chan_out_121)*3 + 0];
        dproduct_121[((threadIdx.y)*dim_1 + idx_chan_out_121)*9 + 2] = l_121_2*dy_1[((idx_batch)*dim_1 + idx_chan_out_121)*3 + 0];
        dproduct_121[((threadIdx.y)*dim_1 + idx_chan_out_121)*9 + 3] = l_121_0*dy_1[((idx_batch)*dim_1 + idx_chan_out_121)*3 + 1];
        dproduct_121[((threadIdx.y)*dim_1 + idx_chan_out_121)*9 + 4] = l_121_1*dy_1[((idx_batch)*dim_1 + idx_chan_out_121)*3 + 1];
        dproduct_121[((threadIdx.y)*dim_1 + idx_chan_out_121)*9 + 5] = l_121_2*dy_1[((idx_batch)*dim_1 + idx_chan_out_121)*3 + 1];
        dproduct_121[((threadIdx.y)*dim_1 + idx_chan_out_121)*9 + 6] = l_121_0*dy_1[((idx_batch)*dim_1 + idx_chan_out_121)*3 + 2];
        dproduct_121[((threadIdx.y)*dim_1 + idx_chan_out_121)*9 + 7] = l_121_1*dy_1[((idx_batch)*dim_1 + idx_chan_out_121)*3 + 2];
        dproduct_121[((threadIdx.y)*dim_1 + idx_chan_out_121)*9 + 8] = l_121_2*dy_1[((idx_batch)*dim_1 + idx_chan_out_121)*3 + 2];
      }
      float l_211_0 = left_211[((idx_batch)*dim_l + threadIdx.y)*9 + 0];
      float l_211_1 = left_211[((idx_batch)*dim_l + threadIdx.y)*9 + 1];
      float l_211_2 = left_211[((idx_batch)*dim_l + threadIdx.y)*9 + 2];
      float l_211_3 = left_211[((idx_batch)*dim_l + threadIdx.y)*9 + 3];
      float l_211_4 = left_211[((idx_batch)*dim_l + threadIdx.y)*9 + 4];
      float l_211_5 = left_211[((idx_batch)*dim_l + threadIdx.y)*9 + 5];
      float l_211_6 = left_211[((idx_batch)*dim_l + threadIdx.y)*9 + 6];
      float l_211_7 = left_211[((idx_batch)*dim_l + threadIdx.y)*9 + 7];
      float l_211_8 = left_211[((idx_batch)*dim_l + threadIdx.y)*9 + 8];
      for (int idx_chan_out_211 = threadIdx.x; idx_chan_out_211 < dim_1; idx_chan_out_211 += blockDim.x) {
        dproduct_211[((threadIdx.y)*dim_1 + idx_chan_out_211)*3 + 0] = l_211_0*dy_1[((idx_batch)*dim_1 + idx_chan_out_211)*3 + 0] + l_211_3*dy_1[((idx_batch)*dim_1 + idx_chan_out_211)*3 + 1] + l_211_6*dy_1[((idx_batch)*dim_1 + idx_chan_out_211)*3 + 2];
        dproduct_211[((threadIdx.y)*dim_1 + idx_chan_out_211)*3 + 1] = l_211_1*dy_1[((idx_batch)*dim_1 + idx_chan_out_211)*3 + 0] + l_211_4*dy_1[((idx_batch)*dim_1 + idx_chan_out_211)*3 + 1] + l_211_7*dy_1[((idx_batch)*dim_1 + idx_chan_out_211)*3 + 2];
        dproduct_211[((threadIdx.y)*dim_1 + idx_chan_out_211)*3 + 2] = l_211_2*dy_1[((idx_batch)*dim_1 + idx_chan_out_211)*3 + 0] + l_211_5*dy_1[((idx_batch)*dim_1 + idx_chan_out_211)*3 + 1] + l_211_8*dy_1[((idx_batch)*dim_1 + idx_chan_out_211)*3 + 2];
      }
    }
    __syncthreads();
    { // linear transforms to compute dx
      for (int idx_chan_in_0 = threadIdx.y; idx_chan_in_0 < dim_0; idx_chan_in_0 += blockDim.y) {
        float dx_o_0_0 = 0.0;
        float accum_101_0 = 0.0;
        for (int idx_l_101 = 0; idx_l_101 < dim_l; idx_l_101 += 1) {
          for (int idx_chan_out_101 = threadIdx.x; idx_chan_out_101 < dim_1; idx_chan_out_101 += blockDim.x) {
            float P_oi_101 = P_101[((idx_chan_out_101)*dim_l + idx_l_101)*dim_0 + idx_chan_in_0];
            accum_101_0 += P_oi_101*dproduct_101[((idx_l_101)*dim_1 + idx_chan_out_101)*1 + 0];
          }
        }
        // reduce across the warp so that first thread in warp will have the sum 
        for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
          accum_101_0 += __shfl_down_sync(0xffffffff, accum_101_0, offset);
        }
        if (threadIdx.x == 0) {
          dx_o_0_0 += accum_101_0;
        }
        if (threadIdx.x == 0) {
          dx_0[((idx_batch)*dim_0 + idx_chan_in_0)*1 + 0] = dx_o_0_0;
        }
      }
      for (int idx_chan_in_1 = threadIdx.y; idx_chan_in_1 < dim_1; idx_chan_in_1 += blockDim.y) {
        float dx_o_1_0 = 0.0;
        float dx_o_1_1 = 0.0;
        float dx_o_1_2 = 0.0;
        float accum_011_0 = 0.0;
        float accum_011_1 = 0.0;
        float accum_011_2 = 0.0;
        for (int idx_l_011 = 0; idx_l_011 < dim_l; idx_l_011 += 1) {
          for (int idx_chan_out_011 = threadIdx.x; idx_chan_out_011 < dim_1; idx_chan_out_011 += blockDim.x) {
            float P_oi_011 = P_011[((idx_chan_out_011)*dim_l + idx_l_011)*dim_1 + idx_chan_in_1];
            accum_011_0 += P_oi_011*dproduct_011[((idx_l_011)*dim_1 + idx_chan_out_011)*3 + 0];
            accum_011_1 += P_oi_011*dproduct_011[((idx_l_011)*dim_1 + idx_chan_out_011)*3 + 1];
            accum_011_2 += P_oi_011*dproduct_011[((idx_l_011)*dim_1 + idx_chan_out_011)*3 + 2];
          }
        }
        // reduce across the warp so that first thread in warp will have the sum 
        for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
          accum_011_0 += __shfl_down_sync(0xffffffff, accum_011_0, offset);
          accum_011_1 += __shfl_down_sync(0xffffffff, accum_011_1, offset);
          accum_011_2 += __shfl_down_sync(0xffffffff, accum_011_2, offset);
        }
        if (threadIdx.x == 0) {
          dx_o_1_0 += accum_011_0;
          dx_o_1_1 += accum_011_1;
          dx_o_1_2 += accum_011_2;
        }
        float accum_211_0 = 0.0;
        float accum_211_1 = 0.0;
        float accum_211_2 = 0.0;
        for (int idx_l_211 = 0; idx_l_211 < dim_l; idx_l_211 += 1) {
          for (int idx_chan_out_211 = threadIdx.x; idx_chan_out_211 < dim_1; idx_chan_out_211 += blockDim.x) {
            float P_oi_211 = P_211[((idx_chan_out_211)*dim_l + idx_l_211)*dim_1 + idx_chan_in_1];
            accum_211_0 += P_oi_211*dproduct_211[((idx_l_211)*dim_1 + idx_chan_out_211)*3 + 0];
            accum_211_1 += P_oi_211*dproduct_211[((idx_l_211)*dim_1 + idx_chan_out_211)*3 + 1];
            accum_211_2 += P_oi_211*dproduct_211[((idx_l_211)*dim_1 + idx_chan_out_211)*3 + 2];
          }
        }
        // reduce across the warp so that first thread in warp will have the sum 
        for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
          accum_211_0 += __shfl_down_sync(0xffffffff, accum_211_0, offset);
          accum_211_1 += __shfl_down_sync(0xffffffff, accum_211_1, offset);
          accum_211_2 += __shfl_down_sync(0xffffffff, accum_211_2, offset);
        }
        if (threadIdx.x == 0) {
          dx_o_1_0 += accum_211_0;
          dx_o_1_1 += accum_211_1;
          dx_o_1_2 += accum_211_2;
        }
        if (threadIdx.x == 0) {
          dx_1[((idx_batch)*dim_1 + idx_chan_in_1)*3 + 0] = dx_o_1_0;
          dx_1[((idx_batch)*dim_1 + idx_chan_in_1)*3 + 1] = dx_o_1_1;
          dx_1[((idx_batch)*dim_1 + idx_chan_in_1)*3 + 2] = dx_o_1_2;
        }
      }
      for (int idx_chan_in_2 = threadIdx.y; idx_chan_in_2 < dim_2; idx_chan_in_2 += blockDim.y) {
        float dx_o_2_0 = 0.0;
        float dx_o_2_1 = 0.0;
        float dx_o_2_2 = 0.0;
        float dx_o_2_3 = 0.0;
        float dx_o_2_4 = 0.0;
        float dx_o_2_5 = 0.0;
        float dx_o_2_6 = 0.0;
        float dx_o_2_7 = 0.0;
        float dx_o_2_8 = 0.0;
        float accum_121_0 = 0.0;
        float accum_121_1 = 0.0;
        float accum_121_2 = 0.0;
        float accum_121_3 = 0.0;
        float accum_121_4 = 0.0;
        float accum_121_5 = 0.0;
        float accum_121_6 = 0.0;
        float accum_121_7 = 0.0;
        float accum_121_8 = 0.0;
        for (int idx_l_121 = 0; idx_l_121 < dim_l; idx_l_121 += 1) {
          for (int idx_chan_out_121 = threadIdx.x; idx_chan_out_121 < dim_1; idx_chan_out_121 += blockDim.x) {
            float P_oi_121 = P_121[((idx_chan_out_121)*dim_l + idx_l_121)*dim_2 + idx_chan_in_2];
            accum_121_0 += P_oi_121*dproduct_121[((idx_l_121)*dim_1 + idx_chan_out_121)*9 + 0];
            accum_121_1 += P_oi_121*dproduct_121[((idx_l_121)*dim_1 + idx_chan_out_121)*9 + 1];
            accum_121_2 += P_oi_121*dproduct_121[((idx_l_121)*dim_1 + idx_chan_out_121)*9 + 2];
            accum_121_3 += P_oi_121*dproduct_121[((idx_l_121)*dim_1 + idx_chan_out_121)*9 + 3];
            accum_121_4 += P_oi_121*dproduct_121[((idx_l_121)*dim_1 + idx_chan_out_121)*9 + 4];
            accum_121_5 += P_oi_121*dproduct_121[((idx_l_121)*dim_1 + idx_chan_out_121)*9 + 5];
            accum_121_6 += P_oi_121*dproduct_121[((idx_l_121)*dim_1 + idx_chan_out_121)*9 + 6];
            accum_121_7 += P_oi_121*dproduct_121[((idx_l_121)*dim_1 + idx_chan_out_121)*9 + 7];
            accum_121_8 += P_oi_121*dproduct_121[((idx_l_121)*dim_1 + idx_chan_out_121)*9 + 8];
          }
        }
        // reduce across the warp so that first thread in warp will have the sum 
        for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
          accum_121_0 += __shfl_down_sync(0xffffffff, accum_121_0, offset);
          accum_121_1 += __shfl_down_sync(0xffffffff, accum_121_1, offset);
          accum_121_2 += __shfl_down_sync(0xffffffff, accum_121_2, offset);
          accum_121_3 += __shfl_down_sync(0xffffffff, accum_121_3, offset);
          accum_121_4 += __shfl_down_sync(0xffffffff, accum_121_4, offset);
          accum_121_5 += __shfl_down_sync(0xffffffff, accum_121_5, offset);
          accum_121_6 += __shfl_down_sync(0xffffffff, accum_121_6, offset);
          accum_121_7 += __shfl_down_sync(0xffffffff, accum_121_7, offset);
          accum_121_8 += __shfl_down_sync(0xffffffff, accum_121_8, offset);
        }
        if (threadIdx.x == 0) {
          dx_o_2_0 += accum_121_0;
          dx_o_2_1 += accum_121_1;
          dx_o_2_2 += accum_121_2;
          dx_o_2_3 += accum_121_3;
          dx_o_2_4 += accum_121_4;
          dx_o_2_5 += accum_121_5;
          dx_o_2_6 += accum_121_6;
          dx_o_2_7 += accum_121_7;
          dx_o_2_8 += accum_121_8;
        }
        if (threadIdx.x == 0) {
          dx_2[((idx_batch)*dim_2 + idx_chan_in_2)*9 + 0] = dx_o_2_0;
          dx_2[((idx_batch)*dim_2 + idx_chan_in_2)*9 + 1] = dx_o_2_1;
          dx_2[((idx_batch)*dim_2 + idx_chan_in_2)*9 + 2] = dx_o_2_2;
          dx_2[((idx_batch)*dim_2 + idx_chan_in_2)*9 + 3] = dx_o_2_3;
          dx_2[((idx_batch)*dim_2 + idx_chan_in_2)*9 + 4] = dx_o_2_4;
          dx_2[((idx_batch)*dim_2 + idx_chan_in_2)*9 + 5] = dx_o_2_5;
          dx_2[((idx_batch)*dim_2 + idx_chan_in_2)*9 + 6] = dx_o_2_6;
          dx_2[((idx_batch)*dim_2 + idx_chan_in_2)*9 + 7] = dx_o_2_7;
          dx_2[((idx_batch)*dim_2 + idx_chan_in_2)*9 + 8] = dx_o_2_8;
        }
      }
    }
  }
}


void ant16_o1_backward(
    int batch, int dim_l, int dim_0, int dim_1, int dim_2,
    const float* dy_1, const float* P_011, const float* left_011, const float* P_101, const float* left_101, const float* P_121, const float* left_121, const float* P_211, const float* left_211,
    float* dx_0, float* dx_1, float* dx_2) {
  
  int p_1 = dim_l*dim_1;
  int sharedmemsz = 0;
  sharedmemsz += 16*p_1;
  dim3 gridsz = dim3(batch);
  dim3 blocksz = dim3(WARPSZ, dim_l);
  ant16_o1_backward_kern<<<gridsz, blocksz, sharedmemsz*sizeof(float)>>>(
      p_1, 
      batch, dim_l, dim_0, dim_1, dim_2,
      dy_1, P_011, left_011, P_101, left_101, P_121, left_121, P_211, left_211,
      dx_0, dx_1, dx_2);
  
}


__global__
void ant16_o1_backleft_kern(
    // <<<(batch), (WARPSZ, dim_l)>>>
    
    int batch, int dim_l, int dim_0, int dim_1, int dim_2,
    const float* x_0, const float* x_1, const float* x_2, const float* dy_1, const float* P_011, const float* P_101, const float* P_121, const float* P_211,
    float* __restrict__ dleft_011, float* __restrict__ dleft_101, float* __restrict__ dleft_121, float* __restrict__ dleft_211) {
  extern __shared__ float s[];
  for (int idx_batch = blockIdx.x; idx_batch < batch; idx_batch += gridDim.x) {
    { // compute left derivative tensor products
      float accum_011_0 = 0.0;
      for (int idx_chan_in_011 = threadIdx.x; idx_chan_in_011 < dim_1; idx_chan_in_011 += blockDim.x) {
        for (int idx_chan_out_011 = 0; idx_chan_out_011 < dim_1; idx_chan_out_011 += 1) {
          float l_011_0 = x_1[((idx_batch)*dim_1 + idx_chan_in_011)*3 + 0]*dy_1[((idx_batch)*dim_1 + idx_chan_out_011)*3 + 0] + x_1[((idx_batch)*dim_1 + idx_chan_in_011)*3 + 1]*dy_1[((idx_batch)*dim_1 + idx_chan_out_011)*3 + 1] + x_1[((idx_batch)*dim_1 + idx_chan_in_011)*3 + 2]*dy_1[((idx_batch)*dim_1 + idx_chan_out_011)*3 + 2];
          float P_oi_011 = P_011[((idx_chan_out_011)*blockDim.y + threadIdx.y)*dim_1 + idx_chan_in_011];
          accum_011_0 += P_oi_011*l_011_0;
        }
      }
      // reduce across the warp so that first thread in warp will have the sum 
      for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
        accum_011_0 += __shfl_down_sync(0xffffffff, accum_011_0, offset);
      }
      if (threadIdx.x == 0) {
        dleft_011[((idx_batch)*dim_l + threadIdx.y)*1 + 0] = accum_011_0;
      }
      float accum_101_0 = 0.0;
      float accum_101_1 = 0.0;
      float accum_101_2 = 0.0;
      for (int idx_chan_in_101 = threadIdx.x; idx_chan_in_101 < dim_0; idx_chan_in_101 += blockDim.x) {
        for (int idx_chan_out_101 = 0; idx_chan_out_101 < dim_1; idx_chan_out_101 += 1) {
          float l_101_0 = x_0[((idx_batch)*dim_0 + idx_chan_in_101)*1 + 0]*dy_1[((idx_batch)*dim_1 + idx_chan_out_101)*3 + 0];
          float l_101_1 = x_0[((idx_batch)*dim_0 + idx_chan_in_101)*1 + 0]*dy_1[((idx_batch)*dim_1 + idx_chan_out_101)*3 + 1];
          float l_101_2 = x_0[((idx_batch)*dim_0 + idx_chan_in_101)*1 + 0]*dy_1[((idx_batch)*dim_1 + idx_chan_out_101)*3 + 2];
          float P_oi_101 = P_101[((idx_chan_out_101)*blockDim.y + threadIdx.y)*dim_0 + idx_chan_in_101];
          accum_101_0 += P_oi_101*l_101_0;
          accum_101_1 += P_oi_101*l_101_1;
          accum_101_2 += P_oi_101*l_101_2;
        }
      }
      // reduce across the warp so that first thread in warp will have the sum 
      for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
        accum_101_0 += __shfl_down_sync(0xffffffff, accum_101_0, offset);
        accum_101_1 += __shfl_down_sync(0xffffffff, accum_101_1, offset);
        accum_101_2 += __shfl_down_sync(0xffffffff, accum_101_2, offset);
      }
      if (threadIdx.x == 0) {
        dleft_101[((idx_batch)*dim_l + threadIdx.y)*3 + 0] = accum_101_0;
        dleft_101[((idx_batch)*dim_l + threadIdx.y)*3 + 1] = accum_101_1;
        dleft_101[((idx_batch)*dim_l + threadIdx.y)*3 + 2] = accum_101_2;
      }
      float accum_121_0 = 0.0;
      float accum_121_1 = 0.0;
      float accum_121_2 = 0.0;
      for (int idx_chan_in_121 = threadIdx.x; idx_chan_in_121 < dim_2; idx_chan_in_121 += blockDim.x) {
        for (int idx_chan_out_121 = 0; idx_chan_out_121 < dim_1; idx_chan_out_121 += 1) {
          float l_121_0 = x_2[((idx_batch)*dim_2 + idx_chan_in_121)*9 + 0]*dy_1[((idx_batch)*dim_1 + idx_chan_out_121)*3 + 0] + x_2[((idx_batch)*dim_2 + idx_chan_in_121)*9 + 3]*dy_1[((idx_batch)*dim_1 + idx_chan_out_121)*3 + 1] + x_2[((idx_batch)*dim_2 + idx_chan_in_121)*9 + 6]*dy_1[((idx_batch)*dim_1 + idx_chan_out_121)*3 + 2];
          float l_121_1 = x_2[((idx_batch)*dim_2 + idx_chan_in_121)*9 + 1]*dy_1[((idx_batch)*dim_1 + idx_chan_out_121)*3 + 0] + x_2[((idx_batch)*dim_2 + idx_chan_in_121)*9 + 4]*dy_1[((idx_batch)*dim_1 + idx_chan_out_121)*3 + 1] + x_2[((idx_batch)*dim_2 + idx_chan_in_121)*9 + 7]*dy_1[((idx_batch)*dim_1 + idx_chan_out_121)*3 + 2];
          float l_121_2 = x_2[((idx_batch)*dim_2 + idx_chan_in_121)*9 + 2]*dy_1[((idx_batch)*dim_1 + idx_chan_out_121)*3 + 0] + x_2[((idx_batch)*dim_2 + idx_chan_in_121)*9 + 5]*dy_1[((idx_batch)*dim_1 + idx_chan_out_121)*3 + 1] + x_2[((idx_batch)*dim_2 + idx_chan_in_121)*9 + 8]*dy_1[((idx_batch)*dim_1 + idx_chan_out_121)*3 + 2];
          float P_oi_121 = P_121[((idx_chan_out_121)*blockDim.y + threadIdx.y)*dim_2 + idx_chan_in_121];
          accum_121_0 += P_oi_121*l_121_0;
          accum_121_1 += P_oi_121*l_121_1;
          accum_121_2 += P_oi_121*l_121_2;
        }
      }
      // reduce across the warp so that first thread in warp will have the sum 
      for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
        accum_121_0 += __shfl_down_sync(0xffffffff, accum_121_0, offset);
        accum_121_1 += __shfl_down_sync(0xffffffff, accum_121_1, offset);
        accum_121_2 += __shfl_down_sync(0xffffffff, accum_121_2, offset);
      }
      if (threadIdx.x == 0) {
        dleft_121[((idx_batch)*dim_l + threadIdx.y)*3 + 0] = accum_121_0;
        dleft_121[((idx_batch)*dim_l + threadIdx.y)*3 + 1] = accum_121_1;
        dleft_121[((idx_batch)*dim_l + threadIdx.y)*3 + 2] = accum_121_2;
      }
      float accum_211_0 = 0.0;
      float accum_211_1 = 0.0;
      float accum_211_2 = 0.0;
      float accum_211_3 = 0.0;
      float accum_211_4 = 0.0;
      float accum_211_5 = 0.0;
      float accum_211_6 = 0.0;
      float accum_211_7 = 0.0;
      float accum_211_8 = 0.0;
      for (int idx_chan_in_211 = threadIdx.x; idx_chan_in_211 < dim_1; idx_chan_in_211 += blockDim.x) {
        for (int idx_chan_out_211 = 0; idx_chan_out_211 < dim_1; idx_chan_out_211 += 1) {
          float l_211_0 = x_1[((idx_batch)*dim_1 + idx_chan_in_211)*3 + 0]*dy_1[((idx_batch)*dim_1 + idx_chan_out_211)*3 + 0];
          float l_211_1 = x_1[((idx_batch)*dim_1 + idx_chan_in_211)*3 + 1]*dy_1[((idx_batch)*dim_1 + idx_chan_out_211)*3 + 0];
          float l_211_2 = x_1[((idx_batch)*dim_1 + idx_chan_in_211)*3 + 2]*dy_1[((idx_batch)*dim_1 + idx_chan_out_211)*3 + 0];
          float l_211_3 = x_1[((idx_batch)*dim_1 + idx_chan_in_211)*3 + 0]*dy_1[((idx_batch)*dim_1 + idx_chan_out_211)*3 + 1];
          float l_211_4 = x_1[((idx_batch)*dim_1 + idx_chan_in_211)*3 + 1]*dy_1[((idx_batch)*dim_1 + idx_chan_out_211)*3 + 1];
          float l_211_5 = x_1[((idx_batch)*dim_1 + idx_chan_in_211)*3 + 2]*dy_1[((idx_batch)*dim_1 + idx_chan_out_211)*3 + 1];
          float l_211_6 = x_1[((idx_batch)*dim_1 + idx_chan_in_211)*3 + 0]*dy_1[((idx_batch)*dim_1 + idx_chan_out_211)*3 + 2];
          float l_211_7 = x_1[((idx_batch)*dim_1 + idx_chan_in_211)*3 + 1]*dy_1[((idx_batch)*dim_1 + idx_chan_out_211)*3 + 2];
          float l_211_8 = x_1[((idx_batch)*dim_1 + idx_chan_in_211)*3 + 2]*dy_1[((idx_batch)*dim_1 + idx_chan_out_211)*3 + 2];
          float P_oi_211 = P_211[((idx_chan_out_211)*blockDim.y + threadIdx.y)*dim_1 + idx_chan_in_211];
          accum_211_0 += P_oi_211*l_211_0;
          accum_211_1 += P_oi_211*l_211_1;
          accum_211_2 += P_oi_211*l_211_2;
          accum_211_3 += P_oi_211*l_211_3;
          accum_211_4 += P_oi_211*l_211_4;
          accum_211_5 += P_oi_211*l_211_5;
          accum_211_6 += P_oi_211*l_211_6;
          accum_211_7 += P_oi_211*l_211_7;
          accum_211_8 += P_oi_211*l_211_8;
        }
      }
      // reduce across the warp so that first thread in warp will have the sum 
      for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
        accum_211_0 += __shfl_down_sync(0xffffffff, accum_211_0, offset);
        accum_211_1 += __shfl_down_sync(0xffffffff, accum_211_1, offset);
        accum_211_2 += __shfl_down_sync(0xffffffff, accum_211_2, offset);
        accum_211_3 += __shfl_down_sync(0xffffffff, accum_211_3, offset);
        accum_211_4 += __shfl_down_sync(0xffffffff, accum_211_4, offset);
        accum_211_5 += __shfl_down_sync(0xffffffff, accum_211_5, offset);
        accum_211_6 += __shfl_down_sync(0xffffffff, accum_211_6, offset);
        accum_211_7 += __shfl_down_sync(0xffffffff, accum_211_7, offset);
        accum_211_8 += __shfl_down_sync(0xffffffff, accum_211_8, offset);
      }
      if (threadIdx.x == 0) {
        dleft_211[((idx_batch)*dim_l + threadIdx.y)*9 + 0] = accum_211_0;
        dleft_211[((idx_batch)*dim_l + threadIdx.y)*9 + 1] = accum_211_1;
        dleft_211[((idx_batch)*dim_l + threadIdx.y)*9 + 2] = accum_211_2;
        dleft_211[((idx_batch)*dim_l + threadIdx.y)*9 + 3] = accum_211_3;
        dleft_211[((idx_batch)*dim_l + threadIdx.y)*9 + 4] = accum_211_4;
        dleft_211[((idx_batch)*dim_l + threadIdx.y)*9 + 5] = accum_211_5;
        dleft_211[((idx_batch)*dim_l + threadIdx.y)*9 + 6] = accum_211_6;
        dleft_211[((idx_batch)*dim_l + threadIdx.y)*9 + 7] = accum_211_7;
        dleft_211[((idx_batch)*dim_l + threadIdx.y)*9 + 8] = accum_211_8;
      }
    }
  }
}


void ant16_o1_backleft(
    int batch, int dim_l, int dim_0, int dim_1, int dim_2,
    const float* x_0, const float* x_1, const float* x_2, const float* dy_1, const float* P_011, const float* P_101, const float* P_121, const float* P_211,
    float* dleft_011, float* dleft_101, float* dleft_121, float* dleft_211) {
  
  
  int sharedmemsz = 0;
  dim3 gridsz = dim3(batch);
  dim3 blocksz = dim3(WARPSZ, dim_l);
  ant16_o1_backleft_kern<<<gridsz, blocksz, sharedmemsz*sizeof(float)>>>(
      
      batch, dim_l, dim_0, dim_1, dim_2,
      x_0, x_1, x_2, dy_1, P_011, P_101, P_121, P_211,
      dleft_011, dleft_101, dleft_121, dleft_211);
  
}


__global__
void ant16_o1_wtsback_kern(
    // <<<(WARPSZ, WARPSZ), (WARPSZ, dim_l)>>>
    
    int batch, int dim_l, int dim_0, int dim_1, int dim_2,
    const float* x_0, const float* x_1, const float* x_2, const float* dy_1, const float* left_011, const float* left_101, const float* left_121, const float* left_211,
    float* __restrict__ dP_011, float* __restrict__ dP_101, float* __restrict__ dP_121, float* __restrict__ dP_211) {
  extern __shared__ float s[];
  for (int idx_chan_in_011 = blockIdx.x; idx_chan_in_011 < dim_1; idx_chan_in_011 += gridDim.x) {
    for (int idx_chan_out_011 = blockIdx.y; idx_chan_out_011 < dim_1; idx_chan_out_011 += gridDim.y) {
      float dP_oi = 0.0;
      for (int idx_batch = threadIdx.x; idx_batch < batch; idx_batch += blockDim.x) {
        dP_oi += (left_011[((idx_batch)*dim_l + threadIdx.y)*1 + 0]*x_1[((idx_batch)*dim_1 + idx_chan_in_011)*3 + 0]*dy_1[((idx_batch)*dim_1 + idx_chan_out_011)*3 + 0] + left_011[((idx_batch)*dim_l + threadIdx.y)*1 + 0]*x_1[((idx_batch)*dim_1 + idx_chan_in_011)*3 + 1]*dy_1[((idx_batch)*dim_1 + idx_chan_out_011)*3 + 1] + left_011[((idx_batch)*dim_l + threadIdx.y)*1 + 0]*x_1[((idx_batch)*dim_1 + idx_chan_in_011)*3 + 2]*dy_1[((idx_batch)*dim_1 + idx_chan_out_011)*3 + 2]);
      }
      // reduce across the warp so that first thread in warp will have the sum 
      for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
        dP_oi += __shfl_down_sync(0xffffffff, dP_oi, offset);
      }
      if (threadIdx.x == 0) {
        dP_011[((idx_chan_out_011)*blockDim.y + threadIdx.y)*dim_1 + idx_chan_in_011] = dP_oi;
      }
    }
  }
  for (int idx_chan_in_101 = blockIdx.x; idx_chan_in_101 < dim_0; idx_chan_in_101 += gridDim.x) {
    for (int idx_chan_out_101 = blockIdx.y; idx_chan_out_101 < dim_1; idx_chan_out_101 += gridDim.y) {
      float dP_oi = 0.0;
      for (int idx_batch = threadIdx.x; idx_batch < batch; idx_batch += blockDim.x) {
        dP_oi += (left_101[((idx_batch)*dim_l + threadIdx.y)*3 + 0]*x_0[((idx_batch)*dim_0 + idx_chan_in_101)*1 + 0]*dy_1[((idx_batch)*dim_1 + idx_chan_out_101)*3 + 0] + left_101[((idx_batch)*dim_l + threadIdx.y)*3 + 1]*x_0[((idx_batch)*dim_0 + idx_chan_in_101)*1 + 0]*dy_1[((idx_batch)*dim_1 + idx_chan_out_101)*3 + 1] + left_101[((idx_batch)*dim_l + threadIdx.y)*3 + 2]*x_0[((idx_batch)*dim_0 + idx_chan_in_101)*1 + 0]*dy_1[((idx_batch)*dim_1 + idx_chan_out_101)*3 + 2]);
      }
      // reduce across the warp so that first thread in warp will have the sum 
      for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
        dP_oi += __shfl_down_sync(0xffffffff, dP_oi, offset);
      }
      if (threadIdx.x == 0) {
        dP_101[((idx_chan_out_101)*blockDim.y + threadIdx.y)*dim_0 + idx_chan_in_101] = dP_oi;
      }
    }
  }
  for (int idx_chan_in_121 = blockIdx.x; idx_chan_in_121 < dim_2; idx_chan_in_121 += gridDim.x) {
    for (int idx_chan_out_121 = blockIdx.y; idx_chan_out_121 < dim_1; idx_chan_out_121 += gridDim.y) {
      float dP_oi = 0.0;
      for (int idx_batch = threadIdx.x; idx_batch < batch; idx_batch += blockDim.x) {
        dP_oi += (left_121[((idx_batch)*dim_l + threadIdx.y)*3 + 0]*x_2[((idx_batch)*dim_2 + idx_chan_in_121)*9 + 0]*dy_1[((idx_batch)*dim_1 + idx_chan_out_121)*3 + 0] + left_121[((idx_batch)*dim_l + threadIdx.y)*3 + 1]*x_2[((idx_batch)*dim_2 + idx_chan_in_121)*9 + 1]*dy_1[((idx_batch)*dim_1 + idx_chan_out_121)*3 + 0] + left_121[((idx_batch)*dim_l + threadIdx.y)*3 + 2]*x_2[((idx_batch)*dim_2 + idx_chan_in_121)*9 + 2]*dy_1[((idx_batch)*dim_1 + idx_chan_out_121)*3 + 0] + left_121[((idx_batch)*dim_l + threadIdx.y)*3 + 0]*x_2[((idx_batch)*dim_2 + idx_chan_in_121)*9 + 3]*dy_1[((idx_batch)*dim_1 + idx_chan_out_121)*3 + 1] + left_121[((idx_batch)*dim_l + threadIdx.y)*3 + 1]*x_2[((idx_batch)*dim_2 + idx_chan_in_121)*9 + 4]*dy_1[((idx_batch)*dim_1 + idx_chan_out_121)*3 + 1] + left_121[((idx_batch)*dim_l + threadIdx.y)*3 + 2]*x_2[((idx_batch)*dim_2 + idx_chan_in_121)*9 + 5]*dy_1[((idx_batch)*dim_1 + idx_chan_out_121)*3 + 1] + left_121[((idx_batch)*dim_l + threadIdx.y)*3 + 0]*x_2[((idx_batch)*dim_2 + idx_chan_in_121)*9 + 6]*dy_1[((idx_batch)*dim_1 + idx_chan_out_121)*3 + 2] + left_121[((idx_batch)*dim_l + threadIdx.y)*3 + 1]*x_2[((idx_batch)*dim_2 + idx_chan_in_121)*9 + 7]*dy_1[((idx_batch)*dim_1 + idx_chan_out_121)*3 + 2] + left_121[((idx_batch)*dim_l + threadIdx.y)*3 + 2]*x_2[((idx_batch)*dim_2 + idx_chan_in_121)*9 + 8]*dy_1[((idx_batch)*dim_1 + idx_chan_out_121)*3 + 2]);
      }
      // reduce across the warp so that first thread in warp will have the sum 
      for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
        dP_oi += __shfl_down_sync(0xffffffff, dP_oi, offset);
      }
      if (threadIdx.x == 0) {
        dP_121[((idx_chan_out_121)*blockDim.y + threadIdx.y)*dim_2 + idx_chan_in_121] = dP_oi;
      }
    }
  }
  for (int idx_chan_in_211 = blockIdx.x; idx_chan_in_211 < dim_1; idx_chan_in_211 += gridDim.x) {
    for (int idx_chan_out_211 = blockIdx.y; idx_chan_out_211 < dim_1; idx_chan_out_211 += gridDim.y) {
      float dP_oi = 0.0;
      for (int idx_batch = threadIdx.x; idx_batch < batch; idx_batch += blockDim.x) {
        dP_oi += (left_211[((idx_batch)*dim_l + threadIdx.y)*9 + 0]*x_1[((idx_batch)*dim_1 + idx_chan_in_211)*3 + 0]*dy_1[((idx_batch)*dim_1 + idx_chan_out_211)*3 + 0] + left_211[((idx_batch)*dim_l + threadIdx.y)*9 + 1]*x_1[((idx_batch)*dim_1 + idx_chan_in_211)*3 + 1]*dy_1[((idx_batch)*dim_1 + idx_chan_out_211)*3 + 0] + left_211[((idx_batch)*dim_l + threadIdx.y)*9 + 2]*x_1[((idx_batch)*dim_1 + idx_chan_in_211)*3 + 2]*dy_1[((idx_batch)*dim_1 + idx_chan_out_211)*3 + 0] + left_211[((idx_batch)*dim_l + threadIdx.y)*9 + 3]*x_1[((idx_batch)*dim_1 + idx_chan_in_211)*3 + 0]*dy_1[((idx_batch)*dim_1 + idx_chan_out_211)*3 + 1] + left_211[((idx_batch)*dim_l + threadIdx.y)*9 + 4]*x_1[((idx_batch)*dim_1 + idx_chan_in_211)*3 + 1]*dy_1[((idx_batch)*dim_1 + idx_chan_out_211)*3 + 1] + left_211[((idx_batch)*dim_l + threadIdx.y)*9 + 5]*x_1[((idx_batch)*dim_1 + idx_chan_in_211)*3 + 2]*dy_1[((idx_batch)*dim_1 + idx_chan_out_211)*3 + 1] + left_211[((idx_batch)*dim_l + threadIdx.y)*9 + 6]*x_1[((idx_batch)*dim_1 + idx_chan_in_211)*3 + 0]*dy_1[((idx_batch)*dim_1 + idx_chan_out_211)*3 + 2] + left_211[((idx_batch)*dim_l + threadIdx.y)*9 + 7]*x_1[((idx_batch)*dim_1 + idx_chan_in_211)*3 + 1]*dy_1[((idx_batch)*dim_1 + idx_chan_out_211)*3 + 2] + left_211[((idx_batch)*dim_l + threadIdx.y)*9 + 8]*x_1[((idx_batch)*dim_1 + idx_chan_in_211)*3 + 2]*dy_1[((idx_batch)*dim_1 + idx_chan_out_211)*3 + 2]);
      }
      // reduce across the warp so that first thread in warp will have the sum 
      for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
        dP_oi += __shfl_down_sync(0xffffffff, dP_oi, offset);
      }
      if (threadIdx.x == 0) {
        dP_211[((idx_chan_out_211)*blockDim.y + threadIdx.y)*dim_1 + idx_chan_in_211] = dP_oi;
      }
    }
  }
}


void ant16_o1_wtsback(
    int batch, int dim_l, int dim_0, int dim_1, int dim_2,
    const float* x_0, const float* x_1, const float* x_2, const float* dy_1, const float* left_011, const float* left_101, const float* left_121, const float* left_211,
    float* dP_011, float* dP_101, float* dP_121, float* dP_211) {
  
  
  int sharedmemsz = 0;
  dim3 gridsz = dim3(WARPSZ, WARPSZ);
  dim3 blocksz = dim3(WARPSZ, dim_l);
  ant16_o1_wtsback_kern<<<gridsz, blocksz, sharedmemsz*sizeof(float)>>>(
      
      batch, dim_l, dim_0, dim_1, dim_2,
      x_0, x_1, x_2, dy_1, left_011, left_101, left_121, left_211,
      dP_011, dP_101, dP_121, dP_211);
  
}


__global__
void ant16_o2_kern(
    // <<<(batch), (WARPSZ, dim_l)>>>
    int p_2, int p_0_base, int p_0, int p_1_base, int p_1, 
    int batch, int dim_l, int dim_0, int dim_1, int dim_2,
    const float* x_0, const float* x_1, const float* x_2, const float* P_022, const float* left_022, const float* P_202, const float* left_202, const float* P_112, const float* left_112, const float* P_222, const float* left_222,
    float* __restrict__ y_2) {
  extern __shared__ float s[];
  float* product_022 = &s[0*p_2]; // size = 9*p_2
  float* product_202 = &s[p_0_base + 0*p_0]; // size = 9*p_0
  float* product_112 = &s[p_1_base + 0*p_1]; // size = 9*p_1
  float* product_222 = &s[9*p_2]; // size = 9*p_2
  for (int idx_batch = blockIdx.x; idx_batch < batch; idx_batch += gridDim.x) {
    { // compute tensor products
      float l_022_0 = left_022[((idx_batch)*dim_l + threadIdx.y)*1 + 0];
      for (int idx_chan_in_022 = threadIdx.x; idx_chan_in_022 < dim_2; idx_chan_in_022 += blockDim.x) {
        product_022[((threadIdx.y)*dim_2 + idx_chan_in_022)*9 + 0] = l_022_0*x_2[((idx_batch)*dim_2 + idx_chan_in_022)*9 + 0];
        product_022[((threadIdx.y)*dim_2 + idx_chan_in_022)*9 + 1] = l_022_0*x_2[((idx_batch)*dim_2 + idx_chan_in_022)*9 + 1];
        product_022[((threadIdx.y)*dim_2 + idx_chan_in_022)*9 + 2] = l_022_0*x_2[((idx_batch)*dim_2 + idx_chan_in_022)*9 + 2];
        product_022[((threadIdx.y)*dim_2 + idx_chan_in_022)*9 + 3] = l_022_0*x_2[((idx_batch)*dim_2 + idx_chan_in_022)*9 + 3];
        product_022[((threadIdx.y)*dim_2 + idx_chan_in_022)*9 + 4] = l_022_0*x_2[((idx_batch)*dim_2 + idx_chan_in_022)*9 + 4];
        product_022[((threadIdx.y)*dim_2 + idx_chan_in_022)*9 + 5] = l_022_0*x_2[((idx_batch)*dim_2 + idx_chan_in_022)*9 + 5];
        product_022[((threadIdx.y)*dim_2 + idx_chan_in_022)*9 + 6] = l_022_0*x_2[((idx_batch)*dim_2 + idx_chan_in_022)*9 + 6];
        product_022[((threadIdx.y)*dim_2 + idx_chan_in_022)*9 + 7] = l_022_0*x_2[((idx_batch)*dim_2 + idx_chan_in_022)*9 + 7];
        product_022[((threadIdx.y)*dim_2 + idx_chan_in_022)*9 + 8] = l_022_0*x_2[((idx_batch)*dim_2 + idx_chan_in_022)*9 + 8];
      }
      float l_202_0 = left_202[((idx_batch)*dim_l + threadIdx.y)*9 + 0];
      float l_202_1 = left_202[((idx_batch)*dim_l + threadIdx.y)*9 + 1];
      float l_202_2 = left_202[((idx_batch)*dim_l + threadIdx.y)*9 + 2];
      float l_202_3 = left_202[((idx_batch)*dim_l + threadIdx.y)*9 + 3];
      float l_202_4 = left_202[((idx_batch)*dim_l + threadIdx.y)*9 + 4];
      float l_202_5 = left_202[((idx_batch)*dim_l + threadIdx.y)*9 + 5];
      float l_202_6 = left_202[((idx_batch)*dim_l + threadIdx.y)*9 + 6];
      float l_202_7 = left_202[((idx_batch)*dim_l + threadIdx.y)*9 + 7];
      float l_202_8 = left_202[((idx_batch)*dim_l + threadIdx.y)*9 + 8];
      for (int idx_chan_in_202 = threadIdx.x; idx_chan_in_202 < dim_0; idx_chan_in_202 += blockDim.x) {
        product_202[((threadIdx.y)*dim_0 + idx_chan_in_202)*9 + 0] = l_202_0*x_0[((idx_batch)*dim_0 + idx_chan_in_202)*1 + 0];
        product_202[((threadIdx.y)*dim_0 + idx_chan_in_202)*9 + 1] = l_202_1*x_0[((idx_batch)*dim_0 + idx_chan_in_202)*1 + 0];
        product_202[((threadIdx.y)*dim_0 + idx_chan_in_202)*9 + 2] = l_202_2*x_0[((idx_batch)*dim_0 + idx_chan_in_202)*1 + 0];
        product_202[((threadIdx.y)*dim_0 + idx_chan_in_202)*9 + 3] = l_202_3*x_0[((idx_batch)*dim_0 + idx_chan_in_202)*1 + 0];
        product_202[((threadIdx.y)*dim_0 + idx_chan_in_202)*9 + 4] = l_202_4*x_0[((idx_batch)*dim_0 + idx_chan_in_202)*1 + 0];
        product_202[((threadIdx.y)*dim_0 + idx_chan_in_202)*9 + 5] = l_202_5*x_0[((idx_batch)*dim_0 + idx_chan_in_202)*1 + 0];
        product_202[((threadIdx.y)*dim_0 + idx_chan_in_202)*9 + 6] = l_202_6*x_0[((idx_batch)*dim_0 + idx_chan_in_202)*1 + 0];
        product_202[((threadIdx.y)*dim_0 + idx_chan_in_202)*9 + 7] = l_202_7*x_0[((idx_batch)*dim_0 + idx_chan_in_202)*1 + 0];
        product_202[((threadIdx.y)*dim_0 + idx_chan_in_202)*9 + 8] = l_202_8*x_0[((idx_batch)*dim_0 + idx_chan_in_202)*1 + 0];
      }
      float l_112_0 = left_112[((idx_batch)*dim_l + threadIdx.y)*3 + 0];
      float l_112_1 = left_112[((idx_batch)*dim_l + threadIdx.y)*3 + 1];
      float l_112_2 = left_112[((idx_batch)*dim_l + threadIdx.y)*3 + 2];
      for (int idx_chan_in_112 = threadIdx.x; idx_chan_in_112 < dim_1; idx_chan_in_112 += blockDim.x) {
        product_112[((threadIdx.y)*dim_1 + idx_chan_in_112)*9 + 0] = l_112_0*x_1[((idx_batch)*dim_1 + idx_chan_in_112)*3 + 0];
        product_112[((threadIdx.y)*dim_1 + idx_chan_in_112)*9 + 1] = l_112_0*x_1[((idx_batch)*dim_1 + idx_chan_in_112)*3 + 1];
        product_112[((threadIdx.y)*dim_1 + idx_chan_in_112)*9 + 2] = l_112_0*x_1[((idx_batch)*dim_1 + idx_chan_in_112)*3 + 2];
        product_112[((threadIdx.y)*dim_1 + idx_chan_in_112)*9 + 3] = l_112_1*x_1[((idx_batch)*dim_1 + idx_chan_in_112)*3 + 0];
        product_112[((threadIdx.y)*dim_1 + idx_chan_in_112)*9 + 4] = l_112_1*x_1[((idx_batch)*dim_1 + idx_chan_in_112)*3 + 1];
        product_112[((threadIdx.y)*dim_1 + idx_chan_in_112)*9 + 5] = l_112_1*x_1[((idx_batch)*dim_1 + idx_chan_in_112)*3 + 2];
        product_112[((threadIdx.y)*dim_1 + idx_chan_in_112)*9 + 6] = l_112_2*x_1[((idx_batch)*dim_1 + idx_chan_in_112)*3 + 0];
        product_112[((threadIdx.y)*dim_1 + idx_chan_in_112)*9 + 7] = l_112_2*x_1[((idx_batch)*dim_1 + idx_chan_in_112)*3 + 1];
        product_112[((threadIdx.y)*dim_1 + idx_chan_in_112)*9 + 8] = l_112_2*x_1[((idx_batch)*dim_1 + idx_chan_in_112)*3 + 2];
      }
      float l_222_0 = left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 0];
      float l_222_1 = left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 1];
      float l_222_2 = left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 2];
      float l_222_3 = left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 3];
      float l_222_4 = left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 4];
      float l_222_5 = left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 5];
      float l_222_6 = left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 6];
      float l_222_7 = left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 7];
      float l_222_8 = left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 8];
      for (int idx_chan_in_222 = threadIdx.x; idx_chan_in_222 < dim_2; idx_chan_in_222 += blockDim.x) {
        product_222[((threadIdx.y)*dim_2 + idx_chan_in_222)*9 + 0] = l_222_0*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 0] + l_222_1*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 1] + l_222_2*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 2];
        product_222[((threadIdx.y)*dim_2 + idx_chan_in_222)*9 + 1] = l_222_0*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 3] + l_222_1*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 4] + l_222_2*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 5];
        product_222[((threadIdx.y)*dim_2 + idx_chan_in_222)*9 + 2] = l_222_0*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 6] + l_222_1*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 7] + l_222_2*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 8];
        product_222[((threadIdx.y)*dim_2 + idx_chan_in_222)*9 + 3] = l_222_3*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 0] + l_222_4*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 1] + l_222_5*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 2];
        product_222[((threadIdx.y)*dim_2 + idx_chan_in_222)*9 + 4] = l_222_3*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 3] + l_222_4*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 4] + l_222_5*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 5];
        product_222[((threadIdx.y)*dim_2 + idx_chan_in_222)*9 + 5] = l_222_3*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 6] + l_222_4*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 7] + l_222_5*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 8];
        product_222[((threadIdx.y)*dim_2 + idx_chan_in_222)*9 + 6] = l_222_6*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 0] + l_222_7*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 1] + l_222_8*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 2];
        product_222[((threadIdx.y)*dim_2 + idx_chan_in_222)*9 + 7] = l_222_6*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 3] + l_222_7*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 4] + l_222_8*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 5];
        product_222[((threadIdx.y)*dim_2 + idx_chan_in_222)*9 + 8] = l_222_6*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 6] + l_222_7*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 7] + l_222_8*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 8];
      }
    }
    __syncthreads();
    { // linear transforms to compute the outputs
      for (int idx_chan_out_2 = threadIdx.y; idx_chan_out_2 < dim_2; idx_chan_out_2 += blockDim.y) {
        float y_o_2_0 = 0.0;
        float y_o_2_1 = 0.0;
        float y_o_2_2 = 0.0;
        float y_o_2_3 = 0.0;
        float y_o_2_4 = 0.0;
        float y_o_2_5 = 0.0;
        float y_o_2_6 = 0.0;
        float y_o_2_7 = 0.0;
        float y_o_2_8 = 0.0;
        float accum_022_0 = 0.0;
        float accum_022_1 = 0.0;
        float accum_022_2 = 0.0;
        float accum_022_3 = 0.0;
        float accum_022_4 = 0.0;
        float accum_022_5 = 0.0;
        float accum_022_6 = 0.0;
        float accum_022_7 = 0.0;
        float accum_022_8 = 0.0;
        for (int idx_chan_in_022 = threadIdx.x; idx_chan_in_022 < dim_l*dim_2; idx_chan_in_022 += blockDim.x) {
          float P_oi_022 = P_022[(idx_chan_out_2)*dim_l*dim_2 + idx_chan_in_022];
          accum_022_0 += P_oi_022*product_022[(idx_chan_in_022)*9 + 0];
          accum_022_1 += P_oi_022*product_022[(idx_chan_in_022)*9 + 1];
          accum_022_2 += P_oi_022*product_022[(idx_chan_in_022)*9 + 2];
          accum_022_3 += P_oi_022*product_022[(idx_chan_in_022)*9 + 3];
          accum_022_4 += P_oi_022*product_022[(idx_chan_in_022)*9 + 4];
          accum_022_5 += P_oi_022*product_022[(idx_chan_in_022)*9 + 5];
          accum_022_6 += P_oi_022*product_022[(idx_chan_in_022)*9 + 6];
          accum_022_7 += P_oi_022*product_022[(idx_chan_in_022)*9 + 7];
          accum_022_8 += P_oi_022*product_022[(idx_chan_in_022)*9 + 8];
        }
        // reduce across the warp so that first thread in warp will have the sum 
        for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
          accum_022_0 += __shfl_down_sync(0xffffffff, accum_022_0, offset);
          accum_022_1 += __shfl_down_sync(0xffffffff, accum_022_1, offset);
          accum_022_2 += __shfl_down_sync(0xffffffff, accum_022_2, offset);
          accum_022_3 += __shfl_down_sync(0xffffffff, accum_022_3, offset);
          accum_022_4 += __shfl_down_sync(0xffffffff, accum_022_4, offset);
          accum_022_5 += __shfl_down_sync(0xffffffff, accum_022_5, offset);
          accum_022_6 += __shfl_down_sync(0xffffffff, accum_022_6, offset);
          accum_022_7 += __shfl_down_sync(0xffffffff, accum_022_7, offset);
          accum_022_8 += __shfl_down_sync(0xffffffff, accum_022_8, offset);
        }
        if (threadIdx.x == 0) {
          y_o_2_0 += accum_022_0;
          y_o_2_1 += accum_022_1;
          y_o_2_2 += accum_022_2;
          y_o_2_3 += accum_022_3;
          y_o_2_4 += accum_022_4;
          y_o_2_5 += accum_022_5;
          y_o_2_6 += accum_022_6;
          y_o_2_7 += accum_022_7;
          y_o_2_8 += accum_022_8;
        }
        float accum_202_0 = 0.0;
        float accum_202_1 = 0.0;
        float accum_202_2 = 0.0;
        float accum_202_3 = 0.0;
        float accum_202_4 = 0.0;
        float accum_202_5 = 0.0;
        float accum_202_6 = 0.0;
        float accum_202_7 = 0.0;
        float accum_202_8 = 0.0;
        for (int idx_chan_in_202 = threadIdx.x; idx_chan_in_202 < dim_l*dim_0; idx_chan_in_202 += blockDim.x) {
          float P_oi_202 = P_202[(idx_chan_out_2)*dim_l*dim_0 + idx_chan_in_202];
          accum_202_0 += P_oi_202*product_202[(idx_chan_in_202)*9 + 0];
          accum_202_1 += P_oi_202*product_202[(idx_chan_in_202)*9 + 1];
          accum_202_2 += P_oi_202*product_202[(idx_chan_in_202)*9 + 2];
          accum_202_3 += P_oi_202*product_202[(idx_chan_in_202)*9 + 3];
          accum_202_4 += P_oi_202*product_202[(idx_chan_in_202)*9 + 4];
          accum_202_5 += P_oi_202*product_202[(idx_chan_in_202)*9 + 5];
          accum_202_6 += P_oi_202*product_202[(idx_chan_in_202)*9 + 6];
          accum_202_7 += P_oi_202*product_202[(idx_chan_in_202)*9 + 7];
          accum_202_8 += P_oi_202*product_202[(idx_chan_in_202)*9 + 8];
        }
        // reduce across the warp so that first thread in warp will have the sum 
        for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
          accum_202_0 += __shfl_down_sync(0xffffffff, accum_202_0, offset);
          accum_202_1 += __shfl_down_sync(0xffffffff, accum_202_1, offset);
          accum_202_2 += __shfl_down_sync(0xffffffff, accum_202_2, offset);
          accum_202_3 += __shfl_down_sync(0xffffffff, accum_202_3, offset);
          accum_202_4 += __shfl_down_sync(0xffffffff, accum_202_4, offset);
          accum_202_5 += __shfl_down_sync(0xffffffff, accum_202_5, offset);
          accum_202_6 += __shfl_down_sync(0xffffffff, accum_202_6, offset);
          accum_202_7 += __shfl_down_sync(0xffffffff, accum_202_7, offset);
          accum_202_8 += __shfl_down_sync(0xffffffff, accum_202_8, offset);
        }
        if (threadIdx.x == 0) {
          y_o_2_0 += accum_202_0;
          y_o_2_1 += accum_202_1;
          y_o_2_2 += accum_202_2;
          y_o_2_3 += accum_202_3;
          y_o_2_4 += accum_202_4;
          y_o_2_5 += accum_202_5;
          y_o_2_6 += accum_202_6;
          y_o_2_7 += accum_202_7;
          y_o_2_8 += accum_202_8;
        }
        float accum_112_0 = 0.0;
        float accum_112_1 = 0.0;
        float accum_112_2 = 0.0;
        float accum_112_3 = 0.0;
        float accum_112_4 = 0.0;
        float accum_112_5 = 0.0;
        float accum_112_6 = 0.0;
        float accum_112_7 = 0.0;
        float accum_112_8 = 0.0;
        for (int idx_chan_in_112 = threadIdx.x; idx_chan_in_112 < dim_l*dim_1; idx_chan_in_112 += blockDim.x) {
          float P_oi_112 = P_112[(idx_chan_out_2)*dim_l*dim_1 + idx_chan_in_112];
          accum_112_0 += P_oi_112*product_112[(idx_chan_in_112)*9 + 0];
          accum_112_1 += P_oi_112*product_112[(idx_chan_in_112)*9 + 1];
          accum_112_2 += P_oi_112*product_112[(idx_chan_in_112)*9 + 2];
          accum_112_3 += P_oi_112*product_112[(idx_chan_in_112)*9 + 3];
          accum_112_4 += P_oi_112*product_112[(idx_chan_in_112)*9 + 4];
          accum_112_5 += P_oi_112*product_112[(idx_chan_in_112)*9 + 5];
          accum_112_6 += P_oi_112*product_112[(idx_chan_in_112)*9 + 6];
          accum_112_7 += P_oi_112*product_112[(idx_chan_in_112)*9 + 7];
          accum_112_8 += P_oi_112*product_112[(idx_chan_in_112)*9 + 8];
        }
        // reduce across the warp so that first thread in warp will have the sum 
        for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
          accum_112_0 += __shfl_down_sync(0xffffffff, accum_112_0, offset);
          accum_112_1 += __shfl_down_sync(0xffffffff, accum_112_1, offset);
          accum_112_2 += __shfl_down_sync(0xffffffff, accum_112_2, offset);
          accum_112_3 += __shfl_down_sync(0xffffffff, accum_112_3, offset);
          accum_112_4 += __shfl_down_sync(0xffffffff, accum_112_4, offset);
          accum_112_5 += __shfl_down_sync(0xffffffff, accum_112_5, offset);
          accum_112_6 += __shfl_down_sync(0xffffffff, accum_112_6, offset);
          accum_112_7 += __shfl_down_sync(0xffffffff, accum_112_7, offset);
          accum_112_8 += __shfl_down_sync(0xffffffff, accum_112_8, offset);
        }
        if (threadIdx.x == 0) {
          y_o_2_0 += accum_112_0;
          y_o_2_1 += accum_112_1;
          y_o_2_2 += accum_112_2;
          y_o_2_3 += accum_112_3;
          y_o_2_4 += accum_112_4;
          y_o_2_5 += accum_112_5;
          y_o_2_6 += accum_112_6;
          y_o_2_7 += accum_112_7;
          y_o_2_8 += accum_112_8;
        }
        float accum_222_0 = 0.0;
        float accum_222_1 = 0.0;
        float accum_222_2 = 0.0;
        float accum_222_3 = 0.0;
        float accum_222_4 = 0.0;
        float accum_222_5 = 0.0;
        float accum_222_6 = 0.0;
        float accum_222_7 = 0.0;
        float accum_222_8 = 0.0;
        for (int idx_chan_in_222 = threadIdx.x; idx_chan_in_222 < dim_l*dim_2; idx_chan_in_222 += blockDim.x) {
          float P_oi_222 = P_222[(idx_chan_out_2)*dim_l*dim_2 + idx_chan_in_222];
          accum_222_0 += P_oi_222*product_222[(idx_chan_in_222)*9 + 0];
          accum_222_1 += P_oi_222*product_222[(idx_chan_in_222)*9 + 1];
          accum_222_2 += P_oi_222*product_222[(idx_chan_in_222)*9 + 2];
          accum_222_3 += P_oi_222*product_222[(idx_chan_in_222)*9 + 3];
          accum_222_4 += P_oi_222*product_222[(idx_chan_in_222)*9 + 4];
          accum_222_5 += P_oi_222*product_222[(idx_chan_in_222)*9 + 5];
          accum_222_6 += P_oi_222*product_222[(idx_chan_in_222)*9 + 6];
          accum_222_7 += P_oi_222*product_222[(idx_chan_in_222)*9 + 7];
          accum_222_8 += P_oi_222*product_222[(idx_chan_in_222)*9 + 8];
        }
        // reduce across the warp so that first thread in warp will have the sum 
        for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
          accum_222_0 += __shfl_down_sync(0xffffffff, accum_222_0, offset);
          accum_222_1 += __shfl_down_sync(0xffffffff, accum_222_1, offset);
          accum_222_2 += __shfl_down_sync(0xffffffff, accum_222_2, offset);
          accum_222_3 += __shfl_down_sync(0xffffffff, accum_222_3, offset);
          accum_222_4 += __shfl_down_sync(0xffffffff, accum_222_4, offset);
          accum_222_5 += __shfl_down_sync(0xffffffff, accum_222_5, offset);
          accum_222_6 += __shfl_down_sync(0xffffffff, accum_222_6, offset);
          accum_222_7 += __shfl_down_sync(0xffffffff, accum_222_7, offset);
          accum_222_8 += __shfl_down_sync(0xffffffff, accum_222_8, offset);
        }
        if (threadIdx.x == 0) {
          y_o_2_0 += accum_222_0;
          y_o_2_1 += accum_222_1;
          y_o_2_2 += accum_222_2;
          y_o_2_3 += accum_222_3;
          y_o_2_4 += accum_222_4;
          y_o_2_5 += accum_222_5;
          y_o_2_6 += accum_222_6;
          y_o_2_7 += accum_222_7;
          y_o_2_8 += accum_222_8;
        }
        if (threadIdx.x == 0) {
          y_2[((idx_batch)*dim_2 + idx_chan_out_2)*9 + 0] = y_o_2_0;
          y_2[((idx_batch)*dim_2 + idx_chan_out_2)*9 + 1] = y_o_2_1;
          y_2[((idx_batch)*dim_2 + idx_chan_out_2)*9 + 2] = y_o_2_2;
          y_2[((idx_batch)*dim_2 + idx_chan_out_2)*9 + 3] = y_o_2_3;
          y_2[((idx_batch)*dim_2 + idx_chan_out_2)*9 + 4] = y_o_2_4;
          y_2[((idx_batch)*dim_2 + idx_chan_out_2)*9 + 5] = y_o_2_5;
          y_2[((idx_batch)*dim_2 + idx_chan_out_2)*9 + 6] = y_o_2_6;
          y_2[((idx_batch)*dim_2 + idx_chan_out_2)*9 + 7] = y_o_2_7;
          y_2[((idx_batch)*dim_2 + idx_chan_out_2)*9 + 8] = y_o_2_8;
        }
      }
    }
  }
}


void ant16_o2(
    int batch, int dim_l, int dim_0, int dim_1, int dim_2,
    const float* x_0, const float* x_1, const float* x_2, const float* P_022, const float* left_022, const float* P_202, const float* left_202, const float* P_112, const float* left_112, const float* P_222, const float* left_222,
    float* y_2) {
  
  int p_2 = dim_l*dim_2;
  int p_0 = dim_l*dim_0;
  int p_1 = dim_l*dim_1;
  int sharedmemsz = 0;
  sharedmemsz += 18*p_2;
  int p_0_base = sharedmemsz;
  sharedmemsz += 9*p_0;
  int p_1_base = sharedmemsz;
  sharedmemsz += 9*p_1;
  dim3 gridsz = dim3(batch);
  dim3 blocksz = dim3(WARPSZ, dim_l);
  ant16_o2_kern<<<gridsz, blocksz, sharedmemsz*sizeof(float)>>>(
      p_2, p_0_base, p_0, p_1_base, p_1, 
      batch, dim_l, dim_0, dim_1, dim_2,
      x_0, x_1, x_2, P_022, left_022, P_202, left_202, P_112, left_112, P_222, left_222,
      y_2);
  
}


__global__
void ant16_o2_backward_kern(
    // <<<(batch), (WARPSZ, dim_l)>>>
    int p_2, 
    int batch, int dim_l, int dim_0, int dim_1, int dim_2,
    const float* dy_2, const float* P_022, const float* left_022, const float* P_202, const float* left_202, const float* P_112, const float* left_112, const float* P_222, const float* left_222,
    float* __restrict__ dx_0, float* __restrict__ dx_1, float* __restrict__ dx_2) {
  extern __shared__ float s[];
  float* dproduct_022 = &s[0*p_2]; // size = 9*p_2
  float* dproduct_202 = &s[9*p_2]; // size = 1*p_2
  float* dproduct_112 = &s[10*p_2]; // size = 3*p_2
  float* dproduct_222 = &s[13*p_2]; // size = 9*p_2
  for (int idx_batch = blockIdx.x; idx_batch < batch; idx_batch += gridDim.x) {
    { // compute tensor products
      float l_022_0 = left_022[((idx_batch)*dim_l + threadIdx.y)*1 + 0];
      for (int idx_chan_out_022 = threadIdx.x; idx_chan_out_022 < dim_2; idx_chan_out_022 += blockDim.x) {
        dproduct_022[((threadIdx.y)*dim_2 + idx_chan_out_022)*9 + 0] = l_022_0*dy_2[((idx_batch)*dim_2 + idx_chan_out_022)*9 + 0];
        dproduct_022[((threadIdx.y)*dim_2 + idx_chan_out_022)*9 + 1] = l_022_0*dy_2[((idx_batch)*dim_2 + idx_chan_out_022)*9 + 1];
        dproduct_022[((threadIdx.y)*dim_2 + idx_chan_out_022)*9 + 2] = l_022_0*dy_2[((idx_batch)*dim_2 + idx_chan_out_022)*9 + 2];
        dproduct_022[((threadIdx.y)*dim_2 + idx_chan_out_022)*9 + 3] = l_022_0*dy_2[((idx_batch)*dim_2 + idx_chan_out_022)*9 + 3];
        dproduct_022[((threadIdx.y)*dim_2 + idx_chan_out_022)*9 + 4] = l_022_0*dy_2[((idx_batch)*dim_2 + idx_chan_out_022)*9 + 4];
        dproduct_022[((threadIdx.y)*dim_2 + idx_chan_out_022)*9 + 5] = l_022_0*dy_2[((idx_batch)*dim_2 + idx_chan_out_022)*9 + 5];
        dproduct_022[((threadIdx.y)*dim_2 + idx_chan_out_022)*9 + 6] = l_022_0*dy_2[((idx_batch)*dim_2 + idx_chan_out_022)*9 + 6];
        dproduct_022[((threadIdx.y)*dim_2 + idx_chan_out_022)*9 + 7] = l_022_0*dy_2[((idx_batch)*dim_2 + idx_chan_out_022)*9 + 7];
        dproduct_022[((threadIdx.y)*dim_2 + idx_chan_out_022)*9 + 8] = l_022_0*dy_2[((idx_batch)*dim_2 + idx_chan_out_022)*9 + 8];
      }
      float l_202_0 = left_202[((idx_batch)*dim_l + threadIdx.y)*9 + 0];
      float l_202_1 = left_202[((idx_batch)*dim_l + threadIdx.y)*9 + 1];
      float l_202_2 = left_202[((idx_batch)*dim_l + threadIdx.y)*9 + 2];
      float l_202_3 = left_202[((idx_batch)*dim_l + threadIdx.y)*9 + 3];
      float l_202_4 = left_202[((idx_batch)*dim_l + threadIdx.y)*9 + 4];
      float l_202_5 = left_202[((idx_batch)*dim_l + threadIdx.y)*9 + 5];
      float l_202_6 = left_202[((idx_batch)*dim_l + threadIdx.y)*9 + 6];
      float l_202_7 = left_202[((idx_batch)*dim_l + threadIdx.y)*9 + 7];
      float l_202_8 = left_202[((idx_batch)*dim_l + threadIdx.y)*9 + 8];
      for (int idx_chan_out_202 = threadIdx.x; idx_chan_out_202 < dim_2; idx_chan_out_202 += blockDim.x) {
        dproduct_202[((threadIdx.y)*dim_2 + idx_chan_out_202)*1 + 0] = l_202_0*dy_2[((idx_batch)*dim_2 + idx_chan_out_202)*9 + 0] + l_202_1*dy_2[((idx_batch)*dim_2 + idx_chan_out_202)*9 + 1] + l_202_2*dy_2[((idx_batch)*dim_2 + idx_chan_out_202)*9 + 2] + l_202_3*dy_2[((idx_batch)*dim_2 + idx_chan_out_202)*9 + 3] + l_202_4*dy_2[((idx_batch)*dim_2 + idx_chan_out_202)*9 + 4] + l_202_5*dy_2[((idx_batch)*dim_2 + idx_chan_out_202)*9 + 5] + l_202_6*dy_2[((idx_batch)*dim_2 + idx_chan_out_202)*9 + 6] + l_202_7*dy_2[((idx_batch)*dim_2 + idx_chan_out_202)*9 + 7] + l_202_8*dy_2[((idx_batch)*dim_2 + idx_chan_out_202)*9 + 8];
      }
      float l_112_0 = left_112[((idx_batch)*dim_l + threadIdx.y)*3 + 0];
      float l_112_1 = left_112[((idx_batch)*dim_l + threadIdx.y)*3 + 1];
      float l_112_2 = left_112[((idx_batch)*dim_l + threadIdx.y)*3 + 2];
      for (int idx_chan_out_112 = threadIdx.x; idx_chan_out_112 < dim_2; idx_chan_out_112 += blockDim.x) {
        dproduct_112[((threadIdx.y)*dim_2 + idx_chan_out_112)*3 + 0] = l_112_0*dy_2[((idx_batch)*dim_2 + idx_chan_out_112)*9 + 0] + l_112_1*dy_2[((idx_batch)*dim_2 + idx_chan_out_112)*9 + 3] + l_112_2*dy_2[((idx_batch)*dim_2 + idx_chan_out_112)*9 + 6];
        dproduct_112[((threadIdx.y)*dim_2 + idx_chan_out_112)*3 + 1] = l_112_0*dy_2[((idx_batch)*dim_2 + idx_chan_out_112)*9 + 1] + l_112_1*dy_2[((idx_batch)*dim_2 + idx_chan_out_112)*9 + 4] + l_112_2*dy_2[((idx_batch)*dim_2 + idx_chan_out_112)*9 + 7];
        dproduct_112[((threadIdx.y)*dim_2 + idx_chan_out_112)*3 + 2] = l_112_0*dy_2[((idx_batch)*dim_2 + idx_chan_out_112)*9 + 2] + l_112_1*dy_2[((idx_batch)*dim_2 + idx_chan_out_112)*9 + 5] + l_112_2*dy_2[((idx_batch)*dim_2 + idx_chan_out_112)*9 + 8];
      }
      float l_222_0 = left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 0];
      float l_222_1 = left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 1];
      float l_222_2 = left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 2];
      float l_222_3 = left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 3];
      float l_222_4 = left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 4];
      float l_222_5 = left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 5];
      float l_222_6 = left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 6];
      float l_222_7 = left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 7];
      float l_222_8 = left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 8];
      for (int idx_chan_out_222 = threadIdx.x; idx_chan_out_222 < dim_2; idx_chan_out_222 += blockDim.x) {
        dproduct_222[((threadIdx.y)*dim_2 + idx_chan_out_222)*9 + 0] = l_222_0*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 0] + l_222_3*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 3] + l_222_6*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 6];
        dproduct_222[((threadIdx.y)*dim_2 + idx_chan_out_222)*9 + 1] = l_222_1*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 0] + l_222_4*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 3] + l_222_7*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 6];
        dproduct_222[((threadIdx.y)*dim_2 + idx_chan_out_222)*9 + 2] = l_222_2*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 0] + l_222_5*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 3] + l_222_8*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 6];
        dproduct_222[((threadIdx.y)*dim_2 + idx_chan_out_222)*9 + 3] = l_222_0*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 1] + l_222_3*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 4] + l_222_6*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 7];
        dproduct_222[((threadIdx.y)*dim_2 + idx_chan_out_222)*9 + 4] = l_222_1*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 1] + l_222_4*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 4] + l_222_7*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 7];
        dproduct_222[((threadIdx.y)*dim_2 + idx_chan_out_222)*9 + 5] = l_222_2*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 1] + l_222_5*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 4] + l_222_8*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 7];
        dproduct_222[((threadIdx.y)*dim_2 + idx_chan_out_222)*9 + 6] = l_222_0*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 2] + l_222_3*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 5] + l_222_6*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 8];
        dproduct_222[((threadIdx.y)*dim_2 + idx_chan_out_222)*9 + 7] = l_222_1*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 2] + l_222_4*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 5] + l_222_7*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 8];
        dproduct_222[((threadIdx.y)*dim_2 + idx_chan_out_222)*9 + 8] = l_222_2*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 2] + l_222_5*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 5] + l_222_8*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 8];
      }
    }
    __syncthreads();
    { // linear transforms to compute dx
      for (int idx_chan_in_0 = threadIdx.y; idx_chan_in_0 < dim_0; idx_chan_in_0 += blockDim.y) {
        float dx_o_0_0 = 0.0;
        float accum_202_0 = 0.0;
        for (int idx_l_202 = 0; idx_l_202 < dim_l; idx_l_202 += 1) {
          for (int idx_chan_out_202 = threadIdx.x; idx_chan_out_202 < dim_2; idx_chan_out_202 += blockDim.x) {
            float P_oi_202 = P_202[((idx_chan_out_202)*dim_l + idx_l_202)*dim_0 + idx_chan_in_0];
            accum_202_0 += P_oi_202*dproduct_202[((idx_l_202)*dim_2 + idx_chan_out_202)*1 + 0];
          }
        }
        // reduce across the warp so that first thread in warp will have the sum 
        for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
          accum_202_0 += __shfl_down_sync(0xffffffff, accum_202_0, offset);
        }
        if (threadIdx.x == 0) {
          dx_o_0_0 += accum_202_0;
        }
        if (threadIdx.x == 0) {
          dx_0[((idx_batch)*dim_0 + idx_chan_in_0)*1 + 0] = dx_o_0_0;
        }
      }
      for (int idx_chan_in_1 = threadIdx.y; idx_chan_in_1 < dim_1; idx_chan_in_1 += blockDim.y) {
        float dx_o_1_0 = 0.0;
        float dx_o_1_1 = 0.0;
        float dx_o_1_2 = 0.0;
        float accum_112_0 = 0.0;
        float accum_112_1 = 0.0;
        float accum_112_2 = 0.0;
        for (int idx_l_112 = 0; idx_l_112 < dim_l; idx_l_112 += 1) {
          for (int idx_chan_out_112 = threadIdx.x; idx_chan_out_112 < dim_2; idx_chan_out_112 += blockDim.x) {
            float P_oi_112 = P_112[((idx_chan_out_112)*dim_l + idx_l_112)*dim_1 + idx_chan_in_1];
            accum_112_0 += P_oi_112*dproduct_112[((idx_l_112)*dim_2 + idx_chan_out_112)*3 + 0];
            accum_112_1 += P_oi_112*dproduct_112[((idx_l_112)*dim_2 + idx_chan_out_112)*3 + 1];
            accum_112_2 += P_oi_112*dproduct_112[((idx_l_112)*dim_2 + idx_chan_out_112)*3 + 2];
          }
        }
        // reduce across the warp so that first thread in warp will have the sum 
        for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
          accum_112_0 += __shfl_down_sync(0xffffffff, accum_112_0, offset);
          accum_112_1 += __shfl_down_sync(0xffffffff, accum_112_1, offset);
          accum_112_2 += __shfl_down_sync(0xffffffff, accum_112_2, offset);
        }
        if (threadIdx.x == 0) {
          dx_o_1_0 += accum_112_0;
          dx_o_1_1 += accum_112_1;
          dx_o_1_2 += accum_112_2;
        }
        if (threadIdx.x == 0) {
          dx_1[((idx_batch)*dim_1 + idx_chan_in_1)*3 + 0] = dx_o_1_0;
          dx_1[((idx_batch)*dim_1 + idx_chan_in_1)*3 + 1] = dx_o_1_1;
          dx_1[((idx_batch)*dim_1 + idx_chan_in_1)*3 + 2] = dx_o_1_2;
        }
      }
      for (int idx_chan_in_2 = threadIdx.y; idx_chan_in_2 < dim_2; idx_chan_in_2 += blockDim.y) {
        float dx_o_2_0 = 0.0;
        float dx_o_2_1 = 0.0;
        float dx_o_2_2 = 0.0;
        float dx_o_2_3 = 0.0;
        float dx_o_2_4 = 0.0;
        float dx_o_2_5 = 0.0;
        float dx_o_2_6 = 0.0;
        float dx_o_2_7 = 0.0;
        float dx_o_2_8 = 0.0;
        float accum_022_0 = 0.0;
        float accum_022_1 = 0.0;
        float accum_022_2 = 0.0;
        float accum_022_3 = 0.0;
        float accum_022_4 = 0.0;
        float accum_022_5 = 0.0;
        float accum_022_6 = 0.0;
        float accum_022_7 = 0.0;
        float accum_022_8 = 0.0;
        for (int idx_l_022 = 0; idx_l_022 < dim_l; idx_l_022 += 1) {
          for (int idx_chan_out_022 = threadIdx.x; idx_chan_out_022 < dim_2; idx_chan_out_022 += blockDim.x) {
            float P_oi_022 = P_022[((idx_chan_out_022)*dim_l + idx_l_022)*dim_2 + idx_chan_in_2];
            accum_022_0 += P_oi_022*dproduct_022[((idx_l_022)*dim_2 + idx_chan_out_022)*9 + 0];
            accum_022_1 += P_oi_022*dproduct_022[((idx_l_022)*dim_2 + idx_chan_out_022)*9 + 1];
            accum_022_2 += P_oi_022*dproduct_022[((idx_l_022)*dim_2 + idx_chan_out_022)*9 + 2];
            accum_022_3 += P_oi_022*dproduct_022[((idx_l_022)*dim_2 + idx_chan_out_022)*9 + 3];
            accum_022_4 += P_oi_022*dproduct_022[((idx_l_022)*dim_2 + idx_chan_out_022)*9 + 4];
            accum_022_5 += P_oi_022*dproduct_022[((idx_l_022)*dim_2 + idx_chan_out_022)*9 + 5];
            accum_022_6 += P_oi_022*dproduct_022[((idx_l_022)*dim_2 + idx_chan_out_022)*9 + 6];
            accum_022_7 += P_oi_022*dproduct_022[((idx_l_022)*dim_2 + idx_chan_out_022)*9 + 7];
            accum_022_8 += P_oi_022*dproduct_022[((idx_l_022)*dim_2 + idx_chan_out_022)*9 + 8];
          }
        }
        // reduce across the warp so that first thread in warp will have the sum 
        for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
          accum_022_0 += __shfl_down_sync(0xffffffff, accum_022_0, offset);
          accum_022_1 += __shfl_down_sync(0xffffffff, accum_022_1, offset);
          accum_022_2 += __shfl_down_sync(0xffffffff, accum_022_2, offset);
          accum_022_3 += __shfl_down_sync(0xffffffff, accum_022_3, offset);
          accum_022_4 += __shfl_down_sync(0xffffffff, accum_022_4, offset);
          accum_022_5 += __shfl_down_sync(0xffffffff, accum_022_5, offset);
          accum_022_6 += __shfl_down_sync(0xffffffff, accum_022_6, offset);
          accum_022_7 += __shfl_down_sync(0xffffffff, accum_022_7, offset);
          accum_022_8 += __shfl_down_sync(0xffffffff, accum_022_8, offset);
        }
        if (threadIdx.x == 0) {
          dx_o_2_0 += accum_022_0;
          dx_o_2_1 += accum_022_1;
          dx_o_2_2 += accum_022_2;
          dx_o_2_3 += accum_022_3;
          dx_o_2_4 += accum_022_4;
          dx_o_2_5 += accum_022_5;
          dx_o_2_6 += accum_022_6;
          dx_o_2_7 += accum_022_7;
          dx_o_2_8 += accum_022_8;
        }
        float accum_222_0 = 0.0;
        float accum_222_1 = 0.0;
        float accum_222_2 = 0.0;
        float accum_222_3 = 0.0;
        float accum_222_4 = 0.0;
        float accum_222_5 = 0.0;
        float accum_222_6 = 0.0;
        float accum_222_7 = 0.0;
        float accum_222_8 = 0.0;
        for (int idx_l_222 = 0; idx_l_222 < dim_l; idx_l_222 += 1) {
          for (int idx_chan_out_222 = threadIdx.x; idx_chan_out_222 < dim_2; idx_chan_out_222 += blockDim.x) {
            float P_oi_222 = P_222[((idx_chan_out_222)*dim_l + idx_l_222)*dim_2 + idx_chan_in_2];
            accum_222_0 += P_oi_222*dproduct_222[((idx_l_222)*dim_2 + idx_chan_out_222)*9 + 0];
            accum_222_1 += P_oi_222*dproduct_222[((idx_l_222)*dim_2 + idx_chan_out_222)*9 + 1];
            accum_222_2 += P_oi_222*dproduct_222[((idx_l_222)*dim_2 + idx_chan_out_222)*9 + 2];
            accum_222_3 += P_oi_222*dproduct_222[((idx_l_222)*dim_2 + idx_chan_out_222)*9 + 3];
            accum_222_4 += P_oi_222*dproduct_222[((idx_l_222)*dim_2 + idx_chan_out_222)*9 + 4];
            accum_222_5 += P_oi_222*dproduct_222[((idx_l_222)*dim_2 + idx_chan_out_222)*9 + 5];
            accum_222_6 += P_oi_222*dproduct_222[((idx_l_222)*dim_2 + idx_chan_out_222)*9 + 6];
            accum_222_7 += P_oi_222*dproduct_222[((idx_l_222)*dim_2 + idx_chan_out_222)*9 + 7];
            accum_222_8 += P_oi_222*dproduct_222[((idx_l_222)*dim_2 + idx_chan_out_222)*9 + 8];
          }
        }
        // reduce across the warp so that first thread in warp will have the sum 
        for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
          accum_222_0 += __shfl_down_sync(0xffffffff, accum_222_0, offset);
          accum_222_1 += __shfl_down_sync(0xffffffff, accum_222_1, offset);
          accum_222_2 += __shfl_down_sync(0xffffffff, accum_222_2, offset);
          accum_222_3 += __shfl_down_sync(0xffffffff, accum_222_3, offset);
          accum_222_4 += __shfl_down_sync(0xffffffff, accum_222_4, offset);
          accum_222_5 += __shfl_down_sync(0xffffffff, accum_222_5, offset);
          accum_222_6 += __shfl_down_sync(0xffffffff, accum_222_6, offset);
          accum_222_7 += __shfl_down_sync(0xffffffff, accum_222_7, offset);
          accum_222_8 += __shfl_down_sync(0xffffffff, accum_222_8, offset);
        }
        if (threadIdx.x == 0) {
          dx_o_2_0 += accum_222_0;
          dx_o_2_1 += accum_222_1;
          dx_o_2_2 += accum_222_2;
          dx_o_2_3 += accum_222_3;
          dx_o_2_4 += accum_222_4;
          dx_o_2_5 += accum_222_5;
          dx_o_2_6 += accum_222_6;
          dx_o_2_7 += accum_222_7;
          dx_o_2_8 += accum_222_8;
        }
        if (threadIdx.x == 0) {
          dx_2[((idx_batch)*dim_2 + idx_chan_in_2)*9 + 0] = dx_o_2_0;
          dx_2[((idx_batch)*dim_2 + idx_chan_in_2)*9 + 1] = dx_o_2_1;
          dx_2[((idx_batch)*dim_2 + idx_chan_in_2)*9 + 2] = dx_o_2_2;
          dx_2[((idx_batch)*dim_2 + idx_chan_in_2)*9 + 3] = dx_o_2_3;
          dx_2[((idx_batch)*dim_2 + idx_chan_in_2)*9 + 4] = dx_o_2_4;
          dx_2[((idx_batch)*dim_2 + idx_chan_in_2)*9 + 5] = dx_o_2_5;
          dx_2[((idx_batch)*dim_2 + idx_chan_in_2)*9 + 6] = dx_o_2_6;
          dx_2[((idx_batch)*dim_2 + idx_chan_in_2)*9 + 7] = dx_o_2_7;
          dx_2[((idx_batch)*dim_2 + idx_chan_in_2)*9 + 8] = dx_o_2_8;
        }
      }
    }
  }
}


void ant16_o2_backward(
    int batch, int dim_l, int dim_0, int dim_1, int dim_2,
    const float* dy_2, const float* P_022, const float* left_022, const float* P_202, const float* left_202, const float* P_112, const float* left_112, const float* P_222, const float* left_222,
    float* dx_0, float* dx_1, float* dx_2) {
  
  int p_2 = dim_l*dim_2;
  int sharedmemsz = 0;
  sharedmemsz += 22*p_2;
  dim3 gridsz = dim3(batch);
  dim3 blocksz = dim3(WARPSZ, dim_l);
  ant16_o2_backward_kern<<<gridsz, blocksz, sharedmemsz*sizeof(float)>>>(
      p_2, 
      batch, dim_l, dim_0, dim_1, dim_2,
      dy_2, P_022, left_022, P_202, left_202, P_112, left_112, P_222, left_222,
      dx_0, dx_1, dx_2);
  
}


__global__
void ant16_o2_backleft_kern(
    // <<<(batch), (WARPSZ, dim_l)>>>
    
    int batch, int dim_l, int dim_0, int dim_1, int dim_2,
    const float* x_0, const float* x_1, const float* x_2, const float* dy_2, const float* P_022, const float* P_202, const float* P_112, const float* P_222,
    float* __restrict__ dleft_022, float* __restrict__ dleft_202, float* __restrict__ dleft_112, float* __restrict__ dleft_222) {
  extern __shared__ float s[];
  for (int idx_batch = blockIdx.x; idx_batch < batch; idx_batch += gridDim.x) {
    { // compute left derivative tensor products
      float accum_022_0 = 0.0;
      for (int idx_chan_in_022 = threadIdx.x; idx_chan_in_022 < dim_2; idx_chan_in_022 += blockDim.x) {
        for (int idx_chan_out_022 = 0; idx_chan_out_022 < dim_2; idx_chan_out_022 += 1) {
          float l_022_0 = x_2[((idx_batch)*dim_2 + idx_chan_in_022)*9 + 0]*dy_2[((idx_batch)*dim_2 + idx_chan_out_022)*9 + 0] + x_2[((idx_batch)*dim_2 + idx_chan_in_022)*9 + 1]*dy_2[((idx_batch)*dim_2 + idx_chan_out_022)*9 + 1] + x_2[((idx_batch)*dim_2 + idx_chan_in_022)*9 + 2]*dy_2[((idx_batch)*dim_2 + idx_chan_out_022)*9 + 2] + x_2[((idx_batch)*dim_2 + idx_chan_in_022)*9 + 3]*dy_2[((idx_batch)*dim_2 + idx_chan_out_022)*9 + 3] + x_2[((idx_batch)*dim_2 + idx_chan_in_022)*9 + 4]*dy_2[((idx_batch)*dim_2 + idx_chan_out_022)*9 + 4] + x_2[((idx_batch)*dim_2 + idx_chan_in_022)*9 + 5]*dy_2[((idx_batch)*dim_2 + idx_chan_out_022)*9 + 5] + x_2[((idx_batch)*dim_2 + idx_chan_in_022)*9 + 6]*dy_2[((idx_batch)*dim_2 + idx_chan_out_022)*9 + 6] + x_2[((idx_batch)*dim_2 + idx_chan_in_022)*9 + 7]*dy_2[((idx_batch)*dim_2 + idx_chan_out_022)*9 + 7] + x_2[((idx_batch)*dim_2 + idx_chan_in_022)*9 + 8]*dy_2[((idx_batch)*dim_2 + idx_chan_out_022)*9 + 8];
          float P_oi_022 = P_022[((idx_chan_out_022)*blockDim.y + threadIdx.y)*dim_2 + idx_chan_in_022];
          accum_022_0 += P_oi_022*l_022_0;
        }
      }
      // reduce across the warp so that first thread in warp will have the sum 
      for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
        accum_022_0 += __shfl_down_sync(0xffffffff, accum_022_0, offset);
      }
      if (threadIdx.x == 0) {
        dleft_022[((idx_batch)*dim_l + threadIdx.y)*1 + 0] = accum_022_0;
      }
      float accum_202_0 = 0.0;
      float accum_202_1 = 0.0;
      float accum_202_2 = 0.0;
      float accum_202_3 = 0.0;
      float accum_202_4 = 0.0;
      float accum_202_5 = 0.0;
      float accum_202_6 = 0.0;
      float accum_202_7 = 0.0;
      float accum_202_8 = 0.0;
      for (int idx_chan_in_202 = threadIdx.x; idx_chan_in_202 < dim_0; idx_chan_in_202 += blockDim.x) {
        for (int idx_chan_out_202 = 0; idx_chan_out_202 < dim_2; idx_chan_out_202 += 1) {
          float l_202_0 = x_0[((idx_batch)*dim_0 + idx_chan_in_202)*1 + 0]*dy_2[((idx_batch)*dim_2 + idx_chan_out_202)*9 + 0];
          float l_202_1 = x_0[((idx_batch)*dim_0 + idx_chan_in_202)*1 + 0]*dy_2[((idx_batch)*dim_2 + idx_chan_out_202)*9 + 1];
          float l_202_2 = x_0[((idx_batch)*dim_0 + idx_chan_in_202)*1 + 0]*dy_2[((idx_batch)*dim_2 + idx_chan_out_202)*9 + 2];
          float l_202_3 = x_0[((idx_batch)*dim_0 + idx_chan_in_202)*1 + 0]*dy_2[((idx_batch)*dim_2 + idx_chan_out_202)*9 + 3];
          float l_202_4 = x_0[((idx_batch)*dim_0 + idx_chan_in_202)*1 + 0]*dy_2[((idx_batch)*dim_2 + idx_chan_out_202)*9 + 4];
          float l_202_5 = x_0[((idx_batch)*dim_0 + idx_chan_in_202)*1 + 0]*dy_2[((idx_batch)*dim_2 + idx_chan_out_202)*9 + 5];
          float l_202_6 = x_0[((idx_batch)*dim_0 + idx_chan_in_202)*1 + 0]*dy_2[((idx_batch)*dim_2 + idx_chan_out_202)*9 + 6];
          float l_202_7 = x_0[((idx_batch)*dim_0 + idx_chan_in_202)*1 + 0]*dy_2[((idx_batch)*dim_2 + idx_chan_out_202)*9 + 7];
          float l_202_8 = x_0[((idx_batch)*dim_0 + idx_chan_in_202)*1 + 0]*dy_2[((idx_batch)*dim_2 + idx_chan_out_202)*9 + 8];
          float P_oi_202 = P_202[((idx_chan_out_202)*blockDim.y + threadIdx.y)*dim_0 + idx_chan_in_202];
          accum_202_0 += P_oi_202*l_202_0;
          accum_202_1 += P_oi_202*l_202_1;
          accum_202_2 += P_oi_202*l_202_2;
          accum_202_3 += P_oi_202*l_202_3;
          accum_202_4 += P_oi_202*l_202_4;
          accum_202_5 += P_oi_202*l_202_5;
          accum_202_6 += P_oi_202*l_202_6;
          accum_202_7 += P_oi_202*l_202_7;
          accum_202_8 += P_oi_202*l_202_8;
        }
      }
      // reduce across the warp so that first thread in warp will have the sum 
      for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
        accum_202_0 += __shfl_down_sync(0xffffffff, accum_202_0, offset);
        accum_202_1 += __shfl_down_sync(0xffffffff, accum_202_1, offset);
        accum_202_2 += __shfl_down_sync(0xffffffff, accum_202_2, offset);
        accum_202_3 += __shfl_down_sync(0xffffffff, accum_202_3, offset);
        accum_202_4 += __shfl_down_sync(0xffffffff, accum_202_4, offset);
        accum_202_5 += __shfl_down_sync(0xffffffff, accum_202_5, offset);
        accum_202_6 += __shfl_down_sync(0xffffffff, accum_202_6, offset);
        accum_202_7 += __shfl_down_sync(0xffffffff, accum_202_7, offset);
        accum_202_8 += __shfl_down_sync(0xffffffff, accum_202_8, offset);
      }
      if (threadIdx.x == 0) {
        dleft_202[((idx_batch)*dim_l + threadIdx.y)*9 + 0] = accum_202_0;
        dleft_202[((idx_batch)*dim_l + threadIdx.y)*9 + 1] = accum_202_1;
        dleft_202[((idx_batch)*dim_l + threadIdx.y)*9 + 2] = accum_202_2;
        dleft_202[((idx_batch)*dim_l + threadIdx.y)*9 + 3] = accum_202_3;
        dleft_202[((idx_batch)*dim_l + threadIdx.y)*9 + 4] = accum_202_4;
        dleft_202[((idx_batch)*dim_l + threadIdx.y)*9 + 5] = accum_202_5;
        dleft_202[((idx_batch)*dim_l + threadIdx.y)*9 + 6] = accum_202_6;
        dleft_202[((idx_batch)*dim_l + threadIdx.y)*9 + 7] = accum_202_7;
        dleft_202[((idx_batch)*dim_l + threadIdx.y)*9 + 8] = accum_202_8;
      }
      float accum_112_0 = 0.0;
      float accum_112_1 = 0.0;
      float accum_112_2 = 0.0;
      for (int idx_chan_in_112 = threadIdx.x; idx_chan_in_112 < dim_1; idx_chan_in_112 += blockDim.x) {
        for (int idx_chan_out_112 = 0; idx_chan_out_112 < dim_2; idx_chan_out_112 += 1) {
          float l_112_0 = x_1[((idx_batch)*dim_1 + idx_chan_in_112)*3 + 0]*dy_2[((idx_batch)*dim_2 + idx_chan_out_112)*9 + 0] + x_1[((idx_batch)*dim_1 + idx_chan_in_112)*3 + 1]*dy_2[((idx_batch)*dim_2 + idx_chan_out_112)*9 + 1] + x_1[((idx_batch)*dim_1 + idx_chan_in_112)*3 + 2]*dy_2[((idx_batch)*dim_2 + idx_chan_out_112)*9 + 2];
          float l_112_1 = x_1[((idx_batch)*dim_1 + idx_chan_in_112)*3 + 0]*dy_2[((idx_batch)*dim_2 + idx_chan_out_112)*9 + 3] + x_1[((idx_batch)*dim_1 + idx_chan_in_112)*3 + 1]*dy_2[((idx_batch)*dim_2 + idx_chan_out_112)*9 + 4] + x_1[((idx_batch)*dim_1 + idx_chan_in_112)*3 + 2]*dy_2[((idx_batch)*dim_2 + idx_chan_out_112)*9 + 5];
          float l_112_2 = x_1[((idx_batch)*dim_1 + idx_chan_in_112)*3 + 0]*dy_2[((idx_batch)*dim_2 + idx_chan_out_112)*9 + 6] + x_1[((idx_batch)*dim_1 + idx_chan_in_112)*3 + 1]*dy_2[((idx_batch)*dim_2 + idx_chan_out_112)*9 + 7] + x_1[((idx_batch)*dim_1 + idx_chan_in_112)*3 + 2]*dy_2[((idx_batch)*dim_2 + idx_chan_out_112)*9 + 8];
          float P_oi_112 = P_112[((idx_chan_out_112)*blockDim.y + threadIdx.y)*dim_1 + idx_chan_in_112];
          accum_112_0 += P_oi_112*l_112_0;
          accum_112_1 += P_oi_112*l_112_1;
          accum_112_2 += P_oi_112*l_112_2;
        }
      }
      // reduce across the warp so that first thread in warp will have the sum 
      for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
        accum_112_0 += __shfl_down_sync(0xffffffff, accum_112_0, offset);
        accum_112_1 += __shfl_down_sync(0xffffffff, accum_112_1, offset);
        accum_112_2 += __shfl_down_sync(0xffffffff, accum_112_2, offset);
      }
      if (threadIdx.x == 0) {
        dleft_112[((idx_batch)*dim_l + threadIdx.y)*3 + 0] = accum_112_0;
        dleft_112[((idx_batch)*dim_l + threadIdx.y)*3 + 1] = accum_112_1;
        dleft_112[((idx_batch)*dim_l + threadIdx.y)*3 + 2] = accum_112_2;
      }
      float accum_222_0 = 0.0;
      float accum_222_1 = 0.0;
      float accum_222_2 = 0.0;
      float accum_222_3 = 0.0;
      float accum_222_4 = 0.0;
      float accum_222_5 = 0.0;
      float accum_222_6 = 0.0;
      float accum_222_7 = 0.0;
      float accum_222_8 = 0.0;
      for (int idx_chan_in_222 = threadIdx.x; idx_chan_in_222 < dim_2; idx_chan_in_222 += blockDim.x) {
        for (int idx_chan_out_222 = 0; idx_chan_out_222 < dim_2; idx_chan_out_222 += 1) {
          float l_222_0 = x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 0]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 0] + x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 3]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 1] + x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 6]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 2];
          float l_222_1 = x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 1]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 0] + x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 4]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 1] + x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 7]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 2];
          float l_222_2 = x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 2]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 0] + x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 5]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 1] + x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 8]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 2];
          float l_222_3 = x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 0]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 3] + x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 3]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 4] + x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 6]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 5];
          float l_222_4 = x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 1]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 3] + x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 4]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 4] + x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 7]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 5];
          float l_222_5 = x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 2]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 3] + x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 5]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 4] + x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 8]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 5];
          float l_222_6 = x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 0]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 6] + x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 3]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 7] + x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 6]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 8];
          float l_222_7 = x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 1]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 6] + x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 4]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 7] + x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 7]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 8];
          float l_222_8 = x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 2]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 6] + x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 5]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 7] + x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 8]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 8];
          float P_oi_222 = P_222[((idx_chan_out_222)*blockDim.y + threadIdx.y)*dim_2 + idx_chan_in_222];
          accum_222_0 += P_oi_222*l_222_0;
          accum_222_1 += P_oi_222*l_222_1;
          accum_222_2 += P_oi_222*l_222_2;
          accum_222_3 += P_oi_222*l_222_3;
          accum_222_4 += P_oi_222*l_222_4;
          accum_222_5 += P_oi_222*l_222_5;
          accum_222_6 += P_oi_222*l_222_6;
          accum_222_7 += P_oi_222*l_222_7;
          accum_222_8 += P_oi_222*l_222_8;
        }
      }
      // reduce across the warp so that first thread in warp will have the sum 
      for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
        accum_222_0 += __shfl_down_sync(0xffffffff, accum_222_0, offset);
        accum_222_1 += __shfl_down_sync(0xffffffff, accum_222_1, offset);
        accum_222_2 += __shfl_down_sync(0xffffffff, accum_222_2, offset);
        accum_222_3 += __shfl_down_sync(0xffffffff, accum_222_3, offset);
        accum_222_4 += __shfl_down_sync(0xffffffff, accum_222_4, offset);
        accum_222_5 += __shfl_down_sync(0xffffffff, accum_222_5, offset);
        accum_222_6 += __shfl_down_sync(0xffffffff, accum_222_6, offset);
        accum_222_7 += __shfl_down_sync(0xffffffff, accum_222_7, offset);
        accum_222_8 += __shfl_down_sync(0xffffffff, accum_222_8, offset);
      }
      if (threadIdx.x == 0) {
        dleft_222[((idx_batch)*dim_l + threadIdx.y)*9 + 0] = accum_222_0;
        dleft_222[((idx_batch)*dim_l + threadIdx.y)*9 + 1] = accum_222_1;
        dleft_222[((idx_batch)*dim_l + threadIdx.y)*9 + 2] = accum_222_2;
        dleft_222[((idx_batch)*dim_l + threadIdx.y)*9 + 3] = accum_222_3;
        dleft_222[((idx_batch)*dim_l + threadIdx.y)*9 + 4] = accum_222_4;
        dleft_222[((idx_batch)*dim_l + threadIdx.y)*9 + 5] = accum_222_5;
        dleft_222[((idx_batch)*dim_l + threadIdx.y)*9 + 6] = accum_222_6;
        dleft_222[((idx_batch)*dim_l + threadIdx.y)*9 + 7] = accum_222_7;
        dleft_222[((idx_batch)*dim_l + threadIdx.y)*9 + 8] = accum_222_8;
      }
    }
  }
}


void ant16_o2_backleft(
    int batch, int dim_l, int dim_0, int dim_1, int dim_2,
    const float* x_0, const float* x_1, const float* x_2, const float* dy_2, const float* P_022, const float* P_202, const float* P_112, const float* P_222,
    float* dleft_022, float* dleft_202, float* dleft_112, float* dleft_222) {
  
  
  int sharedmemsz = 0;
  dim3 gridsz = dim3(batch);
  dim3 blocksz = dim3(WARPSZ, dim_l);
  ant16_o2_backleft_kern<<<gridsz, blocksz, sharedmemsz*sizeof(float)>>>(
      
      batch, dim_l, dim_0, dim_1, dim_2,
      x_0, x_1, x_2, dy_2, P_022, P_202, P_112, P_222,
      dleft_022, dleft_202, dleft_112, dleft_222);
  
}


__global__
void ant16_o2_wtsback_kern(
    // <<<(WARPSZ, WARPSZ), (WARPSZ, dim_l)>>>
    
    int batch, int dim_l, int dim_0, int dim_1, int dim_2,
    const float* x_0, const float* x_1, const float* x_2, const float* dy_2, const float* left_022, const float* left_202, const float* left_112, const float* left_222,
    float* __restrict__ dP_022, float* __restrict__ dP_202, float* __restrict__ dP_112, float* __restrict__ dP_222) {
  extern __shared__ float s[];
  for (int idx_chan_in_022 = blockIdx.x; idx_chan_in_022 < dim_2; idx_chan_in_022 += gridDim.x) {
    for (int idx_chan_out_022 = blockIdx.y; idx_chan_out_022 < dim_2; idx_chan_out_022 += gridDim.y) {
      float dP_oi = 0.0;
      for (int idx_batch = threadIdx.x; idx_batch < batch; idx_batch += blockDim.x) {
        dP_oi += (left_022[((idx_batch)*dim_l + threadIdx.y)*1 + 0]*x_2[((idx_batch)*dim_2 + idx_chan_in_022)*9 + 0]*dy_2[((idx_batch)*dim_2 + idx_chan_out_022)*9 + 0] + left_022[((idx_batch)*dim_l + threadIdx.y)*1 + 0]*x_2[((idx_batch)*dim_2 + idx_chan_in_022)*9 + 1]*dy_2[((idx_batch)*dim_2 + idx_chan_out_022)*9 + 1] + left_022[((idx_batch)*dim_l + threadIdx.y)*1 + 0]*x_2[((idx_batch)*dim_2 + idx_chan_in_022)*9 + 2]*dy_2[((idx_batch)*dim_2 + idx_chan_out_022)*9 + 2] + left_022[((idx_batch)*dim_l + threadIdx.y)*1 + 0]*x_2[((idx_batch)*dim_2 + idx_chan_in_022)*9 + 3]*dy_2[((idx_batch)*dim_2 + idx_chan_out_022)*9 + 3] + left_022[((idx_batch)*dim_l + threadIdx.y)*1 + 0]*x_2[((idx_batch)*dim_2 + idx_chan_in_022)*9 + 4]*dy_2[((idx_batch)*dim_2 + idx_chan_out_022)*9 + 4] + left_022[((idx_batch)*dim_l + threadIdx.y)*1 + 0]*x_2[((idx_batch)*dim_2 + idx_chan_in_022)*9 + 5]*dy_2[((idx_batch)*dim_2 + idx_chan_out_022)*9 + 5] + left_022[((idx_batch)*dim_l + threadIdx.y)*1 + 0]*x_2[((idx_batch)*dim_2 + idx_chan_in_022)*9 + 6]*dy_2[((idx_batch)*dim_2 + idx_chan_out_022)*9 + 6] + left_022[((idx_batch)*dim_l + threadIdx.y)*1 + 0]*x_2[((idx_batch)*dim_2 + idx_chan_in_022)*9 + 7]*dy_2[((idx_batch)*dim_2 + idx_chan_out_022)*9 + 7] + left_022[((idx_batch)*dim_l + threadIdx.y)*1 + 0]*x_2[((idx_batch)*dim_2 + idx_chan_in_022)*9 + 8]*dy_2[((idx_batch)*dim_2 + idx_chan_out_022)*9 + 8]);
      }
      // reduce across the warp so that first thread in warp will have the sum 
      for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
        dP_oi += __shfl_down_sync(0xffffffff, dP_oi, offset);
      }
      if (threadIdx.x == 0) {
        dP_022[((idx_chan_out_022)*blockDim.y + threadIdx.y)*dim_2 + idx_chan_in_022] = dP_oi;
      }
    }
  }
  for (int idx_chan_in_202 = blockIdx.x; idx_chan_in_202 < dim_0; idx_chan_in_202 += gridDim.x) {
    for (int idx_chan_out_202 = blockIdx.y; idx_chan_out_202 < dim_2; idx_chan_out_202 += gridDim.y) {
      float dP_oi = 0.0;
      for (int idx_batch = threadIdx.x; idx_batch < batch; idx_batch += blockDim.x) {
        dP_oi += (left_202[((idx_batch)*dim_l + threadIdx.y)*9 + 0]*x_0[((idx_batch)*dim_0 + idx_chan_in_202)*1 + 0]*dy_2[((idx_batch)*dim_2 + idx_chan_out_202)*9 + 0] + left_202[((idx_batch)*dim_l + threadIdx.y)*9 + 1]*x_0[((idx_batch)*dim_0 + idx_chan_in_202)*1 + 0]*dy_2[((idx_batch)*dim_2 + idx_chan_out_202)*9 + 1] + left_202[((idx_batch)*dim_l + threadIdx.y)*9 + 2]*x_0[((idx_batch)*dim_0 + idx_chan_in_202)*1 + 0]*dy_2[((idx_batch)*dim_2 + idx_chan_out_202)*9 + 2] + left_202[((idx_batch)*dim_l + threadIdx.y)*9 + 3]*x_0[((idx_batch)*dim_0 + idx_chan_in_202)*1 + 0]*dy_2[((idx_batch)*dim_2 + idx_chan_out_202)*9 + 3] + left_202[((idx_batch)*dim_l + threadIdx.y)*9 + 4]*x_0[((idx_batch)*dim_0 + idx_chan_in_202)*1 + 0]*dy_2[((idx_batch)*dim_2 + idx_chan_out_202)*9 + 4] + left_202[((idx_batch)*dim_l + threadIdx.y)*9 + 5]*x_0[((idx_batch)*dim_0 + idx_chan_in_202)*1 + 0]*dy_2[((idx_batch)*dim_2 + idx_chan_out_202)*9 + 5] + left_202[((idx_batch)*dim_l + threadIdx.y)*9 + 6]*x_0[((idx_batch)*dim_0 + idx_chan_in_202)*1 + 0]*dy_2[((idx_batch)*dim_2 + idx_chan_out_202)*9 + 6] + left_202[((idx_batch)*dim_l + threadIdx.y)*9 + 7]*x_0[((idx_batch)*dim_0 + idx_chan_in_202)*1 + 0]*dy_2[((idx_batch)*dim_2 + idx_chan_out_202)*9 + 7] + left_202[((idx_batch)*dim_l + threadIdx.y)*9 + 8]*x_0[((idx_batch)*dim_0 + idx_chan_in_202)*1 + 0]*dy_2[((idx_batch)*dim_2 + idx_chan_out_202)*9 + 8]);
      }
      // reduce across the warp so that first thread in warp will have the sum 
      for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
        dP_oi += __shfl_down_sync(0xffffffff, dP_oi, offset);
      }
      if (threadIdx.x == 0) {
        dP_202[((idx_chan_out_202)*blockDim.y + threadIdx.y)*dim_0 + idx_chan_in_202] = dP_oi;
      }
    }
  }
  for (int idx_chan_in_112 = blockIdx.x; idx_chan_in_112 < dim_1; idx_chan_in_112 += gridDim.x) {
    for (int idx_chan_out_112 = blockIdx.y; idx_chan_out_112 < dim_2; idx_chan_out_112 += gridDim.y) {
      float dP_oi = 0.0;
      for (int idx_batch = threadIdx.x; idx_batch < batch; idx_batch += blockDim.x) {
        dP_oi += (left_112[((idx_batch)*dim_l + threadIdx.y)*3 + 0]*x_1[((idx_batch)*dim_1 + idx_chan_in_112)*3 + 0]*dy_2[((idx_batch)*dim_2 + idx_chan_out_112)*9 + 0] + left_112[((idx_batch)*dim_l + threadIdx.y)*3 + 0]*x_1[((idx_batch)*dim_1 + idx_chan_in_112)*3 + 1]*dy_2[((idx_batch)*dim_2 + idx_chan_out_112)*9 + 1] + left_112[((idx_batch)*dim_l + threadIdx.y)*3 + 0]*x_1[((idx_batch)*dim_1 + idx_chan_in_112)*3 + 2]*dy_2[((idx_batch)*dim_2 + idx_chan_out_112)*9 + 2] + left_112[((idx_batch)*dim_l + threadIdx.y)*3 + 1]*x_1[((idx_batch)*dim_1 + idx_chan_in_112)*3 + 0]*dy_2[((idx_batch)*dim_2 + idx_chan_out_112)*9 + 3] + left_112[((idx_batch)*dim_l + threadIdx.y)*3 + 1]*x_1[((idx_batch)*dim_1 + idx_chan_in_112)*3 + 1]*dy_2[((idx_batch)*dim_2 + idx_chan_out_112)*9 + 4] + left_112[((idx_batch)*dim_l + threadIdx.y)*3 + 1]*x_1[((idx_batch)*dim_1 + idx_chan_in_112)*3 + 2]*dy_2[((idx_batch)*dim_2 + idx_chan_out_112)*9 + 5] + left_112[((idx_batch)*dim_l + threadIdx.y)*3 + 2]*x_1[((idx_batch)*dim_1 + idx_chan_in_112)*3 + 0]*dy_2[((idx_batch)*dim_2 + idx_chan_out_112)*9 + 6] + left_112[((idx_batch)*dim_l + threadIdx.y)*3 + 2]*x_1[((idx_batch)*dim_1 + idx_chan_in_112)*3 + 1]*dy_2[((idx_batch)*dim_2 + idx_chan_out_112)*9 + 7] + left_112[((idx_batch)*dim_l + threadIdx.y)*3 + 2]*x_1[((idx_batch)*dim_1 + idx_chan_in_112)*3 + 2]*dy_2[((idx_batch)*dim_2 + idx_chan_out_112)*9 + 8]);
      }
      // reduce across the warp so that first thread in warp will have the sum 
      for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
        dP_oi += __shfl_down_sync(0xffffffff, dP_oi, offset);
      }
      if (threadIdx.x == 0) {
        dP_112[((idx_chan_out_112)*blockDim.y + threadIdx.y)*dim_1 + idx_chan_in_112] = dP_oi;
      }
    }
  }
  for (int idx_chan_in_222 = blockIdx.x; idx_chan_in_222 < dim_2; idx_chan_in_222 += gridDim.x) {
    for (int idx_chan_out_222 = blockIdx.y; idx_chan_out_222 < dim_2; idx_chan_out_222 += gridDim.y) {
      float dP_oi = 0.0;
      for (int idx_batch = threadIdx.x; idx_batch < batch; idx_batch += blockDim.x) {
        dP_oi += (left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 0]*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 0]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 0] + left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 1]*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 1]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 0] + left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 2]*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 2]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 0] + left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 0]*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 3]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 1] + left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 1]*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 4]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 1] + left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 2]*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 5]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 1] + left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 0]*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 6]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 2] + left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 1]*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 7]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 2] + left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 2]*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 8]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 2] + left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 3]*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 0]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 3] + left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 4]*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 1]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 3] + left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 5]*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 2]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 3] + left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 3]*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 3]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 4] + left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 4]*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 4]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 4] + left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 5]*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 5]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 4] + left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 3]*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 6]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 5] + left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 4]*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 7]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 5] + left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 5]*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 8]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 5] + left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 6]*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 0]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 6] + left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 7]*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 1]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 6] + left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 8]*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 2]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 6] + left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 6]*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 3]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 7] + left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 7]*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 4]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 7] + left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 8]*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 5]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 7] + left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 6]*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 6]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 8] + left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 7]*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 7]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 8] + left_222[((idx_batch)*dim_l + threadIdx.y)*9 + 8]*x_2[((idx_batch)*dim_2 + idx_chan_in_222)*9 + 8]*dy_2[((idx_batch)*dim_2 + idx_chan_out_222)*9 + 8]);
      }
      // reduce across the warp so that first thread in warp will have the sum 
      for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
        dP_oi += __shfl_down_sync(0xffffffff, dP_oi, offset);
      }
      if (threadIdx.x == 0) {
        dP_222[((idx_chan_out_222)*blockDim.y + threadIdx.y)*dim_2 + idx_chan_in_222] = dP_oi;
      }
    }
  }
}


void ant16_o2_wtsback(
    int batch, int dim_l, int dim_0, int dim_1, int dim_2,
    const float* x_0, const float* x_1, const float* x_2, const float* dy_2, const float* left_022, const float* left_202, const float* left_112, const float* left_222,
    float* dP_022, float* dP_202, float* dP_112, float* dP_222) {
  
  
  int sharedmemsz = 0;
  dim3 gridsz = dim3(WARPSZ, WARPSZ);
  dim3 blocksz = dim3(WARPSZ, dim_l);
  ant16_o2_wtsback_kern<<<gridsz, blocksz, sharedmemsz*sizeof(float)>>>(
      
      batch, dim_l, dim_0, dim_1, dim_2,
      x_0, x_1, x_2, dy_2, left_022, left_202, left_112, left_222,
      dP_022, dP_202, dP_112, dP_222);
  
}


__global__
void ant16_oc_kern(
    // <<<(batch), (WARPSZ, dim_l)>>>
    int p_1, 
    int batch, int dim_l, int dim_1, int dim_2,
    const float* x_1, const float* x_2, const float* P_111, const float* left_111, const float* P_212, const float* left_212,
    float* __restrict__ y_1, float* __restrict__ y_2) {
  extern __shared__ float s[];
  float* product_111 = &s[0*p_1]; // size = 3*p_1
  float* product_212 = &s[3*p_1]; // size = 9*p_1
  for (int idx_batch = blockIdx.x; idx_batch < batch; idx_batch += gridDim.x) {
    { // compute tensor products
      float l_111_0 = left_111[((idx_batch)*dim_l + threadIdx.y)*3 + 0];
      float l_111_1 = left_111[((idx_batch)*dim_l + threadIdx.y)*3 + 1];
      float l_111_2 = left_111[((idx_batch)*dim_l + threadIdx.y)*3 + 2];
      for (int idx_chan_in_111 = threadIdx.x; idx_chan_in_111 < dim_1; idx_chan_in_111 += blockDim.x) {
        product_111[((threadIdx.y)*dim_1 + idx_chan_in_111)*3 + 0] = l_111_1*x_1[((idx_batch)*dim_1 + idx_chan_in_111)*3 + 2] + (-1)*l_111_2*x_1[((idx_batch)*dim_1 + idx_chan_in_111)*3 + 1];
        product_111[((threadIdx.y)*dim_1 + idx_chan_in_111)*3 + 1] = (-1)*l_111_0*x_1[((idx_batch)*dim_1 + idx_chan_in_111)*3 + 2] + l_111_2*x_1[((idx_batch)*dim_1 + idx_chan_in_111)*3 + 0];
        product_111[((threadIdx.y)*dim_1 + idx_chan_in_111)*3 + 2] = l_111_0*x_1[((idx_batch)*dim_1 + idx_chan_in_111)*3 + 1] + (-1)*l_111_1*x_1[((idx_batch)*dim_1 + idx_chan_in_111)*3 + 0];
      }
      float l_212_0 = left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 0];
      float l_212_1 = left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 1];
      float l_212_2 = left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 2];
      float l_212_3 = left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 3];
      float l_212_4 = left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 4];
      float l_212_5 = left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 5];
      float l_212_6 = left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 6];
      float l_212_7 = left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 7];
      float l_212_8 = left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 8];
      for (int idx_chan_in_212 = threadIdx.x; idx_chan_in_212 < dim_1; idx_chan_in_212 += blockDim.x) {
        product_212[((threadIdx.y)*dim_1 + idx_chan_in_212)*9 + 0] = l_212_1*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 2] + (-1)*l_212_2*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 1];
        product_212[((threadIdx.y)*dim_1 + idx_chan_in_212)*9 + 1] = (-1)*l_212_0*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 2] + l_212_2*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 0];
        product_212[((threadIdx.y)*dim_1 + idx_chan_in_212)*9 + 2] = l_212_0*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 1] + (-1)*l_212_1*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 0];
        product_212[((threadIdx.y)*dim_1 + idx_chan_in_212)*9 + 3] = l_212_4*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 2] + (-1)*l_212_5*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 1];
        product_212[((threadIdx.y)*dim_1 + idx_chan_in_212)*9 + 4] = (-1)*l_212_3*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 2] + l_212_5*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 0];
        product_212[((threadIdx.y)*dim_1 + idx_chan_in_212)*9 + 5] = l_212_3*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 1] + (-1)*l_212_4*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 0];
        product_212[((threadIdx.y)*dim_1 + idx_chan_in_212)*9 + 6] = l_212_7*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 2] + (-1)*l_212_8*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 1];
        product_212[((threadIdx.y)*dim_1 + idx_chan_in_212)*9 + 7] = (-1)*l_212_6*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 2] + l_212_8*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 0];
        product_212[((threadIdx.y)*dim_1 + idx_chan_in_212)*9 + 8] = l_212_6*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 1] + (-1)*l_212_7*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 0];
      }
    }
    __syncthreads();
    { // linear transforms to compute the outputs
      for (int idx_chan_out_1 = threadIdx.y; idx_chan_out_1 < dim_1; idx_chan_out_1 += blockDim.y) {
        float y_o_1_0 = 0.0;
        float y_o_1_1 = 0.0;
        float y_o_1_2 = 0.0;
        float accum_111_0 = 0.0;
        float accum_111_1 = 0.0;
        float accum_111_2 = 0.0;
        for (int idx_chan_in_111 = threadIdx.x; idx_chan_in_111 < dim_l*dim_1; idx_chan_in_111 += blockDim.x) {
          float P_oi_111 = P_111[(idx_chan_out_1)*dim_l*dim_1 + idx_chan_in_111];
          accum_111_0 += P_oi_111*product_111[(idx_chan_in_111)*3 + 0];
          accum_111_1 += P_oi_111*product_111[(idx_chan_in_111)*3 + 1];
          accum_111_2 += P_oi_111*product_111[(idx_chan_in_111)*3 + 2];
        }
        // reduce across the warp so that first thread in warp will have the sum 
        for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
          accum_111_0 += __shfl_down_sync(0xffffffff, accum_111_0, offset);
          accum_111_1 += __shfl_down_sync(0xffffffff, accum_111_1, offset);
          accum_111_2 += __shfl_down_sync(0xffffffff, accum_111_2, offset);
        }
        if (threadIdx.x == 0) {
          y_o_1_0 += accum_111_0;
          y_o_1_1 += accum_111_1;
          y_o_1_2 += accum_111_2;
        }
        if (threadIdx.x == 0) {
          y_1[((idx_batch)*dim_1 + idx_chan_out_1)*3 + 0] = y_o_1_0;
          y_1[((idx_batch)*dim_1 + idx_chan_out_1)*3 + 1] = y_o_1_1;
          y_1[((idx_batch)*dim_1 + idx_chan_out_1)*3 + 2] = y_o_1_2;
        }
      }
      for (int idx_chan_out_2 = threadIdx.y; idx_chan_out_2 < dim_2; idx_chan_out_2 += blockDim.y) {
        float y_o_2_0 = 0.0;
        float y_o_2_1 = 0.0;
        float y_o_2_2 = 0.0;
        float y_o_2_3 = 0.0;
        float y_o_2_4 = 0.0;
        float y_o_2_5 = 0.0;
        float y_o_2_6 = 0.0;
        float y_o_2_7 = 0.0;
        float y_o_2_8 = 0.0;
        float accum_212_0 = 0.0;
        float accum_212_1 = 0.0;
        float accum_212_2 = 0.0;
        float accum_212_3 = 0.0;
        float accum_212_4 = 0.0;
        float accum_212_5 = 0.0;
        float accum_212_6 = 0.0;
        float accum_212_7 = 0.0;
        float accum_212_8 = 0.0;
        for (int idx_chan_in_212 = threadIdx.x; idx_chan_in_212 < dim_l*dim_1; idx_chan_in_212 += blockDim.x) {
          float P_oi_212 = P_212[(idx_chan_out_2)*dim_l*dim_1 + idx_chan_in_212];
          accum_212_0 += P_oi_212*product_212[(idx_chan_in_212)*9 + 0];
          accum_212_1 += P_oi_212*product_212[(idx_chan_in_212)*9 + 1];
          accum_212_2 += P_oi_212*product_212[(idx_chan_in_212)*9 + 2];
          accum_212_3 += P_oi_212*product_212[(idx_chan_in_212)*9 + 3];
          accum_212_4 += P_oi_212*product_212[(idx_chan_in_212)*9 + 4];
          accum_212_5 += P_oi_212*product_212[(idx_chan_in_212)*9 + 5];
          accum_212_6 += P_oi_212*product_212[(idx_chan_in_212)*9 + 6];
          accum_212_7 += P_oi_212*product_212[(idx_chan_in_212)*9 + 7];
          accum_212_8 += P_oi_212*product_212[(idx_chan_in_212)*9 + 8];
        }
        // reduce across the warp so that first thread in warp will have the sum 
        for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
          accum_212_0 += __shfl_down_sync(0xffffffff, accum_212_0, offset);
          accum_212_1 += __shfl_down_sync(0xffffffff, accum_212_1, offset);
          accum_212_2 += __shfl_down_sync(0xffffffff, accum_212_2, offset);
          accum_212_3 += __shfl_down_sync(0xffffffff, accum_212_3, offset);
          accum_212_4 += __shfl_down_sync(0xffffffff, accum_212_4, offset);
          accum_212_5 += __shfl_down_sync(0xffffffff, accum_212_5, offset);
          accum_212_6 += __shfl_down_sync(0xffffffff, accum_212_6, offset);
          accum_212_7 += __shfl_down_sync(0xffffffff, accum_212_7, offset);
          accum_212_8 += __shfl_down_sync(0xffffffff, accum_212_8, offset);
        }
        if (threadIdx.x == 0) {
          y_o_2_0 += accum_212_0;
          y_o_2_1 += accum_212_1;
          y_o_2_2 += accum_212_2;
          y_o_2_3 += accum_212_3;
          y_o_2_4 += accum_212_4;
          y_o_2_5 += accum_212_5;
          y_o_2_6 += accum_212_6;
          y_o_2_7 += accum_212_7;
          y_o_2_8 += accum_212_8;
        }
        if (threadIdx.x == 0) {
          y_2[((idx_batch)*dim_2 + idx_chan_out_2)*9 + 0] = y_o_2_0;
          y_2[((idx_batch)*dim_2 + idx_chan_out_2)*9 + 1] = y_o_2_1;
          y_2[((idx_batch)*dim_2 + idx_chan_out_2)*9 + 2] = y_o_2_2;
          y_2[((idx_batch)*dim_2 + idx_chan_out_2)*9 + 3] = y_o_2_3;
          y_2[((idx_batch)*dim_2 + idx_chan_out_2)*9 + 4] = y_o_2_4;
          y_2[((idx_batch)*dim_2 + idx_chan_out_2)*9 + 5] = y_o_2_5;
          y_2[((idx_batch)*dim_2 + idx_chan_out_2)*9 + 6] = y_o_2_6;
          y_2[((idx_batch)*dim_2 + idx_chan_out_2)*9 + 7] = y_o_2_7;
          y_2[((idx_batch)*dim_2 + idx_chan_out_2)*9 + 8] = y_o_2_8;
        }
      }
    }
  }
}


void ant16_oc(
    int batch, int dim_l, int dim_1, int dim_2,
    const float* x_1, const float* x_2, const float* P_111, const float* left_111, const float* P_212, const float* left_212,
    float* y_1, float* y_2) {
  
  int p_1 = dim_l*dim_1;
  int sharedmemsz = 0;
  sharedmemsz += 12*p_1;
  dim3 gridsz = dim3(batch);
  dim3 blocksz = dim3(WARPSZ, dim_l);
  ant16_oc_kern<<<gridsz, blocksz, sharedmemsz*sizeof(float)>>>(
      p_1, 
      batch, dim_l, dim_1, dim_2,
      x_1, x_2, P_111, left_111, P_212, left_212,
      y_1, y_2);
  
}


__global__
void ant16_oc_backward_kern(
    // <<<(batch), (WARPSZ, dim_l)>>>
    int p_1, int p_2_base, int p_2, 
    int batch, int dim_l, int dim_1, int dim_2,
    const float* dy_1, const float* dy_2, const float* P_111, const float* left_111, const float* P_212, const float* left_212,
    float* __restrict__ dx_1, float* __restrict__ dx_2) {
  extern __shared__ float s[];
  float* dproduct_111 = &s[0*p_1]; // size = 3*p_1
  float* dproduct_212 = &s[p_2_base + 0*p_2]; // size = 3*p_2
  for (int idx_batch = blockIdx.x; idx_batch < batch; idx_batch += gridDim.x) {
    { // compute tensor products
      float l_111_0 = left_111[((idx_batch)*dim_l + threadIdx.y)*3 + 0];
      float l_111_1 = left_111[((idx_batch)*dim_l + threadIdx.y)*3 + 1];
      float l_111_2 = left_111[((idx_batch)*dim_l + threadIdx.y)*3 + 2];
      for (int idx_chan_out_111 = threadIdx.x; idx_chan_out_111 < dim_1; idx_chan_out_111 += blockDim.x) {
        dproduct_111[((threadIdx.y)*dim_1 + idx_chan_out_111)*3 + 0] = l_111_2*dy_1[((idx_batch)*dim_1 + idx_chan_out_111)*3 + 1] + (-1)*l_111_1*dy_1[((idx_batch)*dim_1 + idx_chan_out_111)*3 + 2];
        dproduct_111[((threadIdx.y)*dim_1 + idx_chan_out_111)*3 + 1] = (-1)*l_111_2*dy_1[((idx_batch)*dim_1 + idx_chan_out_111)*3 + 0] + l_111_0*dy_1[((idx_batch)*dim_1 + idx_chan_out_111)*3 + 2];
        dproduct_111[((threadIdx.y)*dim_1 + idx_chan_out_111)*3 + 2] = l_111_1*dy_1[((idx_batch)*dim_1 + idx_chan_out_111)*3 + 0] + (-1)*l_111_0*dy_1[((idx_batch)*dim_1 + idx_chan_out_111)*3 + 1];
      }
      float l_212_0 = left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 0];
      float l_212_1 = left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 1];
      float l_212_2 = left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 2];
      float l_212_3 = left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 3];
      float l_212_4 = left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 4];
      float l_212_5 = left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 5];
      float l_212_6 = left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 6];
      float l_212_7 = left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 7];
      float l_212_8 = left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 8];
      for (int idx_chan_out_212 = threadIdx.x; idx_chan_out_212 < dim_2; idx_chan_out_212 += blockDim.x) {
        dproduct_212[((threadIdx.y)*dim_2 + idx_chan_out_212)*3 + 0] = l_212_2*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 1] + (-1)*l_212_1*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 2] + l_212_5*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 4] + (-1)*l_212_4*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 5] + l_212_8*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 7] + (-1)*l_212_7*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 8];
        dproduct_212[((threadIdx.y)*dim_2 + idx_chan_out_212)*3 + 1] = (-1)*l_212_2*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 0] + l_212_0*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 2] + (-1)*l_212_5*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 3] + l_212_3*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 5] + (-1)*l_212_8*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 6] + l_212_6*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 8];
        dproduct_212[((threadIdx.y)*dim_2 + idx_chan_out_212)*3 + 2] = l_212_1*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 0] + (-1)*l_212_0*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 1] + l_212_4*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 3] + (-1)*l_212_3*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 4] + l_212_7*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 6] + (-1)*l_212_6*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 7];
      }
    }
    __syncthreads();
    { // linear transforms to compute dx
      for (int idx_chan_in_1 = threadIdx.y; idx_chan_in_1 < dim_1; idx_chan_in_1 += blockDim.y) {
        float dx_o_1_0 = 0.0;
        float dx_o_1_1 = 0.0;
        float dx_o_1_2 = 0.0;
        float accum_111_0 = 0.0;
        float accum_111_1 = 0.0;
        float accum_111_2 = 0.0;
        for (int idx_l_111 = 0; idx_l_111 < dim_l; idx_l_111 += 1) {
          for (int idx_chan_out_111 = threadIdx.x; idx_chan_out_111 < dim_1; idx_chan_out_111 += blockDim.x) {
            float P_oi_111 = P_111[((idx_chan_out_111)*dim_l + idx_l_111)*dim_1 + idx_chan_in_1];
            accum_111_0 += P_oi_111*dproduct_111[((idx_l_111)*dim_1 + idx_chan_out_111)*3 + 0];
            accum_111_1 += P_oi_111*dproduct_111[((idx_l_111)*dim_1 + idx_chan_out_111)*3 + 1];
            accum_111_2 += P_oi_111*dproduct_111[((idx_l_111)*dim_1 + idx_chan_out_111)*3 + 2];
          }
        }
        // reduce across the warp so that first thread in warp will have the sum 
        for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
          accum_111_0 += __shfl_down_sync(0xffffffff, accum_111_0, offset);
          accum_111_1 += __shfl_down_sync(0xffffffff, accum_111_1, offset);
          accum_111_2 += __shfl_down_sync(0xffffffff, accum_111_2, offset);
        }
        if (threadIdx.x == 0) {
          dx_o_1_0 += accum_111_0;
          dx_o_1_1 += accum_111_1;
          dx_o_1_2 += accum_111_2;
        }
        float accum_212_0 = 0.0;
        float accum_212_1 = 0.0;
        float accum_212_2 = 0.0;
        for (int idx_l_212 = 0; idx_l_212 < dim_l; idx_l_212 += 1) {
          for (int idx_chan_out_212 = threadIdx.x; idx_chan_out_212 < dim_2; idx_chan_out_212 += blockDim.x) {
            float P_oi_212 = P_212[((idx_chan_out_212)*dim_l + idx_l_212)*dim_1 + idx_chan_in_1];
            accum_212_0 += P_oi_212*dproduct_212[((idx_l_212)*dim_2 + idx_chan_out_212)*3 + 0];
            accum_212_1 += P_oi_212*dproduct_212[((idx_l_212)*dim_2 + idx_chan_out_212)*3 + 1];
            accum_212_2 += P_oi_212*dproduct_212[((idx_l_212)*dim_2 + idx_chan_out_212)*3 + 2];
          }
        }
        // reduce across the warp so that first thread in warp will have the sum 
        for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
          accum_212_0 += __shfl_down_sync(0xffffffff, accum_212_0, offset);
          accum_212_1 += __shfl_down_sync(0xffffffff, accum_212_1, offset);
          accum_212_2 += __shfl_down_sync(0xffffffff, accum_212_2, offset);
        }
        if (threadIdx.x == 0) {
          dx_o_1_0 += accum_212_0;
          dx_o_1_1 += accum_212_1;
          dx_o_1_2 += accum_212_2;
        }
        if (threadIdx.x == 0) {
          dx_1[((idx_batch)*dim_1 + idx_chan_in_1)*3 + 0] = dx_o_1_0;
          dx_1[((idx_batch)*dim_1 + idx_chan_in_1)*3 + 1] = dx_o_1_1;
          dx_1[((idx_batch)*dim_1 + idx_chan_in_1)*3 + 2] = dx_o_1_2;
        }
      }
      for (int idx_chan_in_2 = threadIdx.y; idx_chan_in_2 < dim_2; idx_chan_in_2 += blockDim.y) {
        float dx_o_2_0 = 0.0;
        float dx_o_2_1 = 0.0;
        float dx_o_2_2 = 0.0;
        float dx_o_2_3 = 0.0;
        float dx_o_2_4 = 0.0;
        float dx_o_2_5 = 0.0;
        float dx_o_2_6 = 0.0;
        float dx_o_2_7 = 0.0;
        float dx_o_2_8 = 0.0;
        if (threadIdx.x == 0) {
          dx_2[((idx_batch)*dim_2 + idx_chan_in_2)*9 + 0] = dx_o_2_0;
          dx_2[((idx_batch)*dim_2 + idx_chan_in_2)*9 + 1] = dx_o_2_1;
          dx_2[((idx_batch)*dim_2 + idx_chan_in_2)*9 + 2] = dx_o_2_2;
          dx_2[((idx_batch)*dim_2 + idx_chan_in_2)*9 + 3] = dx_o_2_3;
          dx_2[((idx_batch)*dim_2 + idx_chan_in_2)*9 + 4] = dx_o_2_4;
          dx_2[((idx_batch)*dim_2 + idx_chan_in_2)*9 + 5] = dx_o_2_5;
          dx_2[((idx_batch)*dim_2 + idx_chan_in_2)*9 + 6] = dx_o_2_6;
          dx_2[((idx_batch)*dim_2 + idx_chan_in_2)*9 + 7] = dx_o_2_7;
          dx_2[((idx_batch)*dim_2 + idx_chan_in_2)*9 + 8] = dx_o_2_8;
        }
      }
    }
  }
}


void ant16_oc_backward(
    int batch, int dim_l, int dim_1, int dim_2,
    const float* dy_1, const float* dy_2, const float* P_111, const float* left_111, const float* P_212, const float* left_212,
    float* dx_1, float* dx_2) {
  
  int p_1 = dim_l*dim_1;
  int p_2 = dim_l*dim_2;
  int sharedmemsz = 0;
  sharedmemsz += 3*p_1;
  int p_2_base = sharedmemsz;
  sharedmemsz += 3*p_2;
  dim3 gridsz = dim3(batch);
  dim3 blocksz = dim3(WARPSZ, dim_l);
  ant16_oc_backward_kern<<<gridsz, blocksz, sharedmemsz*sizeof(float)>>>(
      p_1, p_2_base, p_2, 
      batch, dim_l, dim_1, dim_2,
      dy_1, dy_2, P_111, left_111, P_212, left_212,
      dx_1, dx_2);
  
}


__global__
void ant16_oc_backleft_kern(
    // <<<(batch), (WARPSZ, dim_l)>>>
    
    int batch, int dim_l, int dim_1, int dim_2,
    const float* x_1, const float* x_2, const float* dy_1, const float* dy_2, const float* P_111, const float* P_212,
    float* __restrict__ dleft_111, float* __restrict__ dleft_212) {
  extern __shared__ float s[];
  for (int idx_batch = blockIdx.x; idx_batch < batch; idx_batch += gridDim.x) {
    { // compute left derivative tensor products
      float accum_111_0 = 0.0;
      float accum_111_1 = 0.0;
      float accum_111_2 = 0.0;
      for (int idx_chan_in_111 = threadIdx.x; idx_chan_in_111 < dim_1; idx_chan_in_111 += blockDim.x) {
        for (int idx_chan_out_111 = 0; idx_chan_out_111 < dim_1; idx_chan_out_111 += 1) {
          float l_111_0 = (-1)*x_1[((idx_batch)*dim_1 + idx_chan_in_111)*3 + 2]*dy_1[((idx_batch)*dim_1 + idx_chan_out_111)*3 + 1] + x_1[((idx_batch)*dim_1 + idx_chan_in_111)*3 + 1]*dy_1[((idx_batch)*dim_1 + idx_chan_out_111)*3 + 2];
          float l_111_1 = x_1[((idx_batch)*dim_1 + idx_chan_in_111)*3 + 2]*dy_1[((idx_batch)*dim_1 + idx_chan_out_111)*3 + 0] + (-1)*x_1[((idx_batch)*dim_1 + idx_chan_in_111)*3 + 0]*dy_1[((idx_batch)*dim_1 + idx_chan_out_111)*3 + 2];
          float l_111_2 = (-1)*x_1[((idx_batch)*dim_1 + idx_chan_in_111)*3 + 1]*dy_1[((idx_batch)*dim_1 + idx_chan_out_111)*3 + 0] + x_1[((idx_batch)*dim_1 + idx_chan_in_111)*3 + 0]*dy_1[((idx_batch)*dim_1 + idx_chan_out_111)*3 + 1];
          float P_oi_111 = P_111[((idx_chan_out_111)*blockDim.y + threadIdx.y)*dim_1 + idx_chan_in_111];
          accum_111_0 += P_oi_111*l_111_0;
          accum_111_1 += P_oi_111*l_111_1;
          accum_111_2 += P_oi_111*l_111_2;
        }
      }
      // reduce across the warp so that first thread in warp will have the sum 
      for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
        accum_111_0 += __shfl_down_sync(0xffffffff, accum_111_0, offset);
        accum_111_1 += __shfl_down_sync(0xffffffff, accum_111_1, offset);
        accum_111_2 += __shfl_down_sync(0xffffffff, accum_111_2, offset);
      }
      if (threadIdx.x == 0) {
        dleft_111[((idx_batch)*dim_l + threadIdx.y)*3 + 0] = accum_111_0;
        dleft_111[((idx_batch)*dim_l + threadIdx.y)*3 + 1] = accum_111_1;
        dleft_111[((idx_batch)*dim_l + threadIdx.y)*3 + 2] = accum_111_2;
      }
      float accum_212_0 = 0.0;
      float accum_212_1 = 0.0;
      float accum_212_2 = 0.0;
      float accum_212_3 = 0.0;
      float accum_212_4 = 0.0;
      float accum_212_5 = 0.0;
      float accum_212_6 = 0.0;
      float accum_212_7 = 0.0;
      float accum_212_8 = 0.0;
      for (int idx_chan_in_212 = threadIdx.x; idx_chan_in_212 < dim_1; idx_chan_in_212 += blockDim.x) {
        for (int idx_chan_out_212 = 0; idx_chan_out_212 < dim_2; idx_chan_out_212 += 1) {
          float l_212_0 = (-1)*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 2]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 1] + x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 1]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 2];
          float l_212_1 = x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 2]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 0] + (-1)*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 0]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 2];
          float l_212_2 = (-1)*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 1]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 0] + x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 0]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 1];
          float l_212_3 = (-1)*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 2]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 4] + x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 1]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 5];
          float l_212_4 = x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 2]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 3] + (-1)*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 0]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 5];
          float l_212_5 = (-1)*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 1]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 3] + x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 0]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 4];
          float l_212_6 = (-1)*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 2]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 7] + x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 1]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 8];
          float l_212_7 = x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 2]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 6] + (-1)*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 0]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 8];
          float l_212_8 = (-1)*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 1]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 6] + x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 0]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 7];
          float P_oi_212 = P_212[((idx_chan_out_212)*blockDim.y + threadIdx.y)*dim_1 + idx_chan_in_212];
          accum_212_0 += P_oi_212*l_212_0;
          accum_212_1 += P_oi_212*l_212_1;
          accum_212_2 += P_oi_212*l_212_2;
          accum_212_3 += P_oi_212*l_212_3;
          accum_212_4 += P_oi_212*l_212_4;
          accum_212_5 += P_oi_212*l_212_5;
          accum_212_6 += P_oi_212*l_212_6;
          accum_212_7 += P_oi_212*l_212_7;
          accum_212_8 += P_oi_212*l_212_8;
        }
      }
      // reduce across the warp so that first thread in warp will have the sum 
      for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
        accum_212_0 += __shfl_down_sync(0xffffffff, accum_212_0, offset);
        accum_212_1 += __shfl_down_sync(0xffffffff, accum_212_1, offset);
        accum_212_2 += __shfl_down_sync(0xffffffff, accum_212_2, offset);
        accum_212_3 += __shfl_down_sync(0xffffffff, accum_212_3, offset);
        accum_212_4 += __shfl_down_sync(0xffffffff, accum_212_4, offset);
        accum_212_5 += __shfl_down_sync(0xffffffff, accum_212_5, offset);
        accum_212_6 += __shfl_down_sync(0xffffffff, accum_212_6, offset);
        accum_212_7 += __shfl_down_sync(0xffffffff, accum_212_7, offset);
        accum_212_8 += __shfl_down_sync(0xffffffff, accum_212_8, offset);
      }
      if (threadIdx.x == 0) {
        dleft_212[((idx_batch)*dim_l + threadIdx.y)*9 + 0] = accum_212_0;
        dleft_212[((idx_batch)*dim_l + threadIdx.y)*9 + 1] = accum_212_1;
        dleft_212[((idx_batch)*dim_l + threadIdx.y)*9 + 2] = accum_212_2;
        dleft_212[((idx_batch)*dim_l + threadIdx.y)*9 + 3] = accum_212_3;
        dleft_212[((idx_batch)*dim_l + threadIdx.y)*9 + 4] = accum_212_4;
        dleft_212[((idx_batch)*dim_l + threadIdx.y)*9 + 5] = accum_212_5;
        dleft_212[((idx_batch)*dim_l + threadIdx.y)*9 + 6] = accum_212_6;
        dleft_212[((idx_batch)*dim_l + threadIdx.y)*9 + 7] = accum_212_7;
        dleft_212[((idx_batch)*dim_l + threadIdx.y)*9 + 8] = accum_212_8;
      }
    }
  }
}


void ant16_oc_backleft(
    int batch, int dim_l, int dim_1, int dim_2,
    const float* x_1, const float* x_2, const float* dy_1, const float* dy_2, const float* P_111, const float* P_212,
    float* dleft_111, float* dleft_212) {
  
  
  int sharedmemsz = 0;
  dim3 gridsz = dim3(batch);
  dim3 blocksz = dim3(WARPSZ, dim_l);
  ant16_oc_backleft_kern<<<gridsz, blocksz, sharedmemsz*sizeof(float)>>>(
      
      batch, dim_l, dim_1, dim_2,
      x_1, x_2, dy_1, dy_2, P_111, P_212,
      dleft_111, dleft_212);
  
}


__global__
void ant16_oc_wtsback_kern(
    // <<<(WARPSZ, WARPSZ), (WARPSZ, dim_l)>>>
    
    int batch, int dim_l, int dim_1, int dim_2,
    const float* x_1, const float* x_2, const float* dy_1, const float* dy_2, const float* left_111, const float* left_212,
    float* __restrict__ dP_111, float* __restrict__ dP_212) {
  extern __shared__ float s[];
  for (int idx_chan_in_111 = blockIdx.x; idx_chan_in_111 < dim_1; idx_chan_in_111 += gridDim.x) {
    for (int idx_chan_out_111 = blockIdx.y; idx_chan_out_111 < dim_1; idx_chan_out_111 += gridDim.y) {
      float dP_oi = 0.0;
      for (int idx_batch = threadIdx.x; idx_batch < batch; idx_batch += blockDim.x) {
        dP_oi += (left_111[((idx_batch)*dim_l + threadIdx.y)*3 + 1]*x_1[((idx_batch)*dim_1 + idx_chan_in_111)*3 + 2]*dy_1[((idx_batch)*dim_1 + idx_chan_out_111)*3 + 0] + (-1)*left_111[((idx_batch)*dim_l + threadIdx.y)*3 + 2]*x_1[((idx_batch)*dim_1 + idx_chan_in_111)*3 + 1]*dy_1[((idx_batch)*dim_1 + idx_chan_out_111)*3 + 0] + (-1)*left_111[((idx_batch)*dim_l + threadIdx.y)*3 + 0]*x_1[((idx_batch)*dim_1 + idx_chan_in_111)*3 + 2]*dy_1[((idx_batch)*dim_1 + idx_chan_out_111)*3 + 1] + left_111[((idx_batch)*dim_l + threadIdx.y)*3 + 2]*x_1[((idx_batch)*dim_1 + idx_chan_in_111)*3 + 0]*dy_1[((idx_batch)*dim_1 + idx_chan_out_111)*3 + 1] + left_111[((idx_batch)*dim_l + threadIdx.y)*3 + 0]*x_1[((idx_batch)*dim_1 + idx_chan_in_111)*3 + 1]*dy_1[((idx_batch)*dim_1 + idx_chan_out_111)*3 + 2] + (-1)*left_111[((idx_batch)*dim_l + threadIdx.y)*3 + 1]*x_1[((idx_batch)*dim_1 + idx_chan_in_111)*3 + 0]*dy_1[((idx_batch)*dim_1 + idx_chan_out_111)*3 + 2]);
      }
      // reduce across the warp so that first thread in warp will have the sum 
      for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
        dP_oi += __shfl_down_sync(0xffffffff, dP_oi, offset);
      }
      if (threadIdx.x == 0) {
        dP_111[((idx_chan_out_111)*blockDim.y + threadIdx.y)*dim_1 + idx_chan_in_111] = dP_oi;
      }
    }
  }
  for (int idx_chan_in_212 = blockIdx.x; idx_chan_in_212 < dim_1; idx_chan_in_212 += gridDim.x) {
    for (int idx_chan_out_212 = blockIdx.y; idx_chan_out_212 < dim_2; idx_chan_out_212 += gridDim.y) {
      float dP_oi = 0.0;
      for (int idx_batch = threadIdx.x; idx_batch < batch; idx_batch += blockDim.x) {
        dP_oi += (left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 1]*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 2]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 0] + (-1)*left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 2]*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 1]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 0] + (-1)*left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 0]*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 2]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 1] + left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 2]*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 0]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 1] + left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 0]*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 1]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 2] + (-1)*left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 1]*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 0]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 2] + left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 4]*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 2]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 3] + (-1)*left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 5]*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 1]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 3] + (-1)*left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 3]*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 2]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 4] + left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 5]*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 0]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 4] + left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 3]*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 1]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 5] + (-1)*left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 4]*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 0]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 5] + left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 7]*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 2]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 6] + (-1)*left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 8]*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 1]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 6] + (-1)*left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 6]*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 2]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 7] + left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 8]*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 0]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 7] + left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 6]*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 1]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 8] + (-1)*left_212[((idx_batch)*dim_l + threadIdx.y)*9 + 7]*x_1[((idx_batch)*dim_1 + idx_chan_in_212)*3 + 0]*dy_2[((idx_batch)*dim_2 + idx_chan_out_212)*9 + 8]);
      }
      // reduce across the warp so that first thread in warp will have the sum 
      for (int offset = WARPSZ/2; offset >= 1; offset >>= 1) {
        dP_oi += __shfl_down_sync(0xffffffff, dP_oi, offset);
      }
      if (threadIdx.x == 0) {
        dP_212[((idx_chan_out_212)*blockDim.y + threadIdx.y)*dim_1 + idx_chan_in_212] = dP_oi;
      }
    }
  }
}


void ant16_oc_wtsback(
    int batch, int dim_l, int dim_1, int dim_2,
    const float* x_1, const float* x_2, const float* dy_1, const float* dy_2, const float* left_111, const float* left_212,
    float* dP_111, float* dP_212) {
  
  
  int sharedmemsz = 0;
  dim3 gridsz = dim3(WARPSZ, WARPSZ);
  dim3 blocksz = dim3(WARPSZ, dim_l);
  ant16_oc_wtsback_kern<<<gridsz, blocksz, sharedmemsz*sizeof(float)>>>(
      
      batch, dim_l, dim_1, dim_2,
      x_1, x_2, dy_1, dy_2, left_111, left_212,
      dP_111, dP_212);
  
}


__global__
void bee_fwd_kern(
    // <<<(batch), (WARPSZ)>>>
    
    int batch, int chan,
    const float* l_0, const float* l_1, const float* l_2, const float* r_0, const float* r_1, const float* r_2,
    float* __restrict__ y_000, float* __restrict__ y_110, float* __restrict__ y_220, float* __restrict__ y_011, float* __restrict__ y_101, float* __restrict__ y_121, float* __restrict__ y_211, float* __restrict__ y_022, float* __restrict__ y_202, float* __restrict__ y_112, float* __restrict__ y_222, float* __restrict__ y_111, float* __restrict__ y_212) {
  extern __shared__ float s[];
  for (int idx_batch = blockIdx.x; idx_batch < batch; idx_batch += gridDim.x) {
    for (int idx_chan = threadIdx.x; idx_chan < chan; idx_chan += blockDim.x) {
      y_000[((idx_batch)*chan + idx_chan)*1 + 0] = l_0[((idx_batch)*chan + idx_chan)*1 + 0]*r_0[((idx_batch)*chan + idx_chan)*1 + 0];
      y_110[((idx_batch)*chan + idx_chan)*1 + 0] = l_1[((idx_batch)*chan + idx_chan)*3 + 0]*r_1[((idx_batch)*chan + idx_chan)*3 + 0] + l_1[((idx_batch)*chan + idx_chan)*3 + 1]*r_1[((idx_batch)*chan + idx_chan)*3 + 1] + l_1[((idx_batch)*chan + idx_chan)*3 + 2]*r_1[((idx_batch)*chan + idx_chan)*3 + 2];
      y_220[((idx_batch)*chan + idx_chan)*1 + 0] = l_2[((idx_batch)*chan + idx_chan)*9 + 0]*r_2[((idx_batch)*chan + idx_chan)*9 + 0] + l_2[((idx_batch)*chan + idx_chan)*9 + 1]*r_2[((idx_batch)*chan + idx_chan)*9 + 1] + l_2[((idx_batch)*chan + idx_chan)*9 + 2]*r_2[((idx_batch)*chan + idx_chan)*9 + 2] + l_2[((idx_batch)*chan + idx_chan)*9 + 3]*r_2[((idx_batch)*chan + idx_chan)*9 + 3] + l_2[((idx_batch)*chan + idx_chan)*9 + 4]*r_2[((idx_batch)*chan + idx_chan)*9 + 4] + l_2[((idx_batch)*chan + idx_chan)*9 + 5]*r_2[((idx_batch)*chan + idx_chan)*9 + 5] + l_2[((idx_batch)*chan + idx_chan)*9 + 6]*r_2[((idx_batch)*chan + idx_chan)*9 + 6] + l_2[((idx_batch)*chan + idx_chan)*9 + 7]*r_2[((idx_batch)*chan + idx_chan)*9 + 7] + l_2[((idx_batch)*chan + idx_chan)*9 + 8]*r_2[((idx_batch)*chan + idx_chan)*9 + 8];
      y_011[((idx_batch)*chan + idx_chan)*3 + 0] = l_0[((idx_batch)*chan + idx_chan)*1 + 0]*r_1[((idx_batch)*chan + idx_chan)*3 + 0];
      y_011[((idx_batch)*chan + idx_chan)*3 + 1] = l_0[((idx_batch)*chan + idx_chan)*1 + 0]*r_1[((idx_batch)*chan + idx_chan)*3 + 1];
      y_011[((idx_batch)*chan + idx_chan)*3 + 2] = l_0[((idx_batch)*chan + idx_chan)*1 + 0]*r_1[((idx_batch)*chan + idx_chan)*3 + 2];
      y_101[((idx_batch)*chan + idx_chan)*3 + 0] = l_1[((idx_batch)*chan + idx_chan)*3 + 0]*r_0[((idx_batch)*chan + idx_chan)*1 + 0];
      y_101[((idx_batch)*chan + idx_chan)*3 + 1] = l_1[((idx_batch)*chan + idx_chan)*3 + 1]*r_0[((idx_batch)*chan + idx_chan)*1 + 0];
      y_101[((idx_batch)*chan + idx_chan)*3 + 2] = l_1[((idx_batch)*chan + idx_chan)*3 + 2]*r_0[((idx_batch)*chan + idx_chan)*1 + 0];
      y_121[((idx_batch)*chan + idx_chan)*3 + 0] = l_1[((idx_batch)*chan + idx_chan)*3 + 0]*r_2[((idx_batch)*chan + idx_chan)*9 + 0] + l_1[((idx_batch)*chan + idx_chan)*3 + 1]*r_2[((idx_batch)*chan + idx_chan)*9 + 1] + l_1[((idx_batch)*chan + idx_chan)*3 + 2]*r_2[((idx_batch)*chan + idx_chan)*9 + 2];
      y_121[((idx_batch)*chan + idx_chan)*3 + 1] = l_1[((idx_batch)*chan + idx_chan)*3 + 0]*r_2[((idx_batch)*chan + idx_chan)*9 + 3] + l_1[((idx_batch)*chan + idx_chan)*3 + 1]*r_2[((idx_batch)*chan + idx_chan)*9 + 4] + l_1[((idx_batch)*chan + idx_chan)*3 + 2]*r_2[((idx_batch)*chan + idx_chan)*9 + 5];
      y_121[((idx_batch)*chan + idx_chan)*3 + 2] = l_1[((idx_batch)*chan + idx_chan)*3 + 0]*r_2[((idx_batch)*chan + idx_chan)*9 + 6] + l_1[((idx_batch)*chan + idx_chan)*3 + 1]*r_2[((idx_batch)*chan + idx_chan)*9 + 7] + l_1[((idx_batch)*chan + idx_chan)*3 + 2]*r_2[((idx_batch)*chan + idx_chan)*9 + 8];
      y_211[((idx_batch)*chan + idx_chan)*3 + 0] = l_2[((idx_batch)*chan + idx_chan)*9 + 0]*r_1[((idx_batch)*chan + idx_chan)*3 + 0] + l_2[((idx_batch)*chan + idx_chan)*9 + 1]*r_1[((idx_batch)*chan + idx_chan)*3 + 1] + l_2[((idx_batch)*chan + idx_chan)*9 + 2]*r_1[((idx_batch)*chan + idx_chan)*3 + 2];
      y_211[((idx_batch)*chan + idx_chan)*3 + 1] = l_2[((idx_batch)*chan + idx_chan)*9 + 3]*r_1[((idx_batch)*chan + idx_chan)*3 + 0] + l_2[((idx_batch)*chan + idx_chan)*9 + 4]*r_1[((idx_batch)*chan + idx_chan)*3 + 1] + l_2[((idx_batch)*chan + idx_chan)*9 + 5]*r_1[((idx_batch)*chan + idx_chan)*3 + 2];
      y_211[((idx_batch)*chan + idx_chan)*3 + 2] = l_2[((idx_batch)*chan + idx_chan)*9 + 6]*r_1[((idx_batch)*chan + idx_chan)*3 + 0] + l_2[((idx_batch)*chan + idx_chan)*9 + 7]*r_1[((idx_batch)*chan + idx_chan)*3 + 1] + l_2[((idx_batch)*chan + idx_chan)*9 + 8]*r_1[((idx_batch)*chan + idx_chan)*3 + 2];
      y_022[((idx_batch)*chan + idx_chan)*9 + 0] = l_0[((idx_batch)*chan + idx_chan)*1 + 0]*r_2[((idx_batch)*chan + idx_chan)*9 + 0];
      y_022[((idx_batch)*chan + idx_chan)*9 + 1] = l_0[((idx_batch)*chan + idx_chan)*1 + 0]*r_2[((idx_batch)*chan + idx_chan)*9 + 1];
      y_022[((idx_batch)*chan + idx_chan)*9 + 2] = l_0[((idx_batch)*chan + idx_chan)*1 + 0]*r_2[((idx_batch)*chan + idx_chan)*9 + 2];
      y_022[((idx_batch)*chan + idx_chan)*9 + 3] = l_0[((idx_batch)*chan + idx_chan)*1 + 0]*r_2[((idx_batch)*chan + idx_chan)*9 + 3];
      y_022[((idx_batch)*chan + idx_chan)*9 + 4] = l_0[((idx_batch)*chan + idx_chan)*1 + 0]*r_2[((idx_batch)*chan + idx_chan)*9 + 4];
      y_022[((idx_batch)*chan + idx_chan)*9 + 5] = l_0[((idx_batch)*chan + idx_chan)*1 + 0]*r_2[((idx_batch)*chan + idx_chan)*9 + 5];
      y_022[((idx_batch)*chan + idx_chan)*9 + 6] = l_0[((idx_batch)*chan + idx_chan)*1 + 0]*r_2[((idx_batch)*chan + idx_chan)*9 + 6];
      y_022[((idx_batch)*chan + idx_chan)*9 + 7] = l_0[((idx_batch)*chan + idx_chan)*1 + 0]*r_2[((idx_batch)*chan + idx_chan)*9 + 7];
      y_022[((idx_batch)*chan + idx_chan)*9 + 8] = l_0[((idx_batch)*chan + idx_chan)*1 + 0]*r_2[((idx_batch)*chan + idx_chan)*9 + 8];
      y_202[((idx_batch)*chan + idx_chan)*9 + 0] = l_2[((idx_batch)*chan + idx_chan)*9 + 0]*r_0[((idx_batch)*chan + idx_chan)*1 + 0];
      y_202[((idx_batch)*chan + idx_chan)*9 + 1] = l_2[((idx_batch)*chan + idx_chan)*9 + 1]*r_0[((idx_batch)*chan + idx_chan)*1 + 0];
      y_202[((idx_batch)*chan + idx_chan)*9 + 2] = l_2[((idx_batch)*chan + idx_chan)*9 + 2]*r_0[((idx_batch)*chan + idx_chan)*1 + 0];
      y_202[((idx_batch)*chan + idx_chan)*9 + 3] = l_2[((idx_batch)*chan + idx_chan)*9 + 3]*r_0[((idx_batch)*chan + idx_chan)*1 + 0];
      y_202[((idx_batch)*chan + idx_chan)*9 + 4] = l_2[((idx_batch)*chan + idx_chan)*9 + 4]*r_0[((idx_batch)*chan + idx_chan)*1 + 0];
      y_202[((idx_batch)*chan + idx_chan)*9 + 5] = l_2[((idx_batch)*chan + idx_chan)*9 + 5]*r_0[((idx_batch)*chan + idx_chan)*1 + 0];
      y_202[((idx_batch)*chan + idx_chan)*9 + 6] = l_2[((idx_batch)*chan + idx_chan)*9 + 6]*r_0[((idx_batch)*chan + idx_chan)*1 + 0];
      y_202[((idx_batch)*chan + idx_chan)*9 + 7] = l_2[((idx_batch)*chan + idx_chan)*9 + 7]*r_0[((idx_batch)*chan + idx_chan)*1 + 0];
      y_202[((idx_batch)*chan + idx_chan)*9 + 8] = l_2[((idx_batch)*chan + idx_chan)*9 + 8]*r_0[((idx_batch)*chan + idx_chan)*1 + 0];
      y_112[((idx_batch)*chan + idx_chan)*9 + 0] = l_1[((idx_batch)*chan + idx_chan)*3 + 0]*r_1[((idx_batch)*chan + idx_chan)*3 + 0];
      y_112[((idx_batch)*chan + idx_chan)*9 + 1] = l_1[((idx_batch)*chan + idx_chan)*3 + 0]*r_1[((idx_batch)*chan + idx_chan)*3 + 1];
      y_112[((idx_batch)*chan + idx_chan)*9 + 2] = l_1[((idx_batch)*chan + idx_chan)*3 + 0]*r_1[((idx_batch)*chan + idx_chan)*3 + 2];
      y_112[((idx_batch)*chan + idx_chan)*9 + 3] = l_1[((idx_batch)*chan + idx_chan)*3 + 1]*r_1[((idx_batch)*chan + idx_chan)*3 + 0];
      y_112[((idx_batch)*chan + idx_chan)*9 + 4] = l_1[((idx_batch)*chan + idx_chan)*3 + 1]*r_1[((idx_batch)*chan + idx_chan)*3 + 1];
      y_112[((idx_batch)*chan + idx_chan)*9 + 5] = l_1[((idx_batch)*chan + idx_chan)*3 + 1]*r_1[((idx_batch)*chan + idx_chan)*3 + 2];
      y_112[((idx_batch)*chan + idx_chan)*9 + 6] = l_1[((idx_batch)*chan + idx_chan)*3 + 2]*r_1[((idx_batch)*chan + idx_chan)*3 + 0];
      y_112[((idx_batch)*chan + idx_chan)*9 + 7] = l_1[((idx_batch)*chan + idx_chan)*3 + 2]*r_1[((idx_batch)*chan + idx_chan)*3 + 1];
      y_112[((idx_batch)*chan + idx_chan)*9 + 8] = l_1[((idx_batch)*chan + idx_chan)*3 + 2]*r_1[((idx_batch)*chan + idx_chan)*3 + 2];
      y_222[((idx_batch)*chan + idx_chan)*9 + 0] = l_2[((idx_batch)*chan + idx_chan)*9 + 0]*r_2[((idx_batch)*chan + idx_chan)*9 + 0] + l_2[((idx_batch)*chan + idx_chan)*9 + 1]*r_2[((idx_batch)*chan + idx_chan)*9 + 1] + l_2[((idx_batch)*chan + idx_chan)*9 + 2]*r_2[((idx_batch)*chan + idx_chan)*9 + 2];
      y_222[((idx_batch)*chan + idx_chan)*9 + 1] = l_2[((idx_batch)*chan + idx_chan)*9 + 0]*r_2[((idx_batch)*chan + idx_chan)*9 + 3] + l_2[((idx_batch)*chan + idx_chan)*9 + 1]*r_2[((idx_batch)*chan + idx_chan)*9 + 4] + l_2[((idx_batch)*chan + idx_chan)*9 + 2]*r_2[((idx_batch)*chan + idx_chan)*9 + 5];
      y_222[((idx_batch)*chan + idx_chan)*9 + 2] = l_2[((idx_batch)*chan + idx_chan)*9 + 0]*r_2[((idx_batch)*chan + idx_chan)*9 + 6] + l_2[((idx_batch)*chan + idx_chan)*9 + 1]*r_2[((idx_batch)*chan + idx_chan)*9 + 7] + l_2[((idx_batch)*chan + idx_chan)*9 + 2]*r_2[((idx_batch)*chan + idx_chan)*9 + 8];
      y_222[((idx_batch)*chan + idx_chan)*9 + 3] = l_2[((idx_batch)*chan + idx_chan)*9 + 3]*r_2[((idx_batch)*chan + idx_chan)*9 + 0] + l_2[((idx_batch)*chan + idx_chan)*9 + 4]*r_2[((idx_batch)*chan + idx_chan)*9 + 1] + l_2[((idx_batch)*chan + idx_chan)*9 + 5]*r_2[((idx_batch)*chan + idx_chan)*9 + 2];
      y_222[((idx_batch)*chan + idx_chan)*9 + 4] = l_2[((idx_batch)*chan + idx_chan)*9 + 3]*r_2[((idx_batch)*chan + idx_chan)*9 + 3] + l_2[((idx_batch)*chan + idx_chan)*9 + 4]*r_2[((idx_batch)*chan + idx_chan)*9 + 4] + l_2[((idx_batch)*chan + idx_chan)*9 + 5]*r_2[((idx_batch)*chan + idx_chan)*9 + 5];
      y_222[((idx_batch)*chan + idx_chan)*9 + 5] = l_2[((idx_batch)*chan + idx_chan)*9 + 3]*r_2[((idx_batch)*chan + idx_chan)*9 + 6] + l_2[((idx_batch)*chan + idx_chan)*9 + 4]*r_2[((idx_batch)*chan + idx_chan)*9 + 7] + l_2[((idx_batch)*chan + idx_chan)*9 + 5]*r_2[((idx_batch)*chan + idx_chan)*9 + 8];
      y_222[((idx_batch)*chan + idx_chan)*9 + 6] = l_2[((idx_batch)*chan + idx_chan)*9 + 6]*r_2[((idx_batch)*chan + idx_chan)*9 + 0] + l_2[((idx_batch)*chan + idx_chan)*9 + 7]*r_2[((idx_batch)*chan + idx_chan)*9 + 1] + l_2[((idx_batch)*chan + idx_chan)*9 + 8]*r_2[((idx_batch)*chan + idx_chan)*9 + 2];
      y_222[((idx_batch)*chan + idx_chan)*9 + 7] = l_2[((idx_batch)*chan + idx_chan)*9 + 6]*r_2[((idx_batch)*chan + idx_chan)*9 + 3] + l_2[((idx_batch)*chan + idx_chan)*9 + 7]*r_2[((idx_batch)*chan + idx_chan)*9 + 4] + l_2[((idx_batch)*chan + idx_chan)*9 + 8]*r_2[((idx_batch)*chan + idx_chan)*9 + 5];
      y_222[((idx_batch)*chan + idx_chan)*9 + 8] = l_2[((idx_batch)*chan + idx_chan)*9 + 6]*r_2[((idx_batch)*chan + idx_chan)*9 + 6] + l_2[((idx_batch)*chan + idx_chan)*9 + 7]*r_2[((idx_batch)*chan + idx_chan)*9 + 7] + l_2[((idx_batch)*chan + idx_chan)*9 + 8]*r_2[((idx_batch)*chan + idx_chan)*9 + 8];
      y_111[((idx_batch)*chan + idx_chan)*3 + 0] = l_1[((idx_batch)*chan + idx_chan)*3 + 1]*r_1[((idx_batch)*chan + idx_chan)*3 + 2] + (-1)*l_1[((idx_batch)*chan + idx_chan)*3 + 2]*r_1[((idx_batch)*chan + idx_chan)*3 + 1];
      y_111[((idx_batch)*chan + idx_chan)*3 + 1] = (-1)*l_1[((idx_batch)*chan + idx_chan)*3 + 0]*r_1[((idx_batch)*chan + idx_chan)*3 + 2] + l_1[((idx_batch)*chan + idx_chan)*3 + 2]*r_1[((idx_batch)*chan + idx_chan)*3 + 0];
      y_111[((idx_batch)*chan + idx_chan)*3 + 2] = l_1[((idx_batch)*chan + idx_chan)*3 + 0]*r_1[((idx_batch)*chan + idx_chan)*3 + 1] + (-1)*l_1[((idx_batch)*chan + idx_chan)*3 + 1]*r_1[((idx_batch)*chan + idx_chan)*3 + 0];
      y_212[((idx_batch)*chan + idx_chan)*9 + 0] = l_2[((idx_batch)*chan + idx_chan)*9 + 1]*r_1[((idx_batch)*chan + idx_chan)*3 + 2] + (-1)*l_2[((idx_batch)*chan + idx_chan)*9 + 2]*r_1[((idx_batch)*chan + idx_chan)*3 + 1];
      y_212[((idx_batch)*chan + idx_chan)*9 + 1] = (-1)*l_2[((idx_batch)*chan + idx_chan)*9 + 0]*r_1[((idx_batch)*chan + idx_chan)*3 + 2] + l_2[((idx_batch)*chan + idx_chan)*9 + 2]*r_1[((idx_batch)*chan + idx_chan)*3 + 0];
      y_212[((idx_batch)*chan + idx_chan)*9 + 2] = l_2[((idx_batch)*chan + idx_chan)*9 + 0]*r_1[((idx_batch)*chan + idx_chan)*3 + 1] + (-1)*l_2[((idx_batch)*chan + idx_chan)*9 + 1]*r_1[((idx_batch)*chan + idx_chan)*3 + 0];
      y_212[((idx_batch)*chan + idx_chan)*9 + 3] = l_2[((idx_batch)*chan + idx_chan)*9 + 4]*r_1[((idx_batch)*chan + idx_chan)*3 + 2] + (-1)*l_2[((idx_batch)*chan + idx_chan)*9 + 5]*r_1[((idx_batch)*chan + idx_chan)*3 + 1];
      y_212[((idx_batch)*chan + idx_chan)*9 + 4] = (-1)*l_2[((idx_batch)*chan + idx_chan)*9 + 3]*r_1[((idx_batch)*chan + idx_chan)*3 + 2] + l_2[((idx_batch)*chan + idx_chan)*9 + 5]*r_1[((idx_batch)*chan + idx_chan)*3 + 0];
      y_212[((idx_batch)*chan + idx_chan)*9 + 5] = l_2[((idx_batch)*chan + idx_chan)*9 + 3]*r_1[((idx_batch)*chan + idx_chan)*3 + 1] + (-1)*l_2[((idx_batch)*chan + idx_chan)*9 + 4]*r_1[((idx_batch)*chan + idx_chan)*3 + 0];
      y_212[((idx_batch)*chan + idx_chan)*9 + 6] = l_2[((idx_batch)*chan + idx_chan)*9 + 7]*r_1[((idx_batch)*chan + idx_chan)*3 + 2] + (-1)*l_2[((idx_batch)*chan + idx_chan)*9 + 8]*r_1[((idx_batch)*chan + idx_chan)*3 + 1];
      y_212[((idx_batch)*chan + idx_chan)*9 + 7] = (-1)*l_2[((idx_batch)*chan + idx_chan)*9 + 6]*r_1[((idx_batch)*chan + idx_chan)*3 + 2] + l_2[((idx_batch)*chan + idx_chan)*9 + 8]*r_1[((idx_batch)*chan + idx_chan)*3 + 0];
      y_212[((idx_batch)*chan + idx_chan)*9 + 8] = l_2[((idx_batch)*chan + idx_chan)*9 + 6]*r_1[((idx_batch)*chan + idx_chan)*3 + 1] + (-1)*l_2[((idx_batch)*chan + idx_chan)*9 + 7]*r_1[((idx_batch)*chan + idx_chan)*3 + 0];
    }
  }
}


void bee_fwd(
    int batch, int chan,
    const float* l_0, const float* l_1, const float* l_2, const float* r_0, const float* r_1, const float* r_2,
    float* y_000, float* y_110, float* y_220, float* y_011, float* y_101, float* y_121, float* y_211, float* y_022, float* y_202, float* y_112, float* y_222, float* y_111, float* y_212) {
  
  
  int sharedmemsz = 0;
  dim3 gridsz = dim3(batch);
  dim3 blocksz = dim3(WARPSZ);
  bee_fwd_kern<<<gridsz, blocksz, sharedmemsz*sizeof(float)>>>(
      
      batch, chan,
      l_0, l_1, l_2, r_0, r_1, r_2,
      y_000, y_110, y_220, y_011, y_101, y_121, y_211, y_022, y_202, y_112, y_222, y_111, y_212);
  
}


__global__
void bee_bwl_kern(
    // <<<(batch), (WARPSZ)>>>
    
    int batch, int chan,
    const float* r_0, const float* r_1, const float* r_2, const float* dy_000, const float* dy_110, const float* dy_220, const float* dy_011, const float* dy_101, const float* dy_121, const float* dy_211, const float* dy_022, const float* dy_202, const float* dy_112, const float* dy_222, const float* dy_111, const float* dy_212,
    float* __restrict__ dl_0, float* __restrict__ dl_1, float* __restrict__ dl_2) {
  extern __shared__ float s[];
  for (int idx_batch = blockIdx.x; idx_batch < batch; idx_batch += gridDim.x) {
    for (int idx_chan = threadIdx.x; idx_chan < chan; idx_chan += blockDim.x) {
      float accum_0_0 = 0.0;
      float accum_1_0 = 0.0;
      float accum_1_1 = 0.0;
      float accum_1_2 = 0.0;
      float accum_2_0 = 0.0;
      float accum_2_1 = 0.0;
      float accum_2_2 = 0.0;
      float accum_2_3 = 0.0;
      float accum_2_4 = 0.0;
      float accum_2_5 = 0.0;
      float accum_2_6 = 0.0;
      float accum_2_7 = 0.0;
      float accum_2_8 = 0.0;
      accum_0_0 += r_0[((idx_batch)*chan + idx_chan)*1 + 0]*dy_000[((idx_batch)*chan + idx_chan)*1 + 0];
      accum_1_0 += r_1[((idx_batch)*chan + idx_chan)*3 + 0]*dy_110[((idx_batch)*chan + idx_chan)*1 + 0];
      accum_1_1 += r_1[((idx_batch)*chan + idx_chan)*3 + 1]*dy_110[((idx_batch)*chan + idx_chan)*1 + 0];
      accum_1_2 += r_1[((idx_batch)*chan + idx_chan)*3 + 2]*dy_110[((idx_batch)*chan + idx_chan)*1 + 0];
      accum_2_0 += r_2[((idx_batch)*chan + idx_chan)*9 + 0]*dy_220[((idx_batch)*chan + idx_chan)*1 + 0];
      accum_2_1 += r_2[((idx_batch)*chan + idx_chan)*9 + 1]*dy_220[((idx_batch)*chan + idx_chan)*1 + 0];
      accum_2_2 += r_2[((idx_batch)*chan + idx_chan)*9 + 2]*dy_220[((idx_batch)*chan + idx_chan)*1 + 0];
      accum_2_3 += r_2[((idx_batch)*chan + idx_chan)*9 + 3]*dy_220[((idx_batch)*chan + idx_chan)*1 + 0];
      accum_2_4 += r_2[((idx_batch)*chan + idx_chan)*9 + 4]*dy_220[((idx_batch)*chan + idx_chan)*1 + 0];
      accum_2_5 += r_2[((idx_batch)*chan + idx_chan)*9 + 5]*dy_220[((idx_batch)*chan + idx_chan)*1 + 0];
      accum_2_6 += r_2[((idx_batch)*chan + idx_chan)*9 + 6]*dy_220[((idx_batch)*chan + idx_chan)*1 + 0];
      accum_2_7 += r_2[((idx_batch)*chan + idx_chan)*9 + 7]*dy_220[((idx_batch)*chan + idx_chan)*1 + 0];
      accum_2_8 += r_2[((idx_batch)*chan + idx_chan)*9 + 8]*dy_220[((idx_batch)*chan + idx_chan)*1 + 0];
      accum_0_0 += r_1[((idx_batch)*chan + idx_chan)*3 + 0]*dy_011[((idx_batch)*chan + idx_chan)*3 + 0] + r_1[((idx_batch)*chan + idx_chan)*3 + 1]*dy_011[((idx_batch)*chan + idx_chan)*3 + 1] + r_1[((idx_batch)*chan + idx_chan)*3 + 2]*dy_011[((idx_batch)*chan + idx_chan)*3 + 2];
      accum_1_0 += r_0[((idx_batch)*chan + idx_chan)*1 + 0]*dy_101[((idx_batch)*chan + idx_chan)*3 + 0];
      accum_1_1 += r_0[((idx_batch)*chan + idx_chan)*1 + 0]*dy_101[((idx_batch)*chan + idx_chan)*3 + 1];
      accum_1_2 += r_0[((idx_batch)*chan + idx_chan)*1 + 0]*dy_101[((idx_batch)*chan + idx_chan)*3 + 2];
      accum_1_0 += r_2[((idx_batch)*chan + idx_chan)*9 + 0]*dy_121[((idx_batch)*chan + idx_chan)*3 + 0] + r_2[((idx_batch)*chan + idx_chan)*9 + 3]*dy_121[((idx_batch)*chan + idx_chan)*3 + 1] + r_2[((idx_batch)*chan + idx_chan)*9 + 6]*dy_121[((idx_batch)*chan + idx_chan)*3 + 2];
      accum_1_1 += r_2[((idx_batch)*chan + idx_chan)*9 + 1]*dy_121[((idx_batch)*chan + idx_chan)*3 + 0] + r_2[((idx_batch)*chan + idx_chan)*9 + 4]*dy_121[((idx_batch)*chan + idx_chan)*3 + 1] + r_2[((idx_batch)*chan + idx_chan)*9 + 7]*dy_121[((idx_batch)*chan + idx_chan)*3 + 2];
      accum_1_2 += r_2[((idx_batch)*chan + idx_chan)*9 + 2]*dy_121[((idx_batch)*chan + idx_chan)*3 + 0] + r_2[((idx_batch)*chan + idx_chan)*9 + 5]*dy_121[((idx_batch)*chan + idx_chan)*3 + 1] + r_2[((idx_batch)*chan + idx_chan)*9 + 8]*dy_121[((idx_batch)*chan + idx_chan)*3 + 2];
      accum_2_0 += r_1[((idx_batch)*chan + idx_chan)*3 + 0]*dy_211[((idx_batch)*chan + idx_chan)*3 + 0];
      accum_2_1 += r_1[((idx_batch)*chan + idx_chan)*3 + 1]*dy_211[((idx_batch)*chan + idx_chan)*3 + 0];
      accum_2_2 += r_1[((idx_batch)*chan + idx_chan)*3 + 2]*dy_211[((idx_batch)*chan + idx_chan)*3 + 0];
      accum_2_3 += r_1[((idx_batch)*chan + idx_chan)*3 + 0]*dy_211[((idx_batch)*chan + idx_chan)*3 + 1];
      accum_2_4 += r_1[((idx_batch)*chan + idx_chan)*3 + 1]*dy_211[((idx_batch)*chan + idx_chan)*3 + 1];
      accum_2_5 += r_1[((idx_batch)*chan + idx_chan)*3 + 2]*dy_211[((idx_batch)*chan + idx_chan)*3 + 1];
      accum_2_6 += r_1[((idx_batch)*chan + idx_chan)*3 + 0]*dy_211[((idx_batch)*chan + idx_chan)*3 + 2];
      accum_2_7 += r_1[((idx_batch)*chan + idx_chan)*3 + 1]*dy_211[((idx_batch)*chan + idx_chan)*3 + 2];
      accum_2_8 += r_1[((idx_batch)*chan + idx_chan)*3 + 2]*dy_211[((idx_batch)*chan + idx_chan)*3 + 2];
      accum_0_0 += r_2[((idx_batch)*chan + idx_chan)*9 + 0]*dy_022[((idx_batch)*chan + idx_chan)*9 + 0] + r_2[((idx_batch)*chan + idx_chan)*9 + 1]*dy_022[((idx_batch)*chan + idx_chan)*9 + 1] + r_2[((idx_batch)*chan + idx_chan)*9 + 2]*dy_022[((idx_batch)*chan + idx_chan)*9 + 2] + r_2[((idx_batch)*chan + idx_chan)*9 + 3]*dy_022[((idx_batch)*chan + idx_chan)*9 + 3] + r_2[((idx_batch)*chan + idx_chan)*9 + 4]*dy_022[((idx_batch)*chan + idx_chan)*9 + 4] + r_2[((idx_batch)*chan + idx_chan)*9 + 5]*dy_022[((idx_batch)*chan + idx_chan)*9 + 5] + r_2[((idx_batch)*chan + idx_chan)*9 + 6]*dy_022[((idx_batch)*chan + idx_chan)*9 + 6] + r_2[((idx_batch)*chan + idx_chan)*9 + 7]*dy_022[((idx_batch)*chan + idx_chan)*9 + 7] + r_2[((idx_batch)*chan + idx_chan)*9 + 8]*dy_022[((idx_batch)*chan + idx_chan)*9 + 8];
      accum_2_0 += r_0[((idx_batch)*chan + idx_chan)*1 + 0]*dy_202[((idx_batch)*chan + idx_chan)*9 + 0];
      accum_2_1 += r_0[((idx_batch)*chan + idx_chan)*1 + 0]*dy_202[((idx_batch)*chan + idx_chan)*9 + 1];
      accum_2_2 += r_0[((idx_batch)*chan + idx_chan)*1 + 0]*dy_202[((idx_batch)*chan + idx_chan)*9 + 2];
      accum_2_3 += r_0[((idx_batch)*chan + idx_chan)*1 + 0]*dy_202[((idx_batch)*chan + idx_chan)*9 + 3];
      accum_2_4 += r_0[((idx_batch)*chan + idx_chan)*1 + 0]*dy_202[((idx_batch)*chan + idx_chan)*9 + 4];
      accum_2_5 += r_0[((idx_batch)*chan + idx_chan)*1 + 0]*dy_202[((idx_batch)*chan + idx_chan)*9 + 5];
      accum_2_6 += r_0[((idx_batch)*chan + idx_chan)*1 + 0]*dy_202[((idx_batch)*chan + idx_chan)*9 + 6];
      accum_2_7 += r_0[((idx_batch)*chan + idx_chan)*1 + 0]*dy_202[((idx_batch)*chan + idx_chan)*9 + 7];
      accum_2_8 += r_0[((idx_batch)*chan + idx_chan)*1 + 0]*dy_202[((idx_batch)*chan + idx_chan)*9 + 8];
      accum_1_0 += r_1[((idx_batch)*chan + idx_chan)*3 + 0]*dy_112[((idx_batch)*chan + idx_chan)*9 + 0] + r_1[((idx_batch)*chan + idx_chan)*3 + 1]*dy_112[((idx_batch)*chan + idx_chan)*9 + 1] + r_1[((idx_batch)*chan + idx_chan)*3 + 2]*dy_112[((idx_batch)*chan + idx_chan)*9 + 2];
      accum_1_1 += r_1[((idx_batch)*chan + idx_chan)*3 + 0]*dy_112[((idx_batch)*chan + idx_chan)*9 + 3] + r_1[((idx_batch)*chan + idx_chan)*3 + 1]*dy_112[((idx_batch)*chan + idx_chan)*9 + 4] + r_1[((idx_batch)*chan + idx_chan)*3 + 2]*dy_112[((idx_batch)*chan + idx_chan)*9 + 5];
      accum_1_2 += r_1[((idx_batch)*chan + idx_chan)*3 + 0]*dy_112[((idx_batch)*chan + idx_chan)*9 + 6] + r_1[((idx_batch)*chan + idx_chan)*3 + 1]*dy_112[((idx_batch)*chan + idx_chan)*9 + 7] + r_1[((idx_batch)*chan + idx_chan)*3 + 2]*dy_112[((idx_batch)*chan + idx_chan)*9 + 8];
      accum_2_0 += r_2[((idx_batch)*chan + idx_chan)*9 + 0]*dy_222[((idx_batch)*chan + idx_chan)*9 + 0] + r_2[((idx_batch)*chan + idx_chan)*9 + 3]*dy_222[((idx_batch)*chan + idx_chan)*9 + 1] + r_2[((idx_batch)*chan + idx_chan)*9 + 6]*dy_222[((idx_batch)*chan + idx_chan)*9 + 2];
      accum_2_1 += r_2[((idx_batch)*chan + idx_chan)*9 + 1]*dy_222[((idx_batch)*chan + idx_chan)*9 + 0] + r_2[((idx_batch)*chan + idx_chan)*9 + 4]*dy_222[((idx_batch)*chan + idx_chan)*9 + 1] + r_2[((idx_batch)*chan + idx_chan)*9 + 7]*dy_222[((idx_batch)*chan + idx_chan)*9 + 2];
      accum_2_2 += r_2[((idx_batch)*chan + idx_chan)*9 + 2]*dy_222[((idx_batch)*chan + idx_chan)*9 + 0] + r_2[((idx_batch)*chan + idx_chan)*9 + 5]*dy_222[((idx_batch)*chan + idx_chan)*9 + 1] + r_2[((idx_batch)*chan + idx_chan)*9 + 8]*dy_222[((idx_batch)*chan + idx_chan)*9 + 2];
      accum_2_3 += r_2[((idx_batch)*chan + idx_chan)*9 + 0]*dy_222[((idx_batch)*chan + idx_chan)*9 + 3] + r_2[((idx_batch)*chan + idx_chan)*9 + 3]*dy_222[((idx_batch)*chan + idx_chan)*9 + 4] + r_2[((idx_batch)*chan + idx_chan)*9 + 6]*dy_222[((idx_batch)*chan + idx_chan)*9 + 5];
      accum_2_4 += r_2[((idx_batch)*chan + idx_chan)*9 + 1]*dy_222[((idx_batch)*chan + idx_chan)*9 + 3] + r_2[((idx_batch)*chan + idx_chan)*9 + 4]*dy_222[((idx_batch)*chan + idx_chan)*9 + 4] + r_2[((idx_batch)*chan + idx_chan)*9 + 7]*dy_222[((idx_batch)*chan + idx_chan)*9 + 5];
      accum_2_5 += r_2[((idx_batch)*chan + idx_chan)*9 + 2]*dy_222[((idx_batch)*chan + idx_chan)*9 + 3] + r_2[((idx_batch)*chan + idx_chan)*9 + 5]*dy_222[((idx_batch)*chan + idx_chan)*9 + 4] + r_2[((idx_batch)*chan + idx_chan)*9 + 8]*dy_222[((idx_batch)*chan + idx_chan)*9 + 5];
      accum_2_6 += r_2[((idx_batch)*chan + idx_chan)*9 + 0]*dy_222[((idx_batch)*chan + idx_chan)*9 + 6] + r_2[((idx_batch)*chan + idx_chan)*9 + 3]*dy_222[((idx_batch)*chan + idx_chan)*9 + 7] + r_2[((idx_batch)*chan + idx_chan)*9 + 6]*dy_222[((idx_batch)*chan + idx_chan)*9 + 8];
      accum_2_7 += r_2[((idx_batch)*chan + idx_chan)*9 + 1]*dy_222[((idx_batch)*chan + idx_chan)*9 + 6] + r_2[((idx_batch)*chan + idx_chan)*9 + 4]*dy_222[((idx_batch)*chan + idx_chan)*9 + 7] + r_2[((idx_batch)*chan + idx_chan)*9 + 7]*dy_222[((idx_batch)*chan + idx_chan)*9 + 8];
      accum_2_8 += r_2[((idx_batch)*chan + idx_chan)*9 + 2]*dy_222[((idx_batch)*chan + idx_chan)*9 + 6] + r_2[((idx_batch)*chan + idx_chan)*9 + 5]*dy_222[((idx_batch)*chan + idx_chan)*9 + 7] + r_2[((idx_batch)*chan + idx_chan)*9 + 8]*dy_222[((idx_batch)*chan + idx_chan)*9 + 8];
      accum_1_0 += (-1)*r_1[((idx_batch)*chan + idx_chan)*3 + 2]*dy_111[((idx_batch)*chan + idx_chan)*3 + 1] + r_1[((idx_batch)*chan + idx_chan)*3 + 1]*dy_111[((idx_batch)*chan + idx_chan)*3 + 2];
      accum_1_1 += r_1[((idx_batch)*chan + idx_chan)*3 + 2]*dy_111[((idx_batch)*chan + idx_chan)*3 + 0] + (-1)*r_1[((idx_batch)*chan + idx_chan)*3 + 0]*dy_111[((idx_batch)*chan + idx_chan)*3 + 2];
      accum_1_2 += (-1)*r_1[((idx_batch)*chan + idx_chan)*3 + 1]*dy_111[((idx_batch)*chan + idx_chan)*3 + 0] + r_1[((idx_batch)*chan + idx_chan)*3 + 0]*dy_111[((idx_batch)*chan + idx_chan)*3 + 1];
      accum_2_0 += (-1)*r_1[((idx_batch)*chan + idx_chan)*3 + 2]*dy_212[((idx_batch)*chan + idx_chan)*9 + 1] + r_1[((idx_batch)*chan + idx_chan)*3 + 1]*dy_212[((idx_batch)*chan + idx_chan)*9 + 2];
      accum_2_1 += r_1[((idx_batch)*chan + idx_chan)*3 + 2]*dy_212[((idx_batch)*chan + idx_chan)*9 + 0] + (-1)*r_1[((idx_batch)*chan + idx_chan)*3 + 0]*dy_212[((idx_batch)*chan + idx_chan)*9 + 2];
      accum_2_2 += (-1)*r_1[((idx_batch)*chan + idx_chan)*3 + 1]*dy_212[((idx_batch)*chan + idx_chan)*9 + 0] + r_1[((idx_batch)*chan + idx_chan)*3 + 0]*dy_212[((idx_batch)*chan + idx_chan)*9 + 1];
      accum_2_3 += (-1)*r_1[((idx_batch)*chan + idx_chan)*3 + 2]*dy_212[((idx_batch)*chan + idx_chan)*9 + 4] + r_1[((idx_batch)*chan + idx_chan)*3 + 1]*dy_212[((idx_batch)*chan + idx_chan)*9 + 5];
      accum_2_4 += r_1[((idx_batch)*chan + idx_chan)*3 + 2]*dy_212[((idx_batch)*chan + idx_chan)*9 + 3] + (-1)*r_1[((idx_batch)*chan + idx_chan)*3 + 0]*dy_212[((idx_batch)*chan + idx_chan)*9 + 5];
      accum_2_5 += (-1)*r_1[((idx_batch)*chan + idx_chan)*3 + 1]*dy_212[((idx_batch)*chan + idx_chan)*9 + 3] + r_1[((idx_batch)*chan + idx_chan)*3 + 0]*dy_212[((idx_batch)*chan + idx_chan)*9 + 4];
      accum_2_6 += (-1)*r_1[((idx_batch)*chan + idx_chan)*3 + 2]*dy_212[((idx_batch)*chan + idx_chan)*9 + 7] + r_1[((idx_batch)*chan + idx_chan)*3 + 1]*dy_212[((idx_batch)*chan + idx_chan)*9 + 8];
      accum_2_7 += r_1[((idx_batch)*chan + idx_chan)*3 + 2]*dy_212[((idx_batch)*chan + idx_chan)*9 + 6] + (-1)*r_1[((idx_batch)*chan + idx_chan)*3 + 0]*dy_212[((idx_batch)*chan + idx_chan)*9 + 8];
      accum_2_8 += (-1)*r_1[((idx_batch)*chan + idx_chan)*3 + 1]*dy_212[((idx_batch)*chan + idx_chan)*9 + 6] + r_1[((idx_batch)*chan + idx_chan)*3 + 0]*dy_212[((idx_batch)*chan + idx_chan)*9 + 7];
      dl_0[((idx_batch)*chan + idx_chan)*1 + 0] = accum_0_0;
      dl_1[((idx_batch)*chan + idx_chan)*3 + 0] = accum_1_0;
      dl_1[((idx_batch)*chan + idx_chan)*3 + 1] = accum_1_1;
      dl_1[((idx_batch)*chan + idx_chan)*3 + 2] = accum_1_2;
      dl_2[((idx_batch)*chan + idx_chan)*9 + 0] = accum_2_0;
      dl_2[((idx_batch)*chan + idx_chan)*9 + 1] = accum_2_1;
      dl_2[((idx_batch)*chan + idx_chan)*9 + 2] = accum_2_2;
      dl_2[((idx_batch)*chan + idx_chan)*9 + 3] = accum_2_3;
      dl_2[((idx_batch)*chan + idx_chan)*9 + 4] = accum_2_4;
      dl_2[((idx_batch)*chan + idx_chan)*9 + 5] = accum_2_5;
      dl_2[((idx_batch)*chan + idx_chan)*9 + 6] = accum_2_6;
      dl_2[((idx_batch)*chan + idx_chan)*9 + 7] = accum_2_7;
      dl_2[((idx_batch)*chan + idx_chan)*9 + 8] = accum_2_8;
    }
  }
}


void bee_bwl(
    int batch, int chan,
    const float* r_0, const float* r_1, const float* r_2, const float* dy_000, const float* dy_110, const float* dy_220, const float* dy_011, const float* dy_101, const float* dy_121, const float* dy_211, const float* dy_022, const float* dy_202, const float* dy_112, const float* dy_222, const float* dy_111, const float* dy_212,
    float* dl_0, float* dl_1, float* dl_2) {
  
  
  int sharedmemsz = 0;
  dim3 gridsz = dim3(batch);
  dim3 blocksz = dim3(WARPSZ);
  bee_bwl_kern<<<gridsz, blocksz, sharedmemsz*sizeof(float)>>>(
      
      batch, chan,
      r_0, r_1, r_2, dy_000, dy_110, dy_220, dy_011, dy_101, dy_121, dy_211, dy_022, dy_202, dy_112, dy_222, dy_111, dy_212,
      dl_0, dl_1, dl_2);
  
}


__global__
void bee_bwr_kern(
    // <<<(batch), (WARPSZ)>>>
    
    int batch, int chan,
    const float* l_0, const float* l_1, const float* l_2, const float* dy_000, const float* dy_110, const float* dy_220, const float* dy_011, const float* dy_101, const float* dy_121, const float* dy_211, const float* dy_022, const float* dy_202, const float* dy_112, const float* dy_222, const float* dy_111, const float* dy_212,
    float* __restrict__ dr_0, float* __restrict__ dr_1, float* __restrict__ dr_2) {
  extern __shared__ float s[];
  for (int idx_batch = blockIdx.x; idx_batch < batch; idx_batch += gridDim.x) {
    for (int idx_chan = threadIdx.x; idx_chan < chan; idx_chan += blockDim.x) {
      float accum_0_0 = 0.0;
      float accum_1_0 = 0.0;
      float accum_1_1 = 0.0;
      float accum_1_2 = 0.0;
      float accum_2_0 = 0.0;
      float accum_2_1 = 0.0;
      float accum_2_2 = 0.0;
      float accum_2_3 = 0.0;
      float accum_2_4 = 0.0;
      float accum_2_5 = 0.0;
      float accum_2_6 = 0.0;
      float accum_2_7 = 0.0;
      float accum_2_8 = 0.0;
      accum_0_0 += l_0[((idx_batch)*chan + idx_chan)*1 + 0]*dy_000[((idx_batch)*chan + idx_chan)*1 + 0];
      accum_1_0 += l_1[((idx_batch)*chan + idx_chan)*3 + 0]*dy_110[((idx_batch)*chan + idx_chan)*1 + 0];
      accum_1_1 += l_1[((idx_batch)*chan + idx_chan)*3 + 1]*dy_110[((idx_batch)*chan + idx_chan)*1 + 0];
      accum_1_2 += l_1[((idx_batch)*chan + idx_chan)*3 + 2]*dy_110[((idx_batch)*chan + idx_chan)*1 + 0];
      accum_2_0 += l_2[((idx_batch)*chan + idx_chan)*9 + 0]*dy_220[((idx_batch)*chan + idx_chan)*1 + 0];
      accum_2_1 += l_2[((idx_batch)*chan + idx_chan)*9 + 1]*dy_220[((idx_batch)*chan + idx_chan)*1 + 0];
      accum_2_2 += l_2[((idx_batch)*chan + idx_chan)*9 + 2]*dy_220[((idx_batch)*chan + idx_chan)*1 + 0];
      accum_2_3 += l_2[((idx_batch)*chan + idx_chan)*9 + 3]*dy_220[((idx_batch)*chan + idx_chan)*1 + 0];
      accum_2_4 += l_2[((idx_batch)*chan + idx_chan)*9 + 4]*dy_220[((idx_batch)*chan + idx_chan)*1 + 0];
      accum_2_5 += l_2[((idx_batch)*chan + idx_chan)*9 + 5]*dy_220[((idx_batch)*chan + idx_chan)*1 + 0];
      accum_2_6 += l_2[((idx_batch)*chan + idx_chan)*9 + 6]*dy_220[((idx_batch)*chan + idx_chan)*1 + 0];
      accum_2_7 += l_2[((idx_batch)*chan + idx_chan)*9 + 7]*dy_220[((idx_batch)*chan + idx_chan)*1 + 0];
      accum_2_8 += l_2[((idx_batch)*chan + idx_chan)*9 + 8]*dy_220[((idx_batch)*chan + idx_chan)*1 + 0];
      accum_1_0 += l_0[((idx_batch)*chan + idx_chan)*1 + 0]*dy_011[((idx_batch)*chan + idx_chan)*3 + 0];
      accum_1_1 += l_0[((idx_batch)*chan + idx_chan)*1 + 0]*dy_011[((idx_batch)*chan + idx_chan)*3 + 1];
      accum_1_2 += l_0[((idx_batch)*chan + idx_chan)*1 + 0]*dy_011[((idx_batch)*chan + idx_chan)*3 + 2];
      accum_0_0 += l_1[((idx_batch)*chan + idx_chan)*3 + 0]*dy_101[((idx_batch)*chan + idx_chan)*3 + 0] + l_1[((idx_batch)*chan + idx_chan)*3 + 1]*dy_101[((idx_batch)*chan + idx_chan)*3 + 1] + l_1[((idx_batch)*chan + idx_chan)*3 + 2]*dy_101[((idx_batch)*chan + idx_chan)*3 + 2];
      accum_2_0 += l_1[((idx_batch)*chan + idx_chan)*3 + 0]*dy_121[((idx_batch)*chan + idx_chan)*3 + 0];
      accum_2_1 += l_1[((idx_batch)*chan + idx_chan)*3 + 1]*dy_121[((idx_batch)*chan + idx_chan)*3 + 0];
      accum_2_2 += l_1[((idx_batch)*chan + idx_chan)*3 + 2]*dy_121[((idx_batch)*chan + idx_chan)*3 + 0];
      accum_2_3 += l_1[((idx_batch)*chan + idx_chan)*3 + 0]*dy_121[((idx_batch)*chan + idx_chan)*3 + 1];
      accum_2_4 += l_1[((idx_batch)*chan + idx_chan)*3 + 1]*dy_121[((idx_batch)*chan + idx_chan)*3 + 1];
      accum_2_5 += l_1[((idx_batch)*chan + idx_chan)*3 + 2]*dy_121[((idx_batch)*chan + idx_chan)*3 + 1];
      accum_2_6 += l_1[((idx_batch)*chan + idx_chan)*3 + 0]*dy_121[((idx_batch)*chan + idx_chan)*3 + 2];
      accum_2_7 += l_1[((idx_batch)*chan + idx_chan)*3 + 1]*dy_121[((idx_batch)*chan + idx_chan)*3 + 2];
      accum_2_8 += l_1[((idx_batch)*chan + idx_chan)*3 + 2]*dy_121[((idx_batch)*chan + idx_chan)*3 + 2];
      accum_1_0 += l_2[((idx_batch)*chan + idx_chan)*9 + 0]*dy_211[((idx_batch)*chan + idx_chan)*3 + 0] + l_2[((idx_batch)*chan + idx_chan)*9 + 3]*dy_211[((idx_batch)*chan + idx_chan)*3 + 1] + l_2[((idx_batch)*chan + idx_chan)*9 + 6]*dy_211[((idx_batch)*chan + idx_chan)*3 + 2];
      accum_1_1 += l_2[((idx_batch)*chan + idx_chan)*9 + 1]*dy_211[((idx_batch)*chan + idx_chan)*3 + 0] + l_2[((idx_batch)*chan + idx_chan)*9 + 4]*dy_211[((idx_batch)*chan + idx_chan)*3 + 1] + l_2[((idx_batch)*chan + idx_chan)*9 + 7]*dy_211[((idx_batch)*chan + idx_chan)*3 + 2];
      accum_1_2 += l_2[((idx_batch)*chan + idx_chan)*9 + 2]*dy_211[((idx_batch)*chan + idx_chan)*3 + 0] + l_2[((idx_batch)*chan + idx_chan)*9 + 5]*dy_211[((idx_batch)*chan + idx_chan)*3 + 1] + l_2[((idx_batch)*chan + idx_chan)*9 + 8]*dy_211[((idx_batch)*chan + idx_chan)*3 + 2];
      accum_2_0 += l_0[((idx_batch)*chan + idx_chan)*1 + 0]*dy_022[((idx_batch)*chan + idx_chan)*9 + 0];
      accum_2_1 += l_0[((idx_batch)*chan + idx_chan)*1 + 0]*dy_022[((idx_batch)*chan + idx_chan)*9 + 1];
      accum_2_2 += l_0[((idx_batch)*chan + idx_chan)*1 + 0]*dy_022[((idx_batch)*chan + idx_chan)*9 + 2];
      accum_2_3 += l_0[((idx_batch)*chan + idx_chan)*1 + 0]*dy_022[((idx_batch)*chan + idx_chan)*9 + 3];
      accum_2_4 += l_0[((idx_batch)*chan + idx_chan)*1 + 0]*dy_022[((idx_batch)*chan + idx_chan)*9 + 4];
      accum_2_5 += l_0[((idx_batch)*chan + idx_chan)*1 + 0]*dy_022[((idx_batch)*chan + idx_chan)*9 + 5];
      accum_2_6 += l_0[((idx_batch)*chan + idx_chan)*1 + 0]*dy_022[((idx_batch)*chan + idx_chan)*9 + 6];
      accum_2_7 += l_0[((idx_batch)*chan + idx_chan)*1 + 0]*dy_022[((idx_batch)*chan + idx_chan)*9 + 7];
      accum_2_8 += l_0[((idx_batch)*chan + idx_chan)*1 + 0]*dy_022[((idx_batch)*chan + idx_chan)*9 + 8];
      accum_0_0 += l_2[((idx_batch)*chan + idx_chan)*9 + 0]*dy_202[((idx_batch)*chan + idx_chan)*9 + 0] + l_2[((idx_batch)*chan + idx_chan)*9 + 1]*dy_202[((idx_batch)*chan + idx_chan)*9 + 1] + l_2[((idx_batch)*chan + idx_chan)*9 + 2]*dy_202[((idx_batch)*chan + idx_chan)*9 + 2] + l_2[((idx_batch)*chan + idx_chan)*9 + 3]*dy_202[((idx_batch)*chan + idx_chan)*9 + 3] + l_2[((idx_batch)*chan + idx_chan)*9 + 4]*dy_202[((idx_batch)*chan + idx_chan)*9 + 4] + l_2[((idx_batch)*chan + idx_chan)*9 + 5]*dy_202[((idx_batch)*chan + idx_chan)*9 + 5] + l_2[((idx_batch)*chan + idx_chan)*9 + 6]*dy_202[((idx_batch)*chan + idx_chan)*9 + 6] + l_2[((idx_batch)*chan + idx_chan)*9 + 7]*dy_202[((idx_batch)*chan + idx_chan)*9 + 7] + l_2[((idx_batch)*chan + idx_chan)*9 + 8]*dy_202[((idx_batch)*chan + idx_chan)*9 + 8];
      accum_1_0 += l_1[((idx_batch)*chan + idx_chan)*3 + 0]*dy_112[((idx_batch)*chan + idx_chan)*9 + 0] + l_1[((idx_batch)*chan + idx_chan)*3 + 1]*dy_112[((idx_batch)*chan + idx_chan)*9 + 3] + l_1[((idx_batch)*chan + idx_chan)*3 + 2]*dy_112[((idx_batch)*chan + idx_chan)*9 + 6];
      accum_1_1 += l_1[((idx_batch)*chan + idx_chan)*3 + 0]*dy_112[((idx_batch)*chan + idx_chan)*9 + 1] + l_1[((idx_batch)*chan + idx_chan)*3 + 1]*dy_112[((idx_batch)*chan + idx_chan)*9 + 4] + l_1[((idx_batch)*chan + idx_chan)*3 + 2]*dy_112[((idx_batch)*chan + idx_chan)*9 + 7];
      accum_1_2 += l_1[((idx_batch)*chan + idx_chan)*3 + 0]*dy_112[((idx_batch)*chan + idx_chan)*9 + 2] + l_1[((idx_batch)*chan + idx_chan)*3 + 1]*dy_112[((idx_batch)*chan + idx_chan)*9 + 5] + l_1[((idx_batch)*chan + idx_chan)*3 + 2]*dy_112[((idx_batch)*chan + idx_chan)*9 + 8];
      accum_2_0 += l_2[((idx_batch)*chan + idx_chan)*9 + 0]*dy_222[((idx_batch)*chan + idx_chan)*9 + 0] + l_2[((idx_batch)*chan + idx_chan)*9 + 3]*dy_222[((idx_batch)*chan + idx_chan)*9 + 3] + l_2[((idx_batch)*chan + idx_chan)*9 + 6]*dy_222[((idx_batch)*chan + idx_chan)*9 + 6];
      accum_2_1 += l_2[((idx_batch)*chan + idx_chan)*9 + 1]*dy_222[((idx_batch)*chan + idx_chan)*9 + 0] + l_2[((idx_batch)*chan + idx_chan)*9 + 4]*dy_222[((idx_batch)*chan + idx_chan)*9 + 3] + l_2[((idx_batch)*chan + idx_chan)*9 + 7]*dy_222[((idx_batch)*chan + idx_chan)*9 + 6];
      accum_2_2 += l_2[((idx_batch)*chan + idx_chan)*9 + 2]*dy_222[((idx_batch)*chan + idx_chan)*9 + 0] + l_2[((idx_batch)*chan + idx_chan)*9 + 5]*dy_222[((idx_batch)*chan + idx_chan)*9 + 3] + l_2[((idx_batch)*chan + idx_chan)*9 + 8]*dy_222[((idx_batch)*chan + idx_chan)*9 + 6];
      accum_2_3 += l_2[((idx_batch)*chan + idx_chan)*9 + 0]*dy_222[((idx_batch)*chan + idx_chan)*9 + 1] + l_2[((idx_batch)*chan + idx_chan)*9 + 3]*dy_222[((idx_batch)*chan + idx_chan)*9 + 4] + l_2[((idx_batch)*chan + idx_chan)*9 + 6]*dy_222[((idx_batch)*chan + idx_chan)*9 + 7];
      accum_2_4 += l_2[((idx_batch)*chan + idx_chan)*9 + 1]*dy_222[((idx_batch)*chan + idx_chan)*9 + 1] + l_2[((idx_batch)*chan + idx_chan)*9 + 4]*dy_222[((idx_batch)*chan + idx_chan)*9 + 4] + l_2[((idx_batch)*chan + idx_chan)*9 + 7]*dy_222[((idx_batch)*chan + idx_chan)*9 + 7];
      accum_2_5 += l_2[((idx_batch)*chan + idx_chan)*9 + 2]*dy_222[((idx_batch)*chan + idx_chan)*9 + 1] + l_2[((idx_batch)*chan + idx_chan)*9 + 5]*dy_222[((idx_batch)*chan + idx_chan)*9 + 4] + l_2[((idx_batch)*chan + idx_chan)*9 + 8]*dy_222[((idx_batch)*chan + idx_chan)*9 + 7];
      accum_2_6 += l_2[((idx_batch)*chan + idx_chan)*9 + 0]*dy_222[((idx_batch)*chan + idx_chan)*9 + 2] + l_2[((idx_batch)*chan + idx_chan)*9 + 3]*dy_222[((idx_batch)*chan + idx_chan)*9 + 5] + l_2[((idx_batch)*chan + idx_chan)*9 + 6]*dy_222[((idx_batch)*chan + idx_chan)*9 + 8];
      accum_2_7 += l_2[((idx_batch)*chan + idx_chan)*9 + 1]*dy_222[((idx_batch)*chan + idx_chan)*9 + 2] + l_2[((idx_batch)*chan + idx_chan)*9 + 4]*dy_222[((idx_batch)*chan + idx_chan)*9 + 5] + l_2[((idx_batch)*chan + idx_chan)*9 + 7]*dy_222[((idx_batch)*chan + idx_chan)*9 + 8];
      accum_2_8 += l_2[((idx_batch)*chan + idx_chan)*9 + 2]*dy_222[((idx_batch)*chan + idx_chan)*9 + 2] + l_2[((idx_batch)*chan + idx_chan)*9 + 5]*dy_222[((idx_batch)*chan + idx_chan)*9 + 5] + l_2[((idx_batch)*chan + idx_chan)*9 + 8]*dy_222[((idx_batch)*chan + idx_chan)*9 + 8];
      accum_1_0 += l_1[((idx_batch)*chan + idx_chan)*3 + 2]*dy_111[((idx_batch)*chan + idx_chan)*3 + 1] + (-1)*l_1[((idx_batch)*chan + idx_chan)*3 + 1]*dy_111[((idx_batch)*chan + idx_chan)*3 + 2];
      accum_1_1 += (-1)*l_1[((idx_batch)*chan + idx_chan)*3 + 2]*dy_111[((idx_batch)*chan + idx_chan)*3 + 0] + l_1[((idx_batch)*chan + idx_chan)*3 + 0]*dy_111[((idx_batch)*chan + idx_chan)*3 + 2];
      accum_1_2 += l_1[((idx_batch)*chan + idx_chan)*3 + 1]*dy_111[((idx_batch)*chan + idx_chan)*3 + 0] + (-1)*l_1[((idx_batch)*chan + idx_chan)*3 + 0]*dy_111[((idx_batch)*chan + idx_chan)*3 + 1];
      accum_1_0 += l_2[((idx_batch)*chan + idx_chan)*9 + 2]*dy_212[((idx_batch)*chan + idx_chan)*9 + 1] + (-1)*l_2[((idx_batch)*chan + idx_chan)*9 + 1]*dy_212[((idx_batch)*chan + idx_chan)*9 + 2] + l_2[((idx_batch)*chan + idx_chan)*9 + 5]*dy_212[((idx_batch)*chan + idx_chan)*9 + 4] + (-1)*l_2[((idx_batch)*chan + idx_chan)*9 + 4]*dy_212[((idx_batch)*chan + idx_chan)*9 + 5] + l_2[((idx_batch)*chan + idx_chan)*9 + 8]*dy_212[((idx_batch)*chan + idx_chan)*9 + 7] + (-1)*l_2[((idx_batch)*chan + idx_chan)*9 + 7]*dy_212[((idx_batch)*chan + idx_chan)*9 + 8];
      accum_1_1 += (-1)*l_2[((idx_batch)*chan + idx_chan)*9 + 2]*dy_212[((idx_batch)*chan + idx_chan)*9 + 0] + l_2[((idx_batch)*chan + idx_chan)*9 + 0]*dy_212[((idx_batch)*chan + idx_chan)*9 + 2] + (-1)*l_2[((idx_batch)*chan + idx_chan)*9 + 5]*dy_212[((idx_batch)*chan + idx_chan)*9 + 3] + l_2[((idx_batch)*chan + idx_chan)*9 + 3]*dy_212[((idx_batch)*chan + idx_chan)*9 + 5] + (-1)*l_2[((idx_batch)*chan + idx_chan)*9 + 8]*dy_212[((idx_batch)*chan + idx_chan)*9 + 6] + l_2[((idx_batch)*chan + idx_chan)*9 + 6]*dy_212[((idx_batch)*chan + idx_chan)*9 + 8];
      accum_1_2 += l_2[((idx_batch)*chan + idx_chan)*9 + 1]*dy_212[((idx_batch)*chan + idx_chan)*9 + 0] + (-1)*l_2[((idx_batch)*chan + idx_chan)*9 + 0]*dy_212[((idx_batch)*chan + idx_chan)*9 + 1] + l_2[((idx_batch)*chan + idx_chan)*9 + 4]*dy_212[((idx_batch)*chan + idx_chan)*9 + 3] + (-1)*l_2[((idx_batch)*chan + idx_chan)*9 + 3]*dy_212[((idx_batch)*chan + idx_chan)*9 + 4] + l_2[((idx_batch)*chan + idx_chan)*9 + 7]*dy_212[((idx_batch)*chan + idx_chan)*9 + 6] + (-1)*l_2[((idx_batch)*chan + idx_chan)*9 + 6]*dy_212[((idx_batch)*chan + idx_chan)*9 + 7];
      dr_0[((idx_batch)*chan + idx_chan)*1 + 0] = accum_0_0;
      dr_1[((idx_batch)*chan + idx_chan)*3 + 0] = accum_1_0;
      dr_1[((idx_batch)*chan + idx_chan)*3 + 1] = accum_1_1;
      dr_1[((idx_batch)*chan + idx_chan)*3 + 2] = accum_1_2;
      dr_2[((idx_batch)*chan + idx_chan)*9 + 0] = accum_2_0;
      dr_2[((idx_batch)*chan + idx_chan)*9 + 1] = accum_2_1;
      dr_2[((idx_batch)*chan + idx_chan)*9 + 2] = accum_2_2;
      dr_2[((idx_batch)*chan + idx_chan)*9 + 3] = accum_2_3;
      dr_2[((idx_batch)*chan + idx_chan)*9 + 4] = accum_2_4;
      dr_2[((idx_batch)*chan + idx_chan)*9 + 5] = accum_2_5;
      dr_2[((idx_batch)*chan + idx_chan)*9 + 6] = accum_2_6;
      dr_2[((idx_batch)*chan + idx_chan)*9 + 7] = accum_2_7;
      dr_2[((idx_batch)*chan + idx_chan)*9 + 8] = accum_2_8;
    }
  }
}


void bee_bwr(
    int batch, int chan,
    const float* l_0, const float* l_1, const float* l_2, const float* dy_000, const float* dy_110, const float* dy_220, const float* dy_011, const float* dy_101, const float* dy_121, const float* dy_211, const float* dy_022, const float* dy_202, const float* dy_112, const float* dy_222, const float* dy_111, const float* dy_212,
    float* dr_0, float* dr_1, float* dr_2) {
  
  
  int sharedmemsz = 0;
  dim3 gridsz = dim3(batch);
  dim3 blocksz = dim3(WARPSZ);
  bee_bwr_kern<<<gridsz, blocksz, sharedmemsz*sizeof(float)>>>(
      
      batch, chan,
      l_0, l_1, l_2, dy_000, dy_110, dy_220, dy_011, dy_101, dy_121, dy_211, dy_022, dy_202, dy_112, dy_222, dy_111, dy_212,
      dr_0, dr_1, dr_2);
  
}


void set_kern_attributes() {
  hipFuncSetAttribute(reinterpret_cast<const void*>(fused_tensor_prods_example_kern), hipFuncAttributeMaxDynamicSharedMemorySize, 101376);
  hipFuncSetAttribute(reinterpret_cast<const void*>(fused_tensor_prods_example_backward_kern), hipFuncAttributeMaxDynamicSharedMemorySize, 101376);
  hipFuncSetAttribute(reinterpret_cast<const void*>(fused_tensor_prods_example_backleft_kern), hipFuncAttributeMaxDynamicSharedMemorySize, 101376);
  hipFuncSetAttribute(reinterpret_cast<const void*>(fused_tensor_prods_example_wtsback_kern), hipFuncAttributeMaxDynamicSharedMemorySize, 101376);
  hipFuncSetAttribute(reinterpret_cast<const void*>(ant16_o0_kern), hipFuncAttributeMaxDynamicSharedMemorySize, 101376);
  hipFuncSetAttribute(reinterpret_cast<const void*>(ant16_o0_backward_kern), hipFuncAttributeMaxDynamicSharedMemorySize, 101376);
  hipFuncSetAttribute(reinterpret_cast<const void*>(ant16_o0_backleft_kern), hipFuncAttributeMaxDynamicSharedMemorySize, 101376);
  hipFuncSetAttribute(reinterpret_cast<const void*>(ant16_o0_wtsback_kern), hipFuncAttributeMaxDynamicSharedMemorySize, 101376);
  hipFuncSetAttribute(reinterpret_cast<const void*>(ant16_o1_kern), hipFuncAttributeMaxDynamicSharedMemorySize, 101376);
  hipFuncSetAttribute(reinterpret_cast<const void*>(ant16_o1_backward_kern), hipFuncAttributeMaxDynamicSharedMemorySize, 101376);
  hipFuncSetAttribute(reinterpret_cast<const void*>(ant16_o1_backleft_kern), hipFuncAttributeMaxDynamicSharedMemorySize, 101376);
  hipFuncSetAttribute(reinterpret_cast<const void*>(ant16_o1_wtsback_kern), hipFuncAttributeMaxDynamicSharedMemorySize, 101376);
  hipFuncSetAttribute(reinterpret_cast<const void*>(ant16_o2_kern), hipFuncAttributeMaxDynamicSharedMemorySize, 101376);
  hipFuncSetAttribute(reinterpret_cast<const void*>(ant16_o2_backward_kern), hipFuncAttributeMaxDynamicSharedMemorySize, 101376);
  hipFuncSetAttribute(reinterpret_cast<const void*>(ant16_o2_backleft_kern), hipFuncAttributeMaxDynamicSharedMemorySize, 101376);
  hipFuncSetAttribute(reinterpret_cast<const void*>(ant16_o2_wtsback_kern), hipFuncAttributeMaxDynamicSharedMemorySize, 101376);
  hipFuncSetAttribute(reinterpret_cast<const void*>(ant16_oc_kern), hipFuncAttributeMaxDynamicSharedMemorySize, 101376);
  hipFuncSetAttribute(reinterpret_cast<const void*>(ant16_oc_backward_kern), hipFuncAttributeMaxDynamicSharedMemorySize, 101376);
  hipFuncSetAttribute(reinterpret_cast<const void*>(ant16_oc_backleft_kern), hipFuncAttributeMaxDynamicSharedMemorySize, 101376);
  hipFuncSetAttribute(reinterpret_cast<const void*>(ant16_oc_wtsback_kern), hipFuncAttributeMaxDynamicSharedMemorySize, 101376);
  hipFuncSetAttribute(reinterpret_cast<const void*>(bee_fwd_kern), hipFuncAttributeMaxDynamicSharedMemorySize, 101376);
  hipFuncSetAttribute(reinterpret_cast<const void*>(bee_bwl_kern), hipFuncAttributeMaxDynamicSharedMemorySize, 101376);
  hipFuncSetAttribute(reinterpret_cast<const void*>(bee_bwr_kern), hipFuncAttributeMaxDynamicSharedMemorySize, 101376);
}